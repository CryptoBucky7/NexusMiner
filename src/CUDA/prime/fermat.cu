#include "hip/hip_runtime.h"
/*******************************************************************************************

 Nexus Earth 2018

 [Scale Indefinitely] BlackJack. http://www.opensource.org/licenses/mit-license.php

*******************************************************************************************/
#include <CUDA/include/fermat.h>
#include <CUDA/include/util.h>
#include <CUDA/include/frame_resources.h>

#include <CUDA/include/streams_events.h>

#include <CUDA/include/constants.cuh>

#include <Util/include/debug.h>

#include <stdio.h>
#include <algorithm>

extern struct FrameResource frameResources[GPU_MAX];



hipError_t d_result_event_curr[GPU_MAX][FRAME_COUNT];
hipError_t d_result_event_prev[GPU_MAX][FRAME_COUNT];


uint8_t nOffsetsT;

extern "C" void cuda_set_test_offsets(uint32_t thr_id,
                                       uint32_t *OffsetsT, uint32_t T_count)
{
    nOffsetsT = T_count;

    debug::log(4, FUNCTION, thr_id, "    ", nOffsetsT);

    if(nOffsetsT > 16)
        debug::error(FUNCTION, "test offsets cannot exceed 16");

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_offsetsT), OffsetsT,
        nOffsetsT*sizeof(uint32_t), 0, hipMemcpyHostToDevice));

}


extern "C" void cuda_set_FirstSieveElement(uint32_t thr_id, uint32_t *limbs)
{
    debug::log(4, FUNCTION, thr_id);

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_zFirstSieveElement), limbs,
        WORD_MAX*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

extern "C" void cuda_set_quit(uint32_t quit)
{
       debug::log(4, FUNCTION, quit ? "true" : "false");

       CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_quit), &quit,
                                sizeof(uint32_t), 0, hipMemcpyHostToDevice));

}

__device__ void assign(uint32_t *l, uint32_t *r)
{
  #pragma unroll
  for(uint8_t i = 0; i < WORD_MAX; ++i)
    l[i] = r[i];
}

__device__ int inv2adic(uint32_t x)
{
  uint32_t a;
  a = x;
  x = (((x+2)&4)<<1)+x;
  x *= 2 - a*x;
  x *= 2 - a*x;
  x *= 2 - a*x;
  return -x;
}

__device__ uint32_t cmp_ge_n(uint32_t *x, uint32_t *y)
{
  for(int8_t i = WORD_MAX-1; i >= 0; --i)
  {
    if(x[i] > y[i])
      return 1;

    if(x[i] < y[i])
      return 0;
  }
  return 1;
}

__device__ uint8_t sub_n(uint32_t *z, uint32_t *x, uint32_t *y)
{
  uint32_t temp;
  uint8_t c = 0;

  #pragma unroll
  for(uint8_t i = 0; i < WORD_MAX; ++i)
  {
    temp = x[i] - y[i] - c;
    c = (temp > x[i]);
    z[i] = temp;
  }
  return c;
}

__device__ void sub_ui(uint32_t *z, uint32_t *x, const uint32_t &ui)
{
  uint32_t temp = x[0] - ui;
  uint8_t c = temp > x[0];
  z[0] = temp;

  #pragma unroll
  for(uint8_t i = 1; i < WORD_MAX; ++i)
  {
    temp = x[i] - c;
    c = (temp > x[i]);
    z[i] = temp;
  }
}

__device__ void add_ui(uint32_t *z, uint32_t *x, const uint64_t &ui)
{
  uint32_t temp = x[0] + static_cast<uint32_t>(ui & 0xFFFFFFFF);
  uint8_t c = temp < x[0];
  z[0] = temp;

  temp = x[1] + static_cast<uint32_t>(ui >> 32) + c;
  c = temp < x[1];
  z[1] = temp;

  #pragma unroll
  for(uint8_t i = 2; i < WORD_MAX; ++i)
  {
    temp = x[i] + c;
    c = (temp < x[i]);
    z[i] = temp;
  }
}

__device__ uint32_t addmul_1(uint32_t *z, uint32_t *x, const uint32_t y)
{
  uint64_t prod;
  uint32_t c = 0;

  #pragma unroll
  for(uint8_t i = 0; i < WORD_MAX; ++i)
  {
    prod = static_cast<uint64_t>(x[i]) * static_cast<uint64_t>(y);
    prod += c;
    prod += z[i];
    z[i] = prod;
    c = prod >> 32;
  }

  return c;
}

__device__ void mulredc(uint32_t *z, uint32_t *x, uint32_t *y, uint32_t *n, const uint32_t d, uint32_t *t)
{
  uint32_t m;//, c;
  //uint64_t temp;
  uint8_t i, j;

  #pragma unroll
  for(i = 0; i < WORD_MAX + 2; ++i)
    t[i] = 0;

  for(i = 0; i < WORD_MAX; ++i)
  {
    //c = addmul_1(t, x, y[i]);
    t[WORD_MAX] += addmul_1(t, x, y[i]);
    //temp = static_cast<uint64_t>(t[WORD_MAX]) + c;
    //t[WORD_MAX] = temp;
    //t[WORD_MAX] += c;
    //t[WORD_MAX + 1] = temp >> 32;

    m = t[0]*d;

    //c = addmul_1(t, n, m);
    t[WORD_MAX] += addmul_1(t, n, m);
    //temp = static_cast<uint64_t>(t[WORD_MAX]) + c;
    //t[WORD_MAX] = temp;
    //t[WORD_MAX] += c;
    //t[WORD_MAX + 1] = temp >> 32;

    #pragma unroll
    for(j = 0; j <= WORD_MAX; ++j)
      t[j] = t[j+1];
  }
  if(cmp_ge_n(t, n))
    sub_n(t, t, n);

  #pragma unroll
  for(i = 0; i < WORD_MAX; ++i)
    z[i] = t[i];
}

__device__ void redc(uint32_t *z, uint32_t *x, uint32_t *n, const uint32_t d, uint32_t *t)
{
  uint32_t m;
  uint8_t i, j;

  #pragma unroll
  for(i = 0; i < WORD_MAX; ++i)
    t[i] = x[i];

  t[WORD_MAX] = 0;

  for(i = 0; i < WORD_MAX; ++i)
  {
    m = t[0]*d;
    t[WORD_MAX] = addmul_1(t, n, m);

    for(j = 0; j < WORD_MAX; ++j)
      t[j] = t[j+1];

    t[WORD_MAX] = 0;
  }

  if(cmp_ge_n(t, n))
    sub_n(t, t, n);

  #pragma unroll
  for(i = 0; i < WORD_MAX; ++i)
    z[i] = t[i];
}

__device__ uint16_t bit_count(uint32_t *x)
{
   uint16_t msb = 0; //most significant bit

   uint16_t bits = WORD_MAX << 5;
   uint16_t i;

   #pragma unroll
   for(i = 0; i < bits; ++i)
   {
     if(x[i>>5] & (1 << (i & 31)))
       msb = i;
   }

   return msb + 1; //any number will have at least 1-bit
}

__device__ void lshift(uint32_t *r, uint32_t *a, uint16_t shift)
{
  int8_t i;

  #pragma unroll
  for(i = 0; i < WORD_MAX; ++i)
    r[i] = 0;

  uint8_t k = shift >> 5;
  shift = shift & 31;

  for(i = 0; i < WORD_MAX; ++i)
  {
    uint8_t ik = i + k;
    uint8_t ik1 = ik + 1;

    if(ik1 < WORD_MAX && shift != 0)
      r[ik1] |= (a[i] >> (32-shift));
    if(ik < WORD_MAX)
      r[ik] |= (a[i] << shift);
  }
}

__device__ void rshift(uint32_t *r, uint32_t *a, uint16_t shift)
{
  int8_t i;

  #pragma unroll
  for(i = 0; i < WORD_MAX; ++i)
    r[i] = 0;

  uint8_t k = shift >> 5;
  shift = shift & 31;

  for(i = 0; i < WORD_MAX; ++i)
  {
    int8_t ik = i - k;
    int8_t ik1 = ik - 1;

    if(ik1 >= 0 && shift != 0)
      r[ik1] |= (a[i] << (32-shift));
    if(ik >= 0)
      r[ik] |= (a[i] >> shift);
  }
}

__device__ void lshift1(uint32_t *r, uint32_t *a)
{
  uint32_t t = a[0];
  uint32_t t2;
  uint8_t i = 1;

  r[0] = t << 1;
  for(; i < WORD_MAX; ++i)
  {
    t2 = a[i];
    r[i] = (t2 << 1) | (t >> 31);
    t = t2;
  }
}

__device__ void rshift1(uint32_t *r, uint32_t *a)
{
  uint8_t n = WORD_MAX-1;
  int8_t i = n-1;

  uint32_t t = a[n];
  uint32_t t2;

  r[n] = t >> 1;
  for(; i >= 0; --i)
  {
    t2 = a[i];
    r[i] = (t2 >> 1) | (t << 31);
    t = t2;
  }
}

/* Calculate ABar and BBar for Montgomery Modular Multiplication. */
__device__ void calcBar(uint32_t *a, uint32_t *b, uint32_t *n, uint32_t *t)
{
    #pragma unroll
    for(uint8_t i = 0; i < WORD_MAX; ++i)
        a[i] = 0;

    lshift(t, n, (WORD_MAX<<5) - bit_count(n));
    sub_n(a, a, t);

    while(cmp_ge_n(a, n))  //calculate R mod N;
    {
        rshift1(t, t);
        if(cmp_ge_n(a, t))
          sub_n(a, a, t);
    }

    lshift1(b, a);     //calculate 2R mod N;
    if(cmp_ge_n(b, n))
      sub_n(b, b, n);
}


/* Calculate X = 2^Exp Mod N (Fermat test) */
__device__ void pow2m(uint32_t *X, uint32_t *Exp, uint32_t *N)
{
  uint32_t A[WORD_MAX];
  uint32_t d;
  uint32_t t[WORD_MAX + 1];

  d = inv2adic(N[0]);

  calcBar(X, A, N, t);

  for(int16_t i = bit_count(Exp)-1; i >= 0; --i)
  {
    mulredc(X, X, X, N, d, t);

    if(Exp[i>>5] & (1 << (i & 31)))
      mulredc(X, X, A, N, d, t);
  }

  redc(X, X, N, d, t);
}


/* Test if number p passes Fermat Primality Test base 2. */
__device__ bool fermat_prime(uint32_t *p)
{
  uint32_t e[WORD_MAX];
  uint32_t r[WORD_MAX];

  sub_ui(e, p, 1);
  pow2m(r, e, p);

  uint32_t result = r[0] - 1;

  #pragma unroll
  for(uint8_t i = 1; i < WORD_MAX; ++i)
    result |= r[i];

  return (result == 0);
}

/* Add a Result to the buffer. */
__device__ void add_result(uint64_t *nonce_offsets, uint64_t *nonce_meta, uint32_t *nonce_count,
                           uint64_t &offset, uint64_t &meta, uint32_t max)
{
    uint32_t i = atomicAdd(nonce_count, 1);

    if(i < max)
    {
        nonce_offsets[i] = offset;
        nonce_meta[i] = meta;
    }
    else
        printf("add result: max exceeded.\n");
}

/* Fermat Test and sort offsets into resulting or working buffers. */
__global__ void fermat_kernel(uint64_t *in_nonce_offsets,
                              uint64_t *in_nonce_meta,
                              uint32_t in_nonce_count,
                              uint64_t *out_nonce_offsets,
                              uint64_t *out_nonce_meta,
                              uint32_t *out_nonce_count,
                              uint64_t *g_result_offsets,
                              uint64_t *g_result_meta,
                              uint32_t *g_result_count,
                              uint32_t *g_primes_checked,
                              uint32_t *g_primes_found,
                              uint64_t nPrimorial,
                              uint8_t nTestOffsets,
                              uint8_t nTestLevels,
                              uint8_t o)
{

    /* Compute the global index for this nonce offset. */
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    /* If the quit flag was set, early return to avoid wasting time. */
    if(c_quit)
        return;

    /* Make sure index is not out of bounds. */
    if(idx < in_nonce_count)
    {
        /* Get the nonce offset and meta data. */
        uint64_t nonce_offset = in_nonce_offsets[idx];
        uint64_t nonce_meta =   in_nonce_meta[idx];
        uint32_t p[WORD_MAX];


        /* Decode the nonce meta data. */
        uint32_t combo           =  nonce_meta >> 32;
        uint8_t chain_offset_beg = (nonce_meta >> 24) & 0xFF;
        uint8_t chain_offset_end = (nonce_meta >> 16) & 0xFF;
        uint8_t prime_gap =        (nonce_meta >> 8 ) & 0xFF;
        uint8_t chain_length =      nonce_meta & 0xFF;

        /* Compute the primorial offset from the primorial and
         * offset pattern (i.e 510510*n + [0,4,6,10] ) */
        uint64_t primorial_offset = nPrimorial * nonce_offset;
        primorial_offset += c_offsetsT[chain_offset_end];

        /* Add to the first sieving element to compute prime to test. */
        add_ui(p, c_zFirstSieveElement, primorial_offset);

        if(p[0] % 5 != 0)
        {


            /* Check if prime passes fermat test base 2. */
            if(fermat_prime(p))
            {
                atomicAdd(g_primes_found, 1);
                ++chain_length;
                prime_gap = 0;

                /* If the chain length is satisfied, add it to result buffer. */
                if(chain_length == nTestLevels)
                {
                    /* Encode the nonce meta data. */
                    nonce_meta = 0;
                    nonce_meta |= ((uint64_t)combo << 32);
                    nonce_meta |= ((uint64_t)chain_offset_beg << 24);
                    nonce_meta |= ((uint64_t)chain_offset_end << 16);
                    nonce_meta |= ((uint64_t)prime_gap << 8);
                    nonce_meta |= (uint64_t)chain_length;

                    /* Add to result buffer. */
                    add_result(g_result_offsets, g_result_meta, g_result_count,
                               nonce_offset, nonce_meta, OFFSETS_MAX);

                    //printf("%d: add_result: %016llX\n", o, nonce_offset);
                }
            }
            atomicAdd(g_primes_checked, 1);
        }
        /* Otherwise, if chain length is not satisfied, keep testing. */
        if(chain_length < nTestLevels)
        {
            /* Make sure there are offsets to test in the combo bit mask. */
            if(combo)
            {
                /* Get the next offset index and clear it from the combo bits. */
                uint8_t chain_offset_next = __clz(combo);
                combo ^= 0x80000000 >> chain_offset_next;

                /* If the chain length is zero, shift to the next offset and start over. */
                if(chain_length == 0)
                {
                    chain_offset_beg = chain_offset_next;
                    chain_offset_end = chain_offset_next;
                    prime_gap = 0;
                }

                /* Make sure next offset and beginning are within bounds of testing
                 * for the next round. */
                if(chain_offset_next < nTestOffsets
                && chain_offset_beg <= nTestOffsets - nTestLevels)
                {
                    /* Calculate prime gap to next offset from last prime. */
                    prime_gap += c_offsetsT[chain_offset_next] - c_offsetsT[chain_offset_end];
                    chain_offset_end = chain_offset_next;

                    if(prime_gap <= 12)
                    {
                        /* Encode the nonce meta data. */
                        nonce_meta = 0;
                        nonce_meta |= ((uint64_t)combo << 32);
                        nonce_meta |= ((uint64_t)chain_offset_beg << 24);
                        nonce_meta |= ((uint64_t)chain_offset_end << 16);
                        nonce_meta |= ((uint64_t)prime_gap << 8);
                        nonce_meta |= (uint64_t)chain_length;

                        add_result(out_nonce_offsets, out_nonce_meta, out_nonce_count,
                                   nonce_offset, nonce_meta, OFFSETS_MAX);

                    }
                }
            }
        }
        //atomicAdd(g_primes_checked, 1);
    }
}

__global__ void fermat_launcher(uint64_t *g_nonce_offsets,
                                uint64_t *g_nonce_meta,
                                uint32_t *g_nonce_count,
                                uint64_t *g_result_offsets,
                                uint64_t *g_result_meta,
                                uint32_t *g_result_count,
                                uint32_t *g_primes_checked,
                                uint32_t *g_primes_found,
                                uint64_t nPrimorial,
                                uint8_t nTestOffsets,
                                uint8_t nTestLevels,
                                uint8_t o)
{
    uint8_t buffer_index = o & 1;

    uint64_t *in_nonce_offsets = g_nonce_offsets + buffer_index * OFFSETS_MAX;
    uint64_t *in_nonce_meta    = g_nonce_meta    + buffer_index * OFFSETS_MAX;
    uint32_t *in_nonce_count   = g_nonce_count   + buffer_index * 4;

    buffer_index ^= 1; //flip between two working buffers

    uint64_t *out_nonce_offsets = g_nonce_offsets + buffer_index * OFFSETS_MAX;
    uint64_t *out_nonce_meta    = g_nonce_meta    + buffer_index * OFFSETS_MAX;
    uint32_t *out_nonce_count   = g_nonce_count   + buffer_index * 4;

    uint32_t total_count = in_nonce_count[0];

    if(total_count >= OFFSETS_MAX && threadIdx.x == 0)
    {
        total_count = OFFSETS_MAX;
        printf("[WARNING] Candidates Max Reached. Use more Sieving Primes or Less Offsets.\n");
    }

    /* Clear the counts for each thread. */
    in_nonce_count[threadIdx.x] = 0;
    out_nonce_count[threadIdx.x] = 0;

    if(threadIdx.x == 0 && o == 0)
    {
        *g_result_count = 0;
        *g_primes_checked = 0;
        *g_primes_found = 0;

        //printf("total_count: %d\n", total_count);
    }

    if(total_count > 0 && c_quit == false)
    {
        /* Split the workload into segments and seperate thread launches. */
        uint32_t segment_in_count = (total_count + 5) >> 2;
        uint32_t segment_offset = threadIdx.x * segment_in_count;
        int32_t diff = total_count - segment_offset;
        if(diff < 0)
            diff = 0;

        /* Compute the segment count for each thread launch. */
        segment_in_count = min(segment_in_count, diff);

        /* Launch child processes to test results */
        if(segment_in_count > 0)
        {
            /* Create a temporary stream for the kernel launch. */
            hipStream_t stream;
            hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

            //printf("%d: ranges %d: [%03d %03d]\n", o, threadIdx.x, segment_offset, segment_offset + segment_in_count - 1);

            dim3 block(32);
            dim3 grid((segment_in_count+block.x-1)/block.x);

            /* Launch the fermat kernel for this segment. */
            fermat_kernel<<<grid, block, 0, stream>>>(
                &in_nonce_offsets[segment_offset],  &in_nonce_meta[segment_offset],  segment_in_count,
                &out_nonce_offsets[0], &out_nonce_meta[0],  &out_nonce_count[0],
                g_result_offsets, g_result_meta, g_result_count,
                g_primes_checked, g_primes_found, nPrimorial, nTestOffsets, nTestLevels, threadIdx.x);

            /* Synchronize device across kernel launches. */
            hipDeviceSynchronize();

            /* Destroy the temporary stream. */
            hipStreamDestroy(stream);
        }
    }

    //if(threadIdx.x == 0)
    //{
    //    total_count = 0;
    //    total_count = out_nonce_count[0];

    //    out_nonce_count[0] = total_count;

        //printf("%d: result count: %03d \n", o, *g_result_count);

        //for(int j = 0; j < *g_result_count; ++j)
        //    printf("%016llX ", g_result_offsets[j]);
        //printf("\n");
    //}

}


extern "C" __host__ void cuda_fermat(uint32_t thr_id,
                                     uint32_t sieve_index,
                                     uint32_t test_index,
                                     uint64_t nPrimorial,
                                     uint32_t nTestLevels)
{
    uint32_t curr_sieve = sieve_index % FRAME_COUNT;
    uint32_t curr_test = test_index % FRAME_COUNT;

    uint8_t nComboThreshold = 8;

    uint8_t str_id = 4;

    debug::log(4, FUNCTION, thr_id);

    /* Set the result event switch. */
    d_result_event_curr[thr_id][curr_test] = hipErrorNotReady;
    d_result_event_prev[thr_id][curr_test] = hipErrorNotReady;

    /*Make sure compaction event is finished before testing. */
    CHECK(stream_wait_event(thr_id, curr_sieve, str_id, EVENT::COMPACT));

    //printf("fermat_launcher<<<%d, %d>>>\n", 1, 1);
    for(uint8_t o = 0; o < nComboThreshold; ++o)
    {
        fermat_launcher<<<1, 4, 0, d_Streams[thr_id][str_id]>>>(
                 frameResources[thr_id].d_nonce_offsets[curr_test],
                 frameResources[thr_id].d_nonce_meta[curr_test],
                 frameResources[thr_id].d_nonce_count[curr_test],
                 frameResources[thr_id].d_result_offsets[curr_test],
                 frameResources[thr_id].d_result_meta[curr_test],
                 frameResources[thr_id].d_result_count[curr_test],
                 frameResources[thr_id].d_primes_checked[curr_test],
                 frameResources[thr_id].d_primes_found[curr_test],
                 nPrimorial, nOffsetsT, nTestLevels, o);
    }

    /* Copy the result count. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_result_count[curr_test],
                          frameResources[thr_id].d_result_count[curr_test],
                          sizeof(uint32_t), hipMemcpyDeviceToHost, d_Streams[thr_id][str_id]));

    /* Copy the result offsets. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_result_offsets[curr_test],
                          frameResources[thr_id].d_result_offsets[curr_test],
                          OFFSETS_MAX * sizeof(uint64_t), hipMemcpyDeviceToHost, d_Streams[thr_id][str_id]));

    /* copy the result meta. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_result_meta[curr_test],
                          frameResources[thr_id].d_result_meta[curr_test],
                          OFFSETS_MAX * sizeof(uint64_t), hipMemcpyDeviceToHost, d_Streams[thr_id][str_id]));

    /* Copy the amount of primes checked. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_primes_checked[curr_test],
                          frameResources[thr_id].d_primes_checked[curr_test],
                          sizeof(uint32_t), hipMemcpyDeviceToHost, d_Streams[thr_id][str_id]));

    /* Copy the amount of primes found. */
    CHECK(hipMemcpyAsync(frameResources[thr_id].h_primes_found[curr_test],
                          frameResources[thr_id].d_primes_found[curr_test],
                          sizeof(uint32_t), hipMemcpyDeviceToHost, d_Streams[thr_id][str_id]));

    /* Signal the Fermat event. */
    CHECK(stream_signal_event(thr_id, curr_test, str_id, EVENT::FERMAT));



}

extern "C" void cuda_results(uint32_t thr_id,
                             uint32_t test_index,
                             uint64_t *result_offsets,
                             uint64_t *result_meta,
                             uint32_t *result_count,
                             uint32_t *primes_checked,
                             uint32_t *primes_found)
{
    *result_count = 0;
    *primes_checked = 0;
    *primes_found = 0;

    uint32_t curr_test = test_index % FRAME_COUNT;

    d_result_event_prev[thr_id][curr_test] = d_result_event_curr[thr_id][curr_test];
    d_result_event_curr[thr_id][curr_test] = hipEventQuery(d_Events[thr_id][curr_test][EVENT::FERMAT]);

    if(d_result_event_curr[thr_id][curr_test] == hipSuccess &&
       d_result_event_prev[thr_id][curr_test] == hipErrorNotReady)
    {
        d_result_event_prev[thr_id][curr_test] = hipSuccess;

        *result_count   = *frameResources[thr_id].h_result_count[curr_test];
        *primes_checked = *frameResources[thr_id].h_primes_checked[curr_test];
        *primes_found   = *frameResources[thr_id].h_primes_found[curr_test];

        *frameResources[thr_id].h_result_count[curr_test] = 0;
        *frameResources[thr_id].h_primes_checked[curr_test] = 0;
        *frameResources[thr_id].h_primes_found[curr_test] = 0;

        if(*result_count == 0)
            return;

        uint64_t *pOffsets = frameResources[thr_id].h_result_offsets[curr_test];
        uint64_t *pMeta  =   frameResources[thr_id].h_result_meta[curr_test];

        std::copy(pOffsets, pOffsets + (*result_count), result_offsets);
        std::copy(pMeta,    pMeta    + (*result_count), result_meta);

        debug::log(4, FUNCTION, thr_id, "    ", *result_count, " results");
    }
}

extern "C" void cuda_init_counts(uint32_t thr_id)
{
    uint32_t zero[BUFFER_COUNT] = {0};

    debug::log(4, FUNCTION, thr_id);

    CHECK(hipDeviceSynchronize());

    for(int i = 0; i < FRAME_COUNT; ++i)
    {
        *frameResources[thr_id].h_nonce_count[i] = 0;

        CHECK(hipMemcpy(frameResources[thr_id].d_nonce_count[i],
                         zero,
                         sizeof(uint32_t) * BUFFER_COUNT,
                         hipMemcpyHostToDevice));
    }


}
