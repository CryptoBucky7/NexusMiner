#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include <CUDA/include/util.h>
#include <CUDA/include/frame_resources.h>

#include <Util/include/debug.h>

#include <stdio.h>

int device_map[GPU_MAX] = {0,1,2,3,4,5,6,7};

extern "C" void cuda_reset_device()
{
	hipDeviceReset();
}

extern "C" void cuda_device_synchronize()
{
  	hipDeviceSynchronize();
}

extern "C" void cuda_driver_version(int &major, int &minor)
{
	int version;
	hipError_t err = hipDriverGetVersion(&version);
	if (err != hipSuccess)
	{
		debug::error("Unable to query CUDA driver version! Is an nVidia driver installed?");
		return;
	}

	major = version / 1000;
	minor = version % 100; // same as in deviceQuery sample
	if (major < 5 || (major == 5 && minor < 5))
	{
		debug::error("Driver does not support CUDA 5.5 API! Update your nVidia driver!");
		return;
	}
}

extern "C" uint32_t cuda_device_multiprocessors(uint8_t index)
{
    hipDeviceProp_t props;

	if (hipGetDeviceProperties(&props, index) == hipSuccess)
		return props.multiProcessorCount;

    return 0;
}

extern "C" int cuda_num_devices()
{
    int GPU_N;
    hipError_t err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        debug::log(0, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        return -1;
    }
    return GPU_N;
}


extern "C" std::string cuda_devicename(uint8_t index)
{
	hipDeviceProp_t props;

	if (hipGetDeviceProperties(&props, index) == hipSuccess)
		return std::string(props.name);

	return std::string();
}

extern "C" void cuda_init(uint8_t thr_id)
{
  debug::log(0, "thread ", (uint32_t)thr_id, " maps to CUDA device #", static_cast<uint32_t>(device_map[thr_id]));

  hipSetDevice(device_map[thr_id]);

  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
}

extern "C" void cuda_free(uint8_t thr_id)
{
    debug::log(0, "Device ", static_cast<uint32_t>(device_map[thr_id]), " shutting down...");

    hipSetDevice(device_map[thr_id]);
    hipDeviceSynchronize();
    hipDeviceReset();
}

extern "C" void cuda_shutdown()
{
	hipProfilerStop();
}
