#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <CUDA/include/util.h>
#include <CUDA/include/frame_resources.h>

#include <Util/include/debug.h>

#include <stdio.h>

int device_map[GPU_MAX] = {0,1,2,3,4,5,6,7};

extern "C" void cuda_reset_device()
{
	hipDeviceReset();
}


extern "C" void cuda_device_synchronize()
{
  	hipDeviceSynchronize();
}


extern "C" void cuda_runtime_version(int &major, int &minor)
{
    int runtime_version;
	hipError_t err = hipRuntimeGetVersion(&runtime_version);
	if (err != hipSuccess)
	{
		debug::error("Unable to query CUDA runtime version! Is an Nvidia runtime installed?");
		return;
	}

	major = runtime_version / 1000;
	minor = (runtime_version % 100) / 10; // same as in deviceQuery sample
	if (major < 5 || (major == 5 && minor < 5))
	{
		debug::error("Runtime does not support CUDA 5.5 API! Update your Nvidia runtime!");
		return;
	}
}


extern "C" void cuda_driver_version(int &major, int &minor)
{
	int driver_version;
	hipError_t err = hipDriverGetVersion(&driver_version);
	if (err != hipSuccess)
	{
		debug::error("Unable to query CUDA driver version! Is an Nvidia driver installed?");
		return;
	}

	major = driver_version / 1000;
	minor = (driver_version % 100) / 10; // same as in deviceQuery sample
	if (major < 5 || (major == 5 && minor < 5))
	{
		debug::error("Driver does not support CUDA 5.5 API! Update your Nvidia driver!");
		return;
	}
}


extern "C" uint32_t cuda_device_multiprocessors(uint8_t index)
{
    hipDeviceProp_t props;

	if (hipGetDeviceProperties(&props, index) == hipSuccess)
		return props.multiProcessorCount;

    return 0;
}


extern "C" int cuda_num_devices()
{
    int GPU_N;
    hipError_t err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        debug::log(0, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        return -1;
    }
    return GPU_N;
}


extern "C" std::string cuda_devicename(uint8_t index)
{
	hipDeviceProp_t props;

	if (hipGetDeviceProperties(&props, index) == hipSuccess)
		return std::string(props.name);

	return std::string();
}


extern "C" void cuda_init(uint8_t thr_id)
{
  debug::log(0, "thread ", (uint32_t)thr_id, " maps to CUDA device #", static_cast<uint32_t>(device_map[thr_id]));

  hipSetDevice(device_map[thr_id]);

  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
}


extern "C" void cuda_free(uint8_t thr_id)
{
    debug::log(0, "Device ", static_cast<uint32_t>(device_map[thr_id]), " shutting down...");

    hipSetDevice(device_map[thr_id]);
    hipDeviceSynchronize();
    hipDeviceReset();
}

extern "C" void cuda_shutdown()
{
	hipProfilerStop();
}
