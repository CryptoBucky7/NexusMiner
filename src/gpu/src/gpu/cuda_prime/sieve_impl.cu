#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sieve_impl.cuh"
#include "sieve.hpp"
#include "sieve_small_prime_constants.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>


#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

namespace nexusminer {
    namespace gpu {

        __device__ void cuda_chain_push_back(CudaChain& chain, uint16_t offset);
        __device__ void cuda_chain_open(CudaChain& chain, uint64_t base_offset);
        __device__  bool is_there_still_hope(CudaChain& chain);
        __device__  void get_best_fermat_chain(const CudaChain& chain, uint64_t& base_offset, int& offset, int& best_length);

        __device__ const unsigned int sieve30_offsets[]{ 1,7,11,13,17,19,23,29 };

        //__device__ const unsigned int sieve30_inverse_offsets[]{ 1,13,11,7,23,19,17,29 }; 

        __device__ const unsigned int sieve30_gaps[]{ 6,4,2,4,2,4,6,2 };

        __device__ const unsigned int sieve30_index[]
        { 0,0,1,1,1,1,1, 1, 2, 2, 2, 2, 3, 3, 4, 4, 4, 4, 5, 5, 6, 6, 6, 6, 7, 7, 7, 7, 7, 7 };  //reverse lookup table (offset mod 30 to index)

        //__device__ const unsigned int sieve30_inverse_index[]
        //{ 0,0,3,3,3,3,3, 3, 2, 2, 2, 2, 1, 1, 6, 6, 6, 6, 5, 5, 4, 4, 4, 4, 7, 7, 7, 7, 7, 7 };  //reverse lookup table (prime inverse mod 30 to index)

        __device__ const unsigned int prime_mod30_inverse[]
        { 1,1,13,13,13,13,13, 13, 11, 11, 11, 11, 7, 7, 23, 23, 23, 23, 19, 19, 17, 17, 17, 17, 29, 29, 29, 29, 29, 29 };  //lookup table - prime % 30 to prime inverse % 30

        __device__ const unsigned int next_multiple_mod30_offset[]  //range mod30 to the next highest prime.
        { 1,0,5,4,3,2,1, 0, 3, 2, 1, 0, 1, 0, 3, 2, 1, 0, 1, 0, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0 };

        __device__ const unsigned int sieve120_index[]
        {    0, 0, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 4, 4, 4, 4, 5, 5, 6, 6, 6, 6, 7, 7, 7, 7, 7, 7, 
             8, 8, 9, 9, 9, 9, 9, 9,10,10,10,10,11,11,12,12,12,12,13,13,14,14,14,14,15,15,15,15,15,15,
            16,16,17,17,17,17,17,17,18,18,18,18,19,19,20,20,20,20,21,21,22,22,22,22,23,23,23,23,23,23,
            24,24,25,25,25,25,25,25,26,26,26,26,27,27,28,28,28,28,29,29,30,30,30,30,31,31,31,31,31,31
        };  //reverse lookup table (offset mod 120 to index)


        __device__  const Cuda_sieve::sieve_word_t unset_bit_mask[]{
            ~(1u << 0),  ~(1u << 1),  ~(1u << 2),  ~(1u << 3),  ~(1u << 4),  ~(1u << 5),  ~(1u << 6),  ~(1u << 7), 
            ~(1u << 8),  ~(1u << 9),  ~(1u << 10), ~(1u << 11), ~(1u << 12), ~(1u << 13), ~(1u << 14), ~(1u << 15),
            ~(1u << 16), ~(1u << 17), ~(1u << 18), ~(1u << 19), ~(1u << 20), ~(1u << 21), ~(1u << 22), ~(1u << 23),
            ~(1u << 24), ~(1u << 25), ~(1u << 26), ~(1u << 27), ~(1u << 28), ~(1u << 29), ~(1u << 30), ~(1u << 31)
        };
        
        // cross off small primes.  These primes hit the sieve often.  We iterate through the sieve words and cross them off using 
        // precalculated constants.  start is offset from the sieve start 
        __global__ void sieveSmallPrimes(Cuda_sieve::sieve_word_t* sieve, uint64_t start, uint32_t* small_prime_offsets)
        {

            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;

            const uint32_t increment = Cuda_sieve::m_sieve_word_range;

            //#pragma unroll
            for (uint64_t i = index; i < Cuda_sieve::m_sieve_total_size; i += stride) 
            {
                
                //the offset for the sieve word in process
                uint64_t inc = i * increment;
                //get the correct rotation for the prime mask
                //primes for reference 7,11,13,17,19,23,29,31,37,41,43,47,53,59,61,67,71,73,79,83,89,97,101
                //                     1, 2, 3, 4, 5, 6, 7, 8, 9,10,11,12,13,14,15,16,17,18,19,20,21,22, 23  

                uint16_t index7 = (start + small_prime_offsets[0] + inc) % 7;
                uint16_t index11 = (start + small_prime_offsets[1] + inc) % 11;
                uint16_t index13 = (start + small_prime_offsets[2] + inc) % 13;
                uint16_t index17 = (start + small_prime_offsets[3] + inc) % 17;
                uint16_t index19 = (start + small_prime_offsets[4] + inc) % 19;
                uint16_t index23 = (start + small_prime_offsets[5] + inc) % 23;
                uint16_t index29 = (start + small_prime_offsets[6] + inc) % 29;
                uint16_t index31 = (start + small_prime_offsets[7] + inc) % 31;
                uint16_t index37 = (start + small_prime_offsets[8] + inc) % 37;
                uint16_t index41 = (start + small_prime_offsets[9] + inc) % 41;
                uint16_t index43 = (start + small_prime_offsets[10] + inc) % 43;
                uint16_t index47 = (start + small_prime_offsets[11] + inc) % 47;
                uint16_t index53 = (start + small_prime_offsets[12] + inc) % 53;
                uint16_t index59 = (start + small_prime_offsets[13] + inc) % 59;
                uint16_t index61 = (start + small_prime_offsets[14] + inc) % 61;
                uint16_t index67 = (start + small_prime_offsets[15] + inc) % 67;
                uint16_t index71 = (start + small_prime_offsets[16] + inc) % 71;
                uint16_t index73 = (start + small_prime_offsets[17] + inc) % 73;
                uint16_t index79 = (start + small_prime_offsets[18] + inc) % 79;
                uint16_t index83 = (start + small_prime_offsets[19] + inc) % 83;
                uint16_t index89 = (start + small_prime_offsets[20] + inc) % 89;
                uint16_t index97 = (start + small_prime_offsets[21] + inc) % 97;
                uint16_t index101 = (start + small_prime_offsets[22] + inc) % 101;

               

                //apply the mask.  the mask for the first prime 7 is also used to initialize the sieve (hence no &).
                Cuda_sieve::sieve_word_t word;
                word = p7[index7];
                word &= p11[index11];
                word &= p13[index13];
                word &= p17[index17];
                word &= p19[index19];
                word &= p23[index23];
                word &= p29[index29];
                word &= p31[index31];
                word &= p37[index37];
                word &= p41[index41];
                word &= p43[index43];
                word &= p47[index47];
                word &= p53[index53];
                word &= p59[index59];
                //word &= p61[index61];
                word &= p_61(index61);
                word &= p67[index67];
                word &= p71[index71];
                word &= p73[index73];
                word &= p79[index79];
                word &= p83[index83];
                word &= p89[index89];
                word &= p97[index97];
                word &= p101[index101];

                //save to global memory
                sieve[i] = word;

            }
        }

        //return the offset from x to the next integer multiple of n greater than x that is not divisible by 2, 3, or 5.  
       //x must be a multiple of the primorial 30 and n must be a prime greater than 5.
        template <typename T1, typename T2>
        __device__ __forceinline__ T2 get_offset_to_next_multiple(T1 x, T2 n)
        {
            T2 m = n - static_cast<T2>(x % n);
            //T2 m1 = m + n * next_multiple_mod30_offset[((m % 30) *prime_mod30_inverse[n % 30]) % 30];
            

           if (m % 2 == 0)
            {
                m += n;
            }
           if (m % 3 == 0 || m % 5 == 0)
           {
               m += 2 * n;
           }
           if (m % 3 == 0 || m % 5 == 0)
           {
               m += 2 * n;
           }
            /*while (m % 3 == 0 || m % 5 == 0)
            {
                m += 2 * n;
            }*/
            /*if (m != m1)
            {
                printf("%" PRIu64 " %u %u %u\n", x, m, m1, n);

            }*/
            return m;
        }

        //this is slow. 
        //large primes hit the sieve infrequently with large gaps (> 1 segment) between hits.  The optimizations for
        //medium primes hurts more than helps.  For large primes we simply iterate through multiples of the primes and cross 
        //them off one by one in global memory using atomicAnd.  The memory conflicts between primes should be few because
        // of the infrequency of the hits to the sieve. 
        __global__ void sieveLargePrimes(uint64_t sieve_start_offset, uint32_t* sieving_primes, uint32_t sieving_prime_count,
            uint32_t* starting_multiples, uint32_t* prime_mod_inverses, Cuda_sieve::sieve_word_t* sieve)
        {

            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            uint64_t wheel_index;
            unsigned int next_wheel_gap;
            uint64_t j;
            uint64_t k;
            
            //iterate through each prime starting at the large prime cutoff prime
            for (uint32_t i = index + Cuda_sieve::m_large_prime_cutoff_index; i < sieving_prime_count; i += stride)
            {
                //calculate the starting offset for the current prime
                j = starting_multiples[i];
                if (sieve_start_offset >= j)
                    j = get_offset_to_next_multiple(sieve_start_offset - j, sieving_primes[i]);
                else
                    j -= sieve_start_offset;
                k = sieving_primes[i];
                wheel_index = sieve30_index[(prime_mod_inverses[i] * j) % 30];
                next_wheel_gap = sieve30_gaps[wheel_index];

                while (j < Cuda_sieve::m_sieve_range)
                {
                    //cross off a multiple of the sieving prime
                    uint64_t sieve_index = j / Cuda_sieve::m_sieve_word_range;
                    Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                        (sieve30_index[j % 30] + (8 * (j / Cuda_sieve::m_sieve_byte_range % Cuda_sieve::m_sieve_word_byte_count))));

                    //todo: test lookup table version
                    //Cuda_sieve::sieve_word_t bitmask2 = unset_bit_mask[sieve120_index[j % 120u]];

                    //printf("%" PRIu64 " %u\n", j, bitmask);

                    
                    atomicAnd(&sieve[sieve_index], bitmask);

                    //increment the next multiple of the current prime (rotate the wheel).
                    j += k * next_wheel_gap;
                    wheel_index++;
                    next_wheel_gap = sieve30_gaps[wheel_index % 8];
                }

            }

        }
        
        //get the nth bit from the sieve.
        __device__ __forceinline__ bool get_bit(uint64_t bit_position, Cuda_sieve::sieve_word_t* sieve)
        {
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            
            uint64_t word = bit_position / sieve_bits_per_word;
            unsigned bit_position_in_word = bit_position % sieve_bits_per_word;
            return ((sieve[word] >> bit_position_in_word) & 1) == 1;

        }

        //search the sieve for chains that meet the minimum length requirement.  
        __global__ void find_chain_kernel(Cuda_sieve::sieve_word_t* sieve, CudaChain* chains, uint32_t* chain_index, uint64_t sieve_start_offset,
            unsigned long long* chain_stat_count)
        {

            //const uint64_t sieve_size = Cuda_sieve::m_sieve_total_size;
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            const uint64_t sieve_total_bits = Cuda_sieve::m_sieve_total_size * sieve_bits_per_word;

            
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            unsigned int sieve_offset;
            unsigned int gap;
            uint64_t chain_start, prime_candidate_offset;

            //shared copies of lookup tables
            __shared__ unsigned int sieve30_offsets_shared[8];
            __shared__ unsigned int sieve30_gaps_shared[8];
            //local stats
            __shared__ uint32_t chain_count_shared;
            
            if (threadIdx.x < 8)
            {
                int i = threadIdx.x;
                sieve30_offsets_shared[i] = sieve30_offsets[i];
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            
            if (threadIdx.x == 0)
                chain_count_shared = 0;
            __syncthreads();
           
            //search each sieve location for a possible chain
            for (uint64_t i = index; i < sieve_total_bits; i += stride)
            {
              
                //gross checks to ensure its possible to form a chain
                uint64_t word = i / sieve_bits_per_word;
                if (sieve[word] == 0)
                    continue;
                //check if the next 4 bytes (4*30 = range of 120 integers) has enough prime candidates to form a chain 
                //this is only valid up to min chain length 9.  above 9 requires 5 bytes.
                if (word < Cuda_sieve::m_sieve_total_size - 1)
                {
                    unsigned int next_4_bytes = 0;
                    unsigned int byte_index = (i/8) % 4;
                    next_4_bytes = (sieve[word] >> (byte_index * 8)) & 0xFF;
                    next_4_bytes |= (((sieve[word + (byte_index >= 3 ? 1 : 0)] >> ((byte_index + 1) % 4) * 8) & 0xFF) << 8);
                    next_4_bytes |= (((sieve[word + (byte_index >= 2 ? 1 : 0)] >> ((byte_index + 2) % 4) * 8) & 0xFF) << 16);
                    next_4_bytes |= (((sieve[word + (byte_index >= 1 ? 1 : 0)] >> ((byte_index + 3) % 4) * 8) & 0xFF) << 24);

                    int popc = __popc(next_4_bytes);
                    if (popc < Cuda_sieve::m_min_chain_length)
                        continue;
                }

                //chain must start with a prime
                if (!get_bit(i, sieve))
                {
                    continue;
                }
                //search left for another prime less than max gap away
                uint64_t j = i - 1;
                gap = sieve30_gaps_shared[j % 8];
                while (j < i && gap <= maxGap)
                {
                    if (get_bit(j, sieve))
                    {
                        //there is a valid element to the left.  this is not the first element in a chain. abort.
                        break;
                    }
                    j--;
                    gap += sieve30_gaps_shared[j % 8];
                }
                if (gap <= maxGap)
                    continue;
                //this is the start of a possible chain.  search right
                //where are we in the wheel
                sieve_offset = sieve30_offsets_shared[i % 8u];
                chain_start = sieve_start_offset + i / 8 * 30 + sieve_offset;
                CudaChain current_chain;
                cuda_chain_open(current_chain, chain_start);
                j = i;
                gap = sieve30_gaps_shared[j % 8u];
                j++;
                while (j < sieve_total_bits && gap <= maxGap)
                {
                    if (get_bit(j, sieve))
                    {
                        //another possible candidate.  add it to the chain
                        gap = 0;
                        sieve_offset = sieve30_offsets_shared[j % 8u];
                        prime_candidate_offset = sieve_start_offset + j / 8 * 30 + sieve_offset;
                        uint16_t offset = prime_candidate_offset - chain_start;
                        //printf("%" PRIu64 " %u\n", chain_start, prime_candidate_offset);
                        cuda_chain_push_back(current_chain, offset);
                    }
                    gap += sieve30_gaps_shared[j % 8u];
                    j++;
                        
                }
                //we reached the end of the chain.  check if it meets the length requirement
                if (current_chain.m_offset_count >= Cuda_sieve::m_min_chain_length)
                {
                    //increment the chain list index
                    uint32_t chain_idx = atomicInc(chain_index, Cuda_sieve::m_max_chains);
                    //copy the current chain to the global list
                    chains[chain_idx] = current_chain;
                    //updated block level stats
                    atomicInc(&chain_count_shared, 0xFFFFFFFF);
                }
            }
            //update global chain stats
            __syncthreads();
            if (threadIdx.x == 0)
                atomicAdd(chain_stat_count, chain_count_shared);
        }

        //medium prime sieve.  We use a block of shared memory to sieve in segments.  Each block sieves a different range. 
        //the final results are merged with the global sieve at the end using atomicAnd. 
        __global__ void do_sieve(uint64_t sieve_start_offset, uint32_t* sieving_primes, uint32_t sieving_prime_count,
            uint32_t* starting_multiples, Cuda_sieve::sieve_word_t* sieve_results, uint32_t* multiples)
        {

            const uint32_t segment_size = Cuda_sieve::m_kernel_sieve_size_bytes * Cuda_sieve::m_sieve_byte_range;

            //local shared copy of the sieve
            __shared__ Cuda_sieve::sieve_word_t sieve[Cuda_sieve::m_kernel_sieve_size_words];
            //shared mem lookup tables
            __shared__ unsigned int sieve120_index_shared[120];
            __shared__  Cuda_sieve::sieve_word_t unset_bit_mask_shared[32];
            __shared__ unsigned int sieve30_gaps_shared[8];
            __shared__ unsigned int sieve30_index_shared[30];
            __shared__ unsigned int prime_mod30_inverse_shared[30];
            __shared__ unsigned int next_multiple_mod30_offset_shared[30];

            uint32_t block_id = blockIdx.x;
            uint32_t index = threadIdx.x;
            uint32_t stride = blockDim.x;

            //initialize shared lookup tables.  lookup tables in shared memory are faster than global memory lookup tables.
            for (int i = index; i < 120; i += stride)
            {
                sieve120_index_shared[i] = sieve120_index[i];
            }
            for (int i = index; i < 32; i += stride)
            {
                unset_bit_mask_shared[i] = unset_bit_mask[i];
            }
            for (int i = index; i < 8; i += stride)
            {
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            for (int i = index; i < 30; i += stride)
            {
                sieve30_index_shared[i] = sieve30_index[i];
                prime_mod30_inverse_shared[i] = prime_mod30_inverse[i];
                next_multiple_mod30_offset_shared[i] = next_multiple_mod30_offset[i];
            }

           
            const uint32_t segments = Cuda_sieve::m_kernel_segments_per_block;
            uint32_t sieve_results_index = block_id * Cuda_sieve::m_kernel_sieve_size_words_per_block;

            //each block sieves a different region
            uint64_t start_offset = sieve_start_offset + static_cast<uint64_t>(block_id) * Cuda_sieve::m_kernel_sieve_size_words_per_block * Cuda_sieve::m_sieve_word_range;
            
            uint8_t wheel_index;
            unsigned int next_wheel_gap;
            uint32_t j;
            uint32_t k;
            uint32_t prime_mod_inv;
            uint32_t max_prime_index = min(sieving_prime_count, Cuda_sieve::m_large_prime_cutoff_index);
            for (int s = 0; s < segments; s++)
            {
                
                //everyone in the block initialize part of the shared sieve
                for (int j1 = index; j1 < Cuda_sieve::m_kernel_sieve_size_words; j1 += stride)
                {
                    sieve[j1] = ~0;
                }
                
                __syncthreads();
                for (uint32_t i = index; i < max_prime_index; i += stride)
                {
                    k = sieving_primes[i];
                    //get aligned to this region
                    if (s == 0)
                    {
                        j = starting_multiples[i];
                        //printf("%" PRIu64 " %" PRIu64 " %u\n", start_offset, j, k);
                        //the first time through we need to calculate the starting offsets
                        if (start_offset >= j)
                        {
                            //j = get_offset_to_next_multiple(start_offset - j, sieving_primes[i]);
                            uint64_t x = start_offset - j;
                            //offset to the first integer multiple of the prime above the starting offset
                            uint32_t m = k - (x % k);
                            //find the next integer multiple of the prime that is not divisible by 2,3 or 5
                            m += (m % 2 == 0) ? k : 0;
                            m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                            m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                            j = m;
                            //this does the same thing as above - it gets the next multiple using prime inverse mod 30 and a lookup table. 
                            //prime_mod_inv = prime_mod30_inverse_shared[k % 30];
                            //j = m + k * next_multiple_mod30_offset_shared[((m % 30) * prime_mod_inv) % 30];
                        }

                        else
                            j -= start_offset;
                        
                    }
                    else
                    {
                        j = multiples[block_id* sieving_prime_count +i];
                        //calculating the wheel index each time is faster than saving and retrieving it from global memory each loop
                    }
                    prime_mod_inv = prime_mod30_inverse_shared[k % 30];
                    wheel_index = sieve30_index_shared[(prime_mod_inv * j) % 30];
                    next_wheel_gap = sieve30_gaps_shared[wheel_index];
                        
                    while (j < segment_size)
                    {
                        //cross off a multiple of the sieving prime
                        uint32_t sieve_index = j / Cuda_sieve::m_sieve_word_range;
                        //Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                        //    (sieve30_index[j % 30] + (8 * (j/Cuda_sieve::m_sieve_byte_range % Cuda_sieve::m_sieve_word_byte_count))));
                        
                        Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                            sieve120_index_shared[j % Cuda_sieve::m_sieve_word_range]);
                        //using this lookup table from shared memory is about the same as the hybrid version above
                        //Cuda_sieve::sieve_word_t bitmask = unset_bit_mask_shared[sieve120_index_shared[j % Cuda_sieve::m_sieve_word_range]];
                        
                        //printf("%" PRIu64 " %u\n", j, bitmask);
                            
                        atomicAnd(&sieve[sieve_index], bitmask);
                        
                        //increment the next multiple of the current prime (rotate the wheel).
                        j += k * next_wheel_gap;
                        wheel_index = (wheel_index + 1) % 8;
                        next_wheel_gap = sieve30_gaps_shared[wheel_index];
                    }
                    //save the starting multiple for this prime for the next segment
                    multiples[block_id * sieving_prime_count + i] = j - segment_size;
                    
                }
                __syncthreads();
                

                //merge the sieve results back to global memory
                
                for (uint32_t j2 = index; j2 < Cuda_sieve::m_kernel_sieve_size_words; j2 += stride)
                {
                    if (j2 < Cuda_sieve::m_kernel_sieve_size_words)
                    {
                        sieve_results[sieve_results_index + j2] &= sieve[j2];

                    }
                }
                
                sieve_results_index += Cuda_sieve::m_kernel_sieve_size_words;
                start_offset += segment_size;
            }

        }

        //count the prime candidates in the global sieve
        __global__ void count_prime_candidates(Cuda_sieve::sieve_word_t* sieve, unsigned long long* prime_candidate_count)
        {
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            
            uint64_t count = 0;
            if (index == 0)
                *prime_candidate_count = 0;
            __syncthreads();

            for (uint64_t i = index; i < Cuda_sieve::m_sieve_total_size; i += stride)
            {
                count += __popcll(sieve[i]);
            }
            atomicAdd(prime_candidate_count, count);

        }

        //go through the list of chains.  copy winners to the long chain list.  copy survivors to a temporary chain
        __global__ void filter_busted_chains(CudaChain* chains, uint32_t* chain_index, CudaChain* surviving_chains,
            uint32_t* surviving_chain_index, CudaChain* long_chains, uint32_t* long_chain_index, uint32_t* histogram)
        {
            uint32_t num_threads = blockDim.x;
            uint32_t block_id = blockIdx.x;
            uint32_t index = block_id * num_threads + threadIdx.x;

            if (index >= *chain_index)
                return;
            if (index == 0)
            {
                *surviving_chain_index = 0;
            }
            __syncthreads();
            //printf("%" PRIu64 " %u\n", index, *chain_index);
            if (!is_there_still_hope(chains[index]))
            {
                //this chain is busted.  check how long it is
                //collect stats
                //only count chains 3 or longer to minimize memory accesses
                if (chains[index].m_prime_count >= 3)
                {
                    int chain_length, local_offset;
                    uint64_t base_offset;
                    get_best_fermat_chain(chains[index], base_offset, local_offset, chain_length);
                    uint32_t histogram_chain_length = min(chain_length, Cuda_sieve::chain_histogram_max);
                    if (chain_length >= 3)
                        atomicInc(&histogram[histogram_chain_length], 0xFFFFFFFF);

                    //check for winners
                    if (chain_length >= chains[index].m_min_chain_report_length)
                    {
                        //chain is long. save it. 
                        uint32_t last_long_chain_index = atomicInc(long_chain_index, Cuda_sieve::m_max_long_chains);
                        long_chains[last_long_chain_index] = chains[index];
                    }
                }
            }
            else
            {
                //copy chain to the survival list
                uint32_t last_surviving_chain_index = atomicInc(surviving_chain_index, Cuda_sieve::m_max_chains);
                surviving_chains[last_surviving_chain_index] = chains[index];
            }
        }

        void Cuda_sieve_impl::run_large_prime_sieve(uint64_t sieve_start_offset)
        {
            const int threads = 256;
            const int primes_per_block = 1;
            uint32_t large_prime_count = m_sieving_prime_count - Cuda_sieve::m_large_prime_cutoff_index;
            int blocks = (large_prime_count/ primes_per_block + threads - 1) / threads;
            if (Cuda_sieve::m_large_prime_cutoff_index < m_sieving_prime_count)
            {
                sieveLargePrimes << <blocks, threads >> > (sieve_start_offset, d_sieving_primes, m_sieving_prime_count,
                    d_starting_multiples, d_prime_mod_inverses, d_sieve);
                checkCudaErrors(hipDeviceSynchronize());
            }

        }

        void Cuda_sieve_impl::run_small_prime_sieve(uint64_t sieve_start_offset)
        {
            const int threads = 256;
            const int blocks = (Cuda_sieve::m_sieve_total_size + threads - 1)/threads;
            
            sieveSmallPrimes << <blocks, threads >> > (d_sieve, sieve_start_offset, d_small_prime_offsets);

            checkCudaErrors(hipDeviceSynchronize());
        }

        void Cuda_sieve_impl::run_sieve(uint64_t sieve_start_offset)
        {
            m_sieve_start_offset = sieve_start_offset;
            
            do_sieve <<<Cuda_sieve::m_num_blocks, Cuda_sieve::m_threads_per_block >>> (sieve_start_offset, d_sieving_primes, m_sieving_prime_count,
                d_starting_multiples, d_sieve, d_multiples);

            checkCudaErrors(hipDeviceSynchronize());
        }

        void Cuda_sieve_impl::get_sieve(Cuda_sieve::sieve_word_t sieve[])
        {
            checkCudaErrors(hipMemcpy(sieve, d_sieve, Cuda_sieve::m_sieve_total_size * sizeof(*d_sieve), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::get_prime_candidate_count(uint64_t& prime_candidate_count)
        {
            const int threads = 256;
            const int blocks = 1; // (Cuda_sieve::m_sieve_total_size + threads - 1) / threads;
            count_prime_candidates << <blocks, threads >> > (d_sieve, d_prime_candidate_count);
            checkCudaErrors(hipDeviceSynchronize());
            
            checkCudaErrors(hipMemcpy(&prime_candidate_count, d_prime_candidate_count, sizeof(*d_prime_candidate_count), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::find_chains()
        {
            const int sieve_threads = 128;
            const int checks_per_block = 32;
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            const uint64_t sieve_total_bits = Cuda_sieve::m_sieve_total_size * sieve_bits_per_word;
            const int sieve_blocks = (sieve_total_bits /checks_per_block + sieve_threads - 1)/ sieve_threads;
            find_chain_kernel << <sieve_blocks, sieve_threads >> > (d_sieve, d_chains, d_last_chain_index, m_sieve_start_offset, d_chain_stat_count);

            checkCudaErrors(hipDeviceSynchronize());
            
        }

        void Cuda_sieve_impl::get_chains(CudaChain chains[], uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(chains, d_chains, chain_count * sizeof(*d_chains), hipMemcpyDeviceToHost));
        }

        void Cuda_sieve_impl::get_chain_count(uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToHost));
        }

        //get a pointer to the chain array.  fermat test uses the chain array as input. 
        void Cuda_sieve_impl::get_chain_pointer(CudaChain*& chains_ptr, uint32_t*& chain_count_ptr)
        {
            chains_ptr = d_chains;
            chain_count_ptr = d_last_chain_index;
        }

        //check the list of chains for winners.  save winners and remove losers
        void Cuda_sieve_impl::clean_chains()
        {
            const int threads = 256;
            uint32_t chain_count;
            get_chain_count(chain_count);
            int blocks = (chain_count + threads - 1) / threads;
            //copy surviving chains to a temporary location. 
            filter_busted_chains << <blocks, threads >> > (d_chains, d_last_chain_index, d_good_chains, d_good_chain_index,
                d_long_chains, d_last_long_chain_index, d_chain_histogram);
            checkCudaErrors(hipDeviceSynchronize());
            uint32_t good_chain_count;
            //get the count of good chains from device memory
            checkCudaErrors(hipMemcpy(&good_chain_count, d_good_chain_index, sizeof(*d_good_chain_index), hipMemcpyDeviceToHost));
            //copy the temporary good chain list back to the chain list
            checkCudaErrors(hipMemcpy(d_chains, d_good_chains, good_chain_count*sizeof(*d_chains), hipMemcpyDeviceToDevice));
            //update the chain count
            checkCudaErrors(hipMemcpy(d_last_chain_index, d_good_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToDevice));

        }

        void Cuda_sieve_impl::get_long_chains(CudaChain chains[], uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_long_chain_index, sizeof(*d_last_long_chain_index), hipMemcpyDeviceToHost));
            if (chain_count > 0)
                checkCudaErrors(hipMemcpy(chains, d_long_chains, chain_count * sizeof(*d_long_chains), hipMemcpyDeviceToHost));
            //clear the long chain list
            checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
        }

        //read the histogram
        void Cuda_sieve_impl::get_stats(uint32_t chain_histogram[], uint64_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(chain_histogram, d_chain_histogram, (Cuda_sieve::chain_histogram_max+1) * sizeof(*d_chain_histogram), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_stat_count, sizeof(*d_chain_stat_count), hipMemcpyDeviceToHost));

        }

        //allocate global memory and load values used by the sieve to the gpu 
        void Cuda_sieve_impl::load_sieve(uint32_t primes[], uint32_t prime_count, 
            uint32_t prime_mod_inverses_host[], uint32_t sieve_size, uint16_t device)
        {
          
            m_sieving_prime_count = prime_count;
            m_device = device;
            checkCudaErrors(hipSetDevice(device));
            //allocate memory on the gpu
            checkCudaErrors(hipMalloc(&d_sieving_primes, prime_count * sizeof(*d_sieving_primes)));
            checkCudaErrors(hipMalloc(&d_starting_multiples, prime_count * sizeof(*d_starting_multiples)));
            checkCudaErrors(hipMalloc(&d_prime_mod_inverses, prime_count * sizeof(*d_prime_mod_inverses)));
            checkCudaErrors(hipMalloc(&d_small_prime_offsets, Cuda_sieve::m_small_prime_count * sizeof(*d_small_prime_offsets)));
            checkCudaErrors(hipMalloc(&d_sieve, sieve_size * sizeof(*d_sieve)));
            checkCudaErrors(hipMalloc(&d_multiples, prime_count * Cuda_sieve::m_num_blocks * sizeof(*d_multiples)));
            checkCudaErrors(hipMalloc(&d_chains, Cuda_sieve::m_max_chains * sizeof(*d_chains)));
            checkCudaErrors(hipMalloc(&d_long_chains, Cuda_sieve::m_max_long_chains * sizeof(*d_long_chains)));
            checkCudaErrors(hipMalloc(&d_last_chain_index, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMalloc(&d_last_long_chain_index, sizeof(*d_last_long_chain_index)));
            checkCudaErrors(hipMalloc(&d_prime_candidate_count, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMalloc(&d_good_chain_index, sizeof(*d_good_chain_index)));
            checkCudaErrors(hipMalloc(&d_good_chains, Cuda_sieve::m_max_chains/2 * sizeof(*d_good_chains)));
            checkCudaErrors(hipMalloc(&d_chain_histogram, (Cuda_sieve::chain_histogram_max + 1) * sizeof(*d_chain_histogram)));
            checkCudaErrors(hipMalloc(&d_chain_stat_count, sizeof(*d_chain_stat_count)));


            //copy data to the gpu
            checkCudaErrors(hipMemcpy(d_sieving_primes, primes, prime_count * sizeof(*d_sieving_primes), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_prime_mod_inverses, prime_mod_inverses_host, prime_count * sizeof(*d_prime_mod_inverses), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset(d_last_chain_index, 0, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMemset(d_prime_candidate_count, 0, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
            checkCudaErrors(hipMemset(d_chain_stat_count, 0, sizeof(*d_chain_stat_count)));
            reset_stats();

        }

        //reset sieve with new starting offsets
        void Cuda_sieve_impl::init_sieve(uint32_t starting_multiples[], uint32_t small_prime_offsets[])
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipMemcpy(d_starting_multiples, starting_multiples, m_sieving_prime_count * sizeof(*d_starting_multiples), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_small_prime_offsets, small_prime_offsets, Cuda_sieve::m_small_prime_count * sizeof(*d_small_prime_offsets), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset(d_last_chain_index, 0, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMemset(d_prime_candidate_count, 0, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
        }

        void Cuda_sieve_impl::reset_stats()
        {
            checkCudaErrors(hipMemset(d_chain_histogram, 0, (Cuda_sieve::chain_histogram_max + 1) * sizeof(*d_chain_histogram)));
            checkCudaErrors(hipMemset(d_chain_stat_count, 0, sizeof(*d_chain_stat_count)));

        }

        void Cuda_sieve_impl::free_sieve()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_sieving_primes));
            checkCudaErrors(hipFree(d_starting_multiples));
            checkCudaErrors(hipFree(d_multiples));
            checkCudaErrors(hipFree(d_prime_mod_inverses));
            checkCudaErrors(hipFree(d_sieve));
            checkCudaErrors(hipFree(d_chains));
            checkCudaErrors(hipFree(d_last_chain_index));
            checkCudaErrors(hipFree(d_long_chains));
            checkCudaErrors(hipFree(d_last_long_chain_index));
            checkCudaErrors(hipFree(d_good_chains));
            checkCudaErrors(hipFree(d_good_chain_index));
            checkCudaErrors(hipFree(d_chain_histogram));
        }
    }
}