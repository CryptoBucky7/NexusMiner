#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sieve_impl.cuh"
#include "sieve.hpp"
#include "sieve_small_prime_constants.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>


#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

namespace nexusminer {
    namespace gpu {

        __device__ void cuda_chain_push_back(CudaChain& chain, uint16_t offset);
        __device__ void cuda_chain_open(CudaChain& chain, uint64_t base_offset);
        __device__  bool is_there_still_hope(CudaChain& chain);
        __device__  void get_best_fermat_chain(const CudaChain& chain, uint64_t& base_offset, int& offset, int& best_length);

        __device__ const unsigned int sieve30_offsets[]{ 1,7,11,13,17,19,23,29 };

        //__device__ const unsigned int sieve30_inverse_offsets[]{ 1,13,11,7,23,19,17,29 }; 

        __device__ const unsigned int sieve30_gaps[]{ 6,4,2,4,2,4,6,2 };

        __device__ const unsigned int sieve30_index[]
        { 0,0,1,1,1,1,1, 1, 2, 2, 2, 2, 3, 3, 4, 4, 4, 4, 5, 5, 6, 6, 6, 6, 7, 7, 7, 7, 7, 7 };  //reverse lookup table (offset mod 30 to index)

        //__device__ const unsigned int sieve30_inverse_index[]
        //{ 0,0,3,3,3,3,3, 3, 2, 2, 2, 2, 1, 1, 6, 6, 6, 6, 5, 5, 4, 4, 4, 4, 7, 7, 7, 7, 7, 7 };  //reverse lookup table (prime inverse mod 30 to index)

        __device__ const unsigned int prime_mod30_inverse[]
        { 1,1,13,13,13,13,13, 13, 11, 11, 11, 11, 7, 7, 23, 23, 23, 23, 19, 19, 17, 17, 17, 17, 29, 29, 29, 29, 29, 29 };  //lookup table - prime % 30 to prime inverse % 30

        __device__ const unsigned int next_multiple_mod30_offset[]  //range mod30 to the next highest prime.
        { 1,0,5,4,3,2,1, 0, 3, 2, 1, 0, 1, 0, 3, 2, 1, 0, 1, 0, 3, 2, 1, 0, 5, 4, 3, 2, 1, 0 };

        __device__ const unsigned int sieve120_index[]
        {    0, 0, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 4, 4, 4, 4, 5, 5, 6, 6, 6, 6, 7, 7, 7, 7, 7, 7, 
             8, 8, 9, 9, 9, 9, 9, 9,10,10,10,10,11,11,12,12,12,12,13,13,14,14,14,14,15,15,15,15,15,15,
            16,16,17,17,17,17,17,17,18,18,18,18,19,19,20,20,20,20,21,21,22,22,22,22,23,23,23,23,23,23,
            24,24,25,25,25,25,25,25,26,26,26,26,27,27,28,28,28,28,29,29,30,30,30,30,31,31,31,31,31,31
        };  //reverse lookup table (offset mod 120 to index)


        __device__  const Cuda_sieve::sieve_word_t unset_bit_mask[]{
            ~(1u << 0),  ~(1u << 1),  ~(1u << 2),  ~(1u << 3),  ~(1u << 4),  ~(1u << 5),  ~(1u << 6),  ~(1u << 7), 
            ~(1u << 8),  ~(1u << 9),  ~(1u << 10), ~(1u << 11), ~(1u << 12), ~(1u << 13), ~(1u << 14), ~(1u << 15),
            ~(1u << 16), ~(1u << 17), ~(1u << 18), ~(1u << 19), ~(1u << 20), ~(1u << 21), ~(1u << 22), ~(1u << 23),
            ~(1u << 24), ~(1u << 25), ~(1u << 26), ~(1u << 27), ~(1u << 28), ~(1u << 29), ~(1u << 30), ~(1u << 31)
        };
        
        // cross off small primes.  These primes hit the sieve often.  We iterate through the sieve words and cross them off using 
        // precalculated constants.  start is offset from the sieve start 
        __global__ void sieveSmallPrimes(Cuda_sieve::sieve_word_t* sieve, uint64_t start, uint32_t* small_prime_offsets)
        {

            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;

            const uint32_t increment = Cuda_sieve::m_sieve_word_range;

            //#pragma unroll
            for (uint64_t i = index; i < Cuda_sieve::m_sieve_total_size; i += stride) 
            {
                
                //the offset for the sieve word in process
                uint64_t inc = i * increment;
                //get the correct rotation for the prime mask
                //primes for reference 7,11,13,17,19,23,29,31,37,41,43,47,53,59,61,67,71,73,79,83,89,97,101
                //                     1, 2, 3, 4, 5, 6, 7, 8, 9,10,11,12,13,14,15,16,17,18,19,20,21,22, 23  

                uint16_t index7 = (start + small_prime_offsets[0] + inc) % 7;
                uint16_t index11 = (start + small_prime_offsets[1] + inc) % 11;
                uint16_t index13 = (start + small_prime_offsets[2] + inc) % 13;
                uint16_t index17 = (start + small_prime_offsets[3] + inc) % 17;
                uint16_t index19 = (start + small_prime_offsets[4] + inc) % 19;
                uint16_t index23 = (start + small_prime_offsets[5] + inc) % 23;
                uint16_t index29 = (start + small_prime_offsets[6] + inc) % 29;
                uint16_t index31 = (start + small_prime_offsets[7] + inc) % 31;
                uint16_t index37 = (start + small_prime_offsets[8] + inc) % 37;
                uint16_t index41 = (start + small_prime_offsets[9] + inc) % 41;
                uint16_t index43 = (start + small_prime_offsets[10] + inc) % 43;
                uint16_t index47 = (start + small_prime_offsets[11] + inc) % 47;
                uint16_t index53 = (start + small_prime_offsets[12] + inc) % 53;
                uint16_t index59 = (start + small_prime_offsets[13] + inc) % 59;
                uint16_t index61 = (start + small_prime_offsets[14] + inc) % 61;
                uint16_t index67 = (start + small_prime_offsets[15] + inc) % 67;
                uint16_t index71 = (start + small_prime_offsets[16] + inc) % 71;
                uint16_t index73 = (start + small_prime_offsets[17] + inc) % 73;
                uint16_t index79 = (start + small_prime_offsets[18] + inc) % 79;
                uint16_t index83 = (start + small_prime_offsets[19] + inc) % 83;
                uint16_t index89 = (start + small_prime_offsets[20] + inc) % 89;
                uint16_t index97 = (start + small_prime_offsets[21] + inc) % 97;
                uint16_t index101 = (start + small_prime_offsets[22] + inc) % 101;

               

                //apply the mask.  the mask for the first prime 7 is also used to initialize the sieve (hence no &).
                Cuda_sieve::sieve_word_t word;
                word = p7[index7];
                word &= p11[index11];
                word &= p13[index13];
                word &= p17[index17];
                word &= p19[index19];
                word &= p23[index23];
                word &= p29[index29];
                word &= p31[index31];
                word &= p37[index37];
                word &= p41[index41];
                word &= p43[index43];
                word &= p47[index47];
                word &= p53[index53];
                word &= p59[index59];
                word &= p61[index61];
                word &= p67[index67];
                word &= p71[index71];
                word &= p73[index73];
                word &= p79[index79];
                word &= p83[index83];
                word &= p89[index89];
                word &= p97[index97];
                word &= p101[index101];

                //save to global memory
                sieve[i] = word;

            }
        }

        //return the offset from x to the next integer multiple of n greater than x that is not divisible by 2, 3, or 5.  
       //x must be a multiple of the primorial 30 and n must be a prime greater than 5.
        template <typename T1, typename T2>
        __device__ __forceinline__ T2 get_offset_to_next_multiple(T1 x, T2 n)
        {
            T2 m = n - static_cast<T2>(x % n);            

           if (m % 2 == 0)
            {
                m += n;
            }
           if (m % 3 == 0 || m % 5 == 0)
           {
               m += 2 * n;
           }
           if (m % 3 == 0 || m % 5 == 0)
           {
               m += 2 * n;
           }
           
            return m;
        }

        //large primes hit the sieve no more than once per segment.  The large prime kernel works on a shared copy 
        //of the sieve one segment at a time.  The word and bit where the primes hit the segment are stored in the bucket array. 
        //The buckets must be filled prior to calling this kernel. We iterate through the hits in the bucket and cross off composites.  
        __global__ void sieveLargePrimes(uint64_t sieve_start_offset, uint32_t* sieving_primes,
            uint32_t* starting_multiples, uint32_t* large_prime_buckets, uint32_t* bucket_indices, Cuda_sieve::sieve_word_t* sieve_results)
        {
            //each kernel block works on one segment of the sieve.  
            unsigned int num_blocks = gridDim.x;
            unsigned int num_threads = blockDim.x;
            unsigned int block_id = blockIdx.x/Cuda_sieve::m_kernel_segments_per_block;
            unsigned int segment_id = blockIdx.x % Cuda_sieve::m_kernel_segments_per_block;
            unsigned int index = threadIdx.x;
            unsigned int stride = num_threads;

            if (block_id >= Cuda_sieve::m_num_blocks)
                return;

            //local shared copy of one segment of the sieve
            __shared__ Cuda_sieve::sieve_word_t sieve[Cuda_sieve::m_kernel_sieve_size_words];
            
            //each thread in the block initialize part of the shared sieve
            for (int j = index; j < Cuda_sieve::m_kernel_sieve_size_words; j += stride)
            {
                sieve[j] = ~0;
            }
            __syncthreads();

            //the number of sieve hits in this segment
            unsigned int sieve_hits = bucket_indices[block_id * Cuda_sieve::m_kernel_segments_per_block + segment_id];
            /*if (index == 0 && block_id == 0)
                printf("block %u segment %u sieve hits %u\n", block_id, segment_id, sieve_hits);*/
            uint32_t z = block_id;
            uint32_t y = segment_id;
            uint32_t x;
            const uint32_t zmax = Cuda_sieve::m_num_blocks;
            const uint32_t ymax = Cuda_sieve::m_kernel_segments_per_block;
            const uint32_t xmax = Cuda_sieve::m_large_prime_bucket_size;
            //iterate through the sieve hits
            for (unsigned int i = index; i < sieve_hits; i+=stride)
            {
                //unpack the sieve word and bit from the bucket data
                x = i;
                uint32_t bucket_data = large_prime_buckets[z * xmax * ymax + y * xmax + x];
                uint32_t sieve_word = (bucket_data >> 16) & 0x0000FFFF;
                uint32_t sieve_bit = bucket_data & 0x0000FFFF;
                //cross off the bit in the shared sieve
                uint32_t bit_mask = ~(1u << sieve_bit);
                atomicAnd(&sieve[sieve_word], bit_mask);
                //if (block_id == 1)
                //    printf("block %u segment %u bucket index %u word %u bit %u\n", block_id, segment_id, i, sieve_word, sieve_bit);
            }

            __syncthreads();

            //merge the sieve results back to global memory
            uint32_t sieve_results_index = blockIdx.x * Cuda_sieve::m_kernel_sieve_size_words;
            for (unsigned int j = index; j < Cuda_sieve::m_kernel_sieve_size_words; j += stride)
            {
                sieve_results[sieve_results_index + j] &= sieve[j];
            }

        }
        
        //get the nth bit from the sieve.
        __device__ __forceinline__ bool get_bit(uint64_t bit_position, Cuda_sieve::sieve_word_t* sieve)
        {
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            
            uint64_t word = bit_position / sieve_bits_per_word;
            unsigned bit_position_in_word = bit_position % sieve_bits_per_word;
            return ((sieve[word] >> bit_position_in_word) & 1) == 1;

        }

        //search the sieve for chains that meet the minimum length requirement.  
        __global__ void find_chain_kernel(Cuda_sieve::sieve_word_t* sieve, CudaChain* chains, uint32_t* chain_index, uint64_t sieve_start_offset,
            unsigned long long* chain_stat_count)
        {

            //const uint64_t sieve_size = Cuda_sieve::m_sieve_total_size;
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            const uint64_t sieve_total_bits = Cuda_sieve::m_sieve_total_size * sieve_bits_per_word;

            
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            unsigned int sieve_offset;
            unsigned int gap;
            uint64_t chain_start, prime_candidate_offset;

            //shared copies of lookup tables
            __shared__ unsigned int sieve30_offsets_shared[8];
            __shared__ unsigned int sieve30_gaps_shared[8];
            //local stats
            __shared__ uint32_t chain_count_shared;
            
            if (threadIdx.x < 8)
            {
                int i = threadIdx.x;
                sieve30_offsets_shared[i] = sieve30_offsets[i];
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            
            if (threadIdx.x == 0)
                chain_count_shared = 0;
            __syncthreads();
           
            //search each sieve location for a possible chain
            for (uint64_t i = index; i < sieve_total_bits; i += stride)
            {
              
                //gross checks to ensure its possible to form a chain
                uint64_t word = i / sieve_bits_per_word;
                if (sieve[word] == 0)
                    continue;
                //check if the next 4 bytes (4*30 = range of 120 integers) has enough prime candidates to form a chain 
                //this is only valid up to min chain length 9.  above 9 requires 5 bytes.
                if (word < Cuda_sieve::m_sieve_total_size - 1)
                {
                    unsigned int next_4_bytes = 0;
                    unsigned int byte_index = (i/8) % 4;
                    next_4_bytes = (sieve[word] >> (byte_index * 8)) & 0xFF;
                    next_4_bytes |= (((sieve[word + (byte_index >= 3 ? 1 : 0)] >> ((byte_index + 1) % 4) * 8) & 0xFF) << 8);
                    next_4_bytes |= (((sieve[word + (byte_index >= 2 ? 1 : 0)] >> ((byte_index + 2) % 4) * 8) & 0xFF) << 16);
                    next_4_bytes |= (((sieve[word + (byte_index >= 1 ? 1 : 0)] >> ((byte_index + 3) % 4) * 8) & 0xFF) << 24);

                    int popc = __popc(next_4_bytes);
                    if (popc < Cuda_sieve::m_min_chain_length)
                        continue;
                }

                //chain must start with a prime
                if (!get_bit(i, sieve))
                {
                    continue;
                }
                //search left for another prime less than max gap away
                uint64_t j = i - 1;
                gap = sieve30_gaps_shared[j % 8];
                while (j < i && gap <= maxGap)
                {
                    if (get_bit(j, sieve))
                    {
                        //there is a valid element to the left.  this is not the first element in a chain. abort.
                        break;
                    }
                    j--;
                    gap += sieve30_gaps_shared[j % 8];
                }
                if (gap <= maxGap)
                    continue;
                //this is the start of a possible chain.  search right
                //where are we in the wheel
                sieve_offset = sieve30_offsets_shared[i % 8u];
                chain_start = sieve_start_offset + i / 8 * 30 + sieve_offset;
                CudaChain current_chain;
                cuda_chain_open(current_chain, chain_start);
                j = i;
                gap = sieve30_gaps_shared[j % 8u];
                j++;
                while (j < sieve_total_bits && gap <= maxGap)
                {
                    if (get_bit(j, sieve))
                    {
                        //another possible candidate.  add it to the chain
                        gap = 0;
                        sieve_offset = sieve30_offsets_shared[j % 8u];
                        prime_candidate_offset = sieve_start_offset + j / 8 * 30 + sieve_offset;
                        uint16_t offset = prime_candidate_offset - chain_start;
                        //printf("%" PRIu64 " %u\n", chain_start, prime_candidate_offset);
                        cuda_chain_push_back(current_chain, offset);
                    }
                    gap += sieve30_gaps_shared[j % 8u];
                    j++;
                        
                }
                //we reached the end of the chain.  check if it meets the length requirement
                if (current_chain.m_offset_count >= Cuda_sieve::m_min_chain_length)
                {
                    //increment the chain list index
                    uint32_t chain_idx = atomicInc(chain_index, Cuda_sieve::m_max_chains);
                    //copy the current chain to the global list
                    chains[chain_idx] = current_chain;
                    //updated block level stats
                    atomicInc(&chain_count_shared, 0xFFFFFFFF);
                }
            }
            //update global chain stats
            __syncthreads();
            if (threadIdx.x == 0)
                atomicAdd(chain_stat_count, chain_count_shared);
        }

        //medium prime sieve.  We use a block of shared memory to sieve in segments.  Each block sieves a different range. 
        //the final results are merged with the global sieve at the end using atomicAnd. 
        __global__ void do_sieve(uint64_t sieve_start_offset, uint32_t* sieving_primes, uint32_t sieving_prime_count,
            uint32_t* starting_multiples, Cuda_sieve::sieve_word_t* sieve_results, uint32_t* multiples)
        {

            const uint32_t segment_size = Cuda_sieve::m_kernel_sieve_size_bytes * Cuda_sieve::m_sieve_byte_range;

            //local shared copy of the sieve
            __shared__ Cuda_sieve::sieve_word_t sieve[Cuda_sieve::m_kernel_sieve_size_words];
            //shared mem lookup tables
            __shared__ unsigned int sieve120_index_shared[120];
            //__shared__  Cuda_sieve::sieve_word_t unset_bit_mask_shared[32];
            __shared__ unsigned int sieve30_gaps_shared[8];
            __shared__ unsigned int sieve30_index_shared[30];
            __shared__ unsigned int prime_mod30_inverse_shared[30];
            //__shared__ unsigned int next_multiple_mod30_offset_shared[30];
           
            uint32_t block_id = blockIdx.x;
            uint32_t index = threadIdx.x;
            uint32_t stride = blockDim.x;
            uint32_t num_threads = blockDim.x;

            //initialize shared lookup tables.  lookup tables in shared memory are faster than global memory lookup tables.
            for (int i = index; i < 120; i += stride)
            {
                sieve120_index_shared[i] = sieve120_index[i];
            }
            /*for (int i = index; i < 32; i += stride)
            {
                unset_bit_mask_shared[i] = unset_bit_mask[i];
            }*/
            for (int i = index; i < 8; i += stride)
            {
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            for (int i = index; i < 30; i += stride)
            {
                sieve30_index_shared[i] = sieve30_index[i];
                prime_mod30_inverse_shared[i] = prime_mod30_inverse[i];
                //next_multiple_mod30_offset_shared[i] = next_multiple_mod30_offset[i];
            }

           
            const uint32_t segments = Cuda_sieve::m_kernel_segments_per_block;
            uint32_t sieve_results_index = block_id * Cuda_sieve::m_kernel_sieve_size_words_per_block;
            const uint32_t primes_per_thread = (sieving_prime_count + num_threads - 1) / num_threads;
            //each block sieves a different region
            uint64_t start_offset = sieve_start_offset + static_cast<uint64_t>(block_id) * Cuda_sieve::m_kernel_sieve_size_words_per_block * Cuda_sieve::m_sieve_word_range;
            
            uint8_t wheel_index;
            unsigned int next_wheel_gap;
            uint32_t j;
            uint32_t k;
            uint32_t prime_mod_inv;          
            for (int s = 0; s < segments; s++)
            {
                //everyone in the block initialize part of the shared sieve
                for (int j1 = index; j1 < Cuda_sieve::m_kernel_sieve_size_words; j1 += stride)
                {
                    sieve[j1] = ~0;
                }
                __syncthreads();

                for (uint32_t i = index; i < sieving_prime_count; i += stride)
                //for (uint32_t prime_index_offset = 0; prime_index_offset < primes_per_thread; prime_index_offset++)
                {
                    //uint32_t i = primes_per_thread * index + prime_index_offset;
                    /*if (i >= sieving_prime_count)
                        break;*/

                    k = sieving_primes[i];
                    
                    //get aligned to this region
                    if (s == 0)
                    {
                        j = starting_multiples[i];
                        
                        //printf("%" PRIu64 " %" PRIu64 " %u\n", start_offset, j, k);
                        //the first time through we need to calculate the starting offsets
                        if (start_offset >= j)
                        {
                            //j = get_offset_to_next_multiple(start_offset - j, sieving_primes[i]);
                            uint64_t x = start_offset - j;
                            //offset to the first integer multiple of the prime above the starting offset
                            uint32_t m = k - (x % k);
                            //find the next integer multiple of the prime that is not divisible by 2,3 or 5
                            m += (m % 2 == 0) ? k : 0;
                            m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                            m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                            j = m;
                            //this does the same thing as above - it gets the next multiple using prime inverse mod 30 and a lookup table. 
                            //prime_mod_inv = prime_mod30_inverse_shared[k % 30];
                            //j = m + k * next_multiple_mod30_offset_shared[((m % 30) * prime_mod_inv) % 30];
                        }

                        else
                            j -= start_offset;
                        
                    }
                    else
                    {
                        j = multiples[block_id * sieving_prime_count + i];
                        //calculating the wheel index each time is faster than saving and retrieving it from global memory each loop
                    }
                    prime_mod_inv = prime_mod30_inverse_shared[k % 30];
                    wheel_index = sieve30_index_shared[(prime_mod_inv * j) % 30];
                    next_wheel_gap = sieve30_gaps_shared[wheel_index];
                        
                    while (j < segment_size)
                    {
                        //cross off a multiple of the sieving prime
                        uint32_t sieve_index = j / Cuda_sieve::m_sieve_word_range;
                        //Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                        //    (sieve30_index[j % 30] + (8 * (j/Cuda_sieve::m_sieve_byte_range % Cuda_sieve::m_sieve_word_byte_count))));
                        
                        Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                            sieve120_index_shared[j % Cuda_sieve::m_sieve_word_range]);
                        //using this lookup table from shared memory is about the same as the hybrid version above
                        //Cuda_sieve::sieve_word_t bitmask = unset_bit_mask_shared[sieve120_index_shared[j % Cuda_sieve::m_sieve_word_range]];
                        
                        //printf("%" PRIu64 " %u\n", j, bitmask);
                        //if (k >= 821999 && block_id == 1)
                        //    printf("med sieve prime %u block %u segment %u sieve word %u bit %u\n", k, block_id, s, sieve_index, sieve120_index_shared[j % Cuda_sieve::m_sieve_word_range]);
                            
                        atomicAnd(&sieve[sieve_index], bitmask);
                        
                        //increment the next multiple of the current prime (rotate the wheel).
                        j += k * next_wheel_gap;
                        wheel_index = (wheel_index + 1) % 8;
                        next_wheel_gap = sieve30_gaps_shared[wheel_index];
                    }
                    //save the starting multiple for this prime for the next segment
                    multiples[block_id * sieving_prime_count + i] = j - segment_size;
                    
                }
                __syncthreads();
                

                //merge the sieve results back to global memory
                
                for (uint32_t j2 = index; j2 < Cuda_sieve::m_kernel_sieve_size_words; j2 += stride)
                {
                    if (j2 < Cuda_sieve::m_kernel_sieve_size_words)
                    {
                        sieve_results[sieve_results_index + j2] &= sieve[j2];

                    }
                }
                
                sieve_results_index += Cuda_sieve::m_kernel_sieve_size_words;
                start_offset += segment_size;
            }

        }

        //count the prime candidates in the global sieve
        __global__ void count_prime_candidates(Cuda_sieve::sieve_word_t* sieve, unsigned long long* prime_candidate_count)
        {
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            
            uint64_t count = 0;
            if (index == 0)
                *prime_candidate_count = 0;
            __syncthreads();

            for (uint64_t i = index; i < Cuda_sieve::m_sieve_total_size; i += stride)
            {
                count += __popcll(sieve[i]);
            }
            atomicAdd(prime_candidate_count, count);

        }

        //go through the list of chains.  copy winners to the long chain list.  copy survivors to a temporary chain
        __global__ void filter_busted_chains(CudaChain* chains, uint32_t* chain_index, CudaChain* surviving_chains,
            uint32_t* surviving_chain_index, CudaChain* long_chains, uint32_t* long_chain_index, uint32_t* histogram)
        {
            uint32_t num_threads = blockDim.x;
            uint32_t block_id = blockIdx.x;
            uint32_t index = block_id * num_threads + threadIdx.x;

            if (index >= *chain_index)
                return;
            if (index == 0)
            {
                *surviving_chain_index = 0;
            }
            __syncthreads();
            //printf("%" PRIu64 " %u\n", index, *chain_index);
            if (!is_there_still_hope(chains[index]))
            {
                //this chain is busted.  check how long it is
                //collect stats
                //only count chains 3 or longer to minimize memory accesses
                if (chains[index].m_prime_count >= 3)
                {
                    int chain_length, local_offset;
                    uint64_t base_offset;
                    get_best_fermat_chain(chains[index], base_offset, local_offset, chain_length);
                    uint32_t histogram_chain_length = min(chain_length, Cuda_sieve::chain_histogram_max);
                    if (chain_length >= 3)
                        atomicInc(&histogram[histogram_chain_length], 0xFFFFFFFF);

                    //check for winners
                    if (chain_length >= chains[index].m_min_chain_report_length)
                    {
                        //chain is long. save it. 
                        uint32_t last_long_chain_index = atomicInc(long_chain_index, Cuda_sieve::m_max_long_chains);
                        long_chains[last_long_chain_index] = chains[index];
                    }
                }
            }
            else
            {
                //copy chain to the survival list
                uint32_t last_surviving_chain_index = atomicInc(surviving_chain_index, Cuda_sieve::m_max_chains);
                surviving_chains[last_surviving_chain_index] = chains[index];
            }
        }

        //sort large primes into buckets by where they hit the sieve
        __global__ void sort_large_primes(uint64_t sieve_start_offset, uint32_t* large_primes,
            uint32_t* starting_multiples, uint32_t* large_prime_buckets, uint32_t* bucket_indices)
        {
            int num_blocks = gridDim.x;
            int num_threads = blockDim.x;
            int block_id = blockIdx.x;
            int index = threadIdx.x;
            int stride = num_threads;
            
            const uint32_t segment_size = Cuda_sieve::m_kernel_sieve_size_bytes * Cuda_sieve::m_sieve_byte_range;
            const uint32_t segments = Cuda_sieve::m_kernel_segments_per_block;
            const uint32_t block_range = segments * segment_size;

            //each block sieves a different region
            uint64_t start_offset = sieve_start_offset + static_cast<uint64_t>(block_id) * block_range;

            //shared mem lookup tables
            __shared__ unsigned int sieve30_gaps_shared[8];
            __shared__ unsigned int sieve30_index_shared[30];
            __shared__ unsigned int prime_mod30_inverse_shared[30];
            __shared__ unsigned int sieve120_index_shared[120];
            __shared__ unsigned int max_bucket;
            uint32_t bucket_index = 0;

            if (index == 0)
            {
                max_bucket = 0;
            }

            //initialize shared lookup tables.  lookup tables in shared memory are faster than global memory lookup tables.
            for (int i = index; i < 8; i += stride)
            {
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            for (int i = index; i < 30; i += stride)
            {
                sieve30_index_shared[i] = sieve30_index[i];
                prime_mod30_inverse_shared[i] = prime_mod30_inverse[i];
            }
            for (int i = index; i < 120; i += stride)
            {
                sieve120_index_shared[i] = sieve120_index[i];
            }

            //reset the bucket indices
            for (int i = index; i < Cuda_sieve::m_kernel_segments_per_block; i += stride)
            {
                bucket_indices[block_id* Cuda_sieve::m_kernel_segments_per_block + i] = 0;
            }
            __syncthreads();

            //iterate through the list of primes
            for (uint32_t i = index; i < Cuda_sieve::m_large_prime_count; i += stride)
            {
                uint32_t k = large_primes[i];
                uint32_t j = starting_multiples[i];

                //calculate the starting offsets for this block
                if (start_offset >= j)
                {
                    uint64_t x = start_offset - j;
                    //offset to the first integer multiple of the prime above the starting offset
                    uint32_t m = k - (x % k);
                    //find the next integer multiple of the prime that is not divisible by 2,3 or 5
                    m += (m % 2 == 0) ? k : 0;
                    m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                    m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                    j = m;
                }
                else
                    j -= start_offset;
                
                uint32_t prime_mod_inv = prime_mod30_inverse_shared[k % 30];
                uint8_t wheel_index = sieve30_index_shared[(prime_mod_inv * j) % 30];
                unsigned int next_wheel_gap = sieve30_gaps_shared[wheel_index];
                uint32_t next_segment = j / segment_size;
                uint32_t segment_offset = j % segment_size;
                while (next_segment < segments)
                {
                    //which word within the segment does the prime hit
                    uint32_t sieve_word = segment_offset / Cuda_sieve::m_sieve_word_range;
                    //which bit within the word does the prime hit
                    uint32_t sieve_bit = sieve120_index_shared[segment_offset % Cuda_sieve::m_sieve_word_range];
                    //pack the word index and bit into one 32 bit word
                    uint32_t sieve_segment_hit = (sieve_word << 16) | sieve_bit;
                    //add the sieve hit to the segment's bucket
                    bucket_index = atomicInc(&bucket_indices[block_id * Cuda_sieve::m_kernel_segments_per_block + next_segment], 0xFFFFFFFF);
                    //we are indexing a 1D array as if it were a 3D array. 
                    uint32_t z = block_id;
                    uint32_t y = next_segment;
                    uint32_t x = bucket_index;
                    const uint32_t zmax = Cuda_sieve::m_num_blocks;
                    const uint32_t ymax = segments;
                    const uint32_t xmax = Cuda_sieve::m_large_prime_bucket_size;
                    large_prime_buckets[z*xmax*ymax + y*xmax + x] = sieve_segment_hit;
                    
                    //if (/*k == 821999 &&*/ block_id == 1)
                    //    printf("Block %i Prime %u start offset %" PRIu64 " j %u next segment %u sieve word %u sieve bit %u bucket data %x bucket_index %u\n",
                    //        block_id, k, start_offset, j, next_segment, sieve_word, sieve_bit, sieve_segment_hit, bucket_index);
                   
                   
                    //increment the next multiple of the current prime (rotate the wheel).
                    j += k * next_wheel_gap;
                    wheel_index = (wheel_index + 1) % 8;
                    next_wheel_gap = sieve30_gaps_shared[wheel_index];
                    next_segment = j / segment_size;
                    segment_offset = j % segment_size;
                }
                //atomicMax(&max_bucket, bucket_index);
            }
           /* __syncthreads();
            if (index == 0)
                printf("block %u max bucket %u\n", block_id, max_bucket);*/

        }

        void Cuda_sieve_impl::run_large_prime_sieve(uint64_t sieve_start_offset)
        {

            int threads = 1024;
            //one kernel block per sieve block
            int blocks = Cuda_sieve::m_num_blocks;
            
            sort_large_primes << <blocks, threads >> > (sieve_start_offset, d_large_primes, d_large_prime_starting_multiples,
                d_large_prime_buckets, d_bucket_indices);

            //one kernel block per sieve segment
            blocks = Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block;
            threads = 512;
            sieveLargePrimes << <blocks, threads >> > (sieve_start_offset, d_large_primes, d_large_prime_starting_multiples, 
                d_large_prime_buckets, d_bucket_indices, d_sieve);
            

        }

        void Cuda_sieve_impl::run_small_prime_sieve(uint64_t sieve_start_offset)
        {
            const int threads = 256;
            const int blocks = (Cuda_sieve::m_sieve_total_size + threads - 1)/threads;
            
            sieveSmallPrimes << <blocks, threads >> > (d_sieve, sieve_start_offset, d_small_prime_offsets);

            //checkCudaErrors(hipDeviceSynchronize());
        }

        void Cuda_sieve_impl::run_sieve(uint64_t sieve_start_offset)
        {
            m_sieve_start_offset = sieve_start_offset;
            
            do_sieve <<<Cuda_sieve::m_num_blocks, Cuda_sieve::m_threads_per_block >>> (sieve_start_offset, d_sieving_primes, m_sieving_prime_count,
                d_starting_multiples, d_sieve, d_multiples);

            //checkCudaErrors(hipDeviceSynchronize());
        }

        void Cuda_sieve_impl::get_sieve(Cuda_sieve::sieve_word_t sieve[])
        {
            checkCudaErrors(hipMemcpy(sieve, d_sieve, Cuda_sieve::m_sieve_total_size * sizeof(*d_sieve), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::get_prime_candidate_count(uint64_t& prime_candidate_count)
        {
            const int threads = 256;
            const int blocks = 1; // (Cuda_sieve::m_sieve_total_size + threads - 1) / threads;
            count_prime_candidates << <blocks, threads >> > (d_sieve, d_prime_candidate_count);
            checkCudaErrors(hipDeviceSynchronize());
            
            checkCudaErrors(hipMemcpy(&prime_candidate_count, d_prime_candidate_count, sizeof(*d_prime_candidate_count), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::find_chains()
        {
            const int sieve_threads = 128;
            const int checks_per_block = 32;
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            const uint64_t sieve_total_bits = Cuda_sieve::m_sieve_total_size * sieve_bits_per_word;
            const int sieve_blocks = (sieve_total_bits /checks_per_block + sieve_threads - 1)/ sieve_threads;
            find_chain_kernel << <sieve_blocks, sieve_threads >> > (d_sieve, d_chains, d_last_chain_index, m_sieve_start_offset, d_chain_stat_count);

            //checkCudaErrors(hipDeviceSynchronize());
            
        }

        void Cuda_sieve_impl::get_chains(CudaChain chains[], uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(chains, d_chains, chain_count * sizeof(*d_chains), hipMemcpyDeviceToHost));
        }

        void Cuda_sieve_impl::get_chain_count(uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToHost));
        }

        //get a pointer to the chain array.  fermat test uses the chain array as input. 
        void Cuda_sieve_impl::get_chain_pointer(CudaChain*& chains_ptr, uint32_t*& chain_count_ptr)
        {
            chains_ptr = d_chains;
            chain_count_ptr = d_last_chain_index;
        }

        //check the list of chains for winners.  save winners and remove losers
        void Cuda_sieve_impl::clean_chains()
        {
            const int threads = 256;
            uint32_t chain_count;
            get_chain_count(chain_count);
            int blocks = (chain_count + threads - 1) / threads;
            //copy surviving chains to a temporary location. 
            filter_busted_chains << <blocks, threads >> > (d_chains, d_last_chain_index, d_good_chains, d_good_chain_index,
                d_long_chains, d_last_long_chain_index, d_chain_histogram);
            //checkCudaErrors(hipDeviceSynchronize());
            uint32_t good_chain_count;
            //get the count of good chains from device memory
            checkCudaErrors(hipMemcpy(&good_chain_count, d_good_chain_index, sizeof(*d_good_chain_index), hipMemcpyDeviceToHost));
            //copy the temporary good chain list back to the chain list
            checkCudaErrors(hipMemcpyAsync(d_chains, d_good_chains, good_chain_count*sizeof(*d_chains), hipMemcpyDeviceToDevice));
            //update the chain count
            checkCudaErrors(hipMemcpy(d_last_chain_index, d_good_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToDevice));

        }

        void Cuda_sieve_impl::get_long_chains(CudaChain chains[], uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_long_chain_index, sizeof(*d_last_long_chain_index), hipMemcpyDeviceToHost));
            if (chain_count > 0)
            {
                checkCudaErrors(hipMemcpy(chains, d_long_chains, chain_count * sizeof(*d_long_chains), hipMemcpyDeviceToHost));
                //clear the long chain list
                checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
            }
        }

        //read the histogram
        void Cuda_sieve_impl::get_stats(uint32_t chain_histogram[], uint64_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(chain_histogram, d_chain_histogram, (Cuda_sieve::chain_histogram_max+1) * sizeof(*d_chain_histogram), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_stat_count, sizeof(*d_chain_stat_count), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::synchronize()
        {
            checkCudaErrors(hipDeviceSynchronize());
        }

        //allocate global memory and load values used by the sieve to the gpu 
        void Cuda_sieve_impl::load_sieve(uint32_t primes[], uint32_t prime_count, uint32_t large_primes[], uint32_t sieve_size, uint16_t device)
        {
          
            m_sieving_prime_count = prime_count;
            m_device = device;
            checkCudaErrors(hipSetDevice(device));
            //allocate memory on the gpu
            checkCudaErrors(hipMalloc(&d_sieving_primes, prime_count * sizeof(*d_sieving_primes)));
            checkCudaErrors(hipMalloc(&d_starting_multiples, prime_count * sizeof(*d_starting_multiples)));
            checkCudaErrors(hipMalloc(&d_small_prime_offsets, Cuda_sieve::m_small_prime_count * sizeof(*d_small_prime_offsets)));
            checkCudaErrors(hipMalloc(&d_large_primes, Cuda_sieve::m_large_prime_count * sizeof(*d_large_primes)));
            checkCudaErrors(hipMalloc(&d_large_prime_starting_multiples, Cuda_sieve::m_large_prime_count * sizeof(*d_large_prime_starting_multiples)));
            checkCudaErrors(hipMalloc(&d_large_prime_buckets, Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block
                * Cuda_sieve::m_large_prime_bucket_size * sizeof(*d_large_prime_buckets)));
            checkCudaErrors(hipMalloc(&d_bucket_indices, Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block * sizeof(*d_bucket_indices)));
            checkCudaErrors(hipMalloc(&d_sieve, sieve_size * sizeof(*d_sieve)));
            checkCudaErrors(hipMalloc(&d_multiples, prime_count * Cuda_sieve::m_num_blocks * sizeof(*d_multiples)));
            checkCudaErrors(hipMalloc(&d_chains, Cuda_sieve::m_max_chains * sizeof(*d_chains)));
            checkCudaErrors(hipMalloc(&d_long_chains, Cuda_sieve::m_max_long_chains * sizeof(*d_long_chains)));
            checkCudaErrors(hipMalloc(&d_last_chain_index, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMalloc(&d_last_long_chain_index, sizeof(*d_last_long_chain_index)));
            checkCudaErrors(hipMalloc(&d_prime_candidate_count, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMalloc(&d_good_chain_index, sizeof(*d_good_chain_index)));
            checkCudaErrors(hipMalloc(&d_good_chains, Cuda_sieve::m_max_chains/2 * sizeof(*d_good_chains)));
            checkCudaErrors(hipMalloc(&d_chain_histogram, (Cuda_sieve::chain_histogram_max + 1) * sizeof(*d_chain_histogram)));
            checkCudaErrors(hipMalloc(&d_chain_stat_count, sizeof(*d_chain_stat_count)));


            //copy data to the gpu
            checkCudaErrors(hipMemcpy(d_sieving_primes, primes, prime_count * sizeof(*d_sieving_primes), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_large_primes, large_primes, Cuda_sieve::m_large_prime_count * sizeof(*d_large_primes), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset(d_last_chain_index, 0, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMemset(d_prime_candidate_count, 0, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
            checkCudaErrors(hipMemset(d_chain_stat_count, 0, sizeof(*d_chain_stat_count)));
            reset_stats();

        }

        //reset sieve with new starting offsets
        void Cuda_sieve_impl::init_sieve(uint32_t starting_multiples[], uint32_t small_prime_offsets[], uint32_t large_prime_multiples[])
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipMemcpy(d_starting_multiples, starting_multiples, m_sieving_prime_count * sizeof(*d_starting_multiples), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_large_prime_starting_multiples, large_prime_multiples, Cuda_sieve::m_large_prime_count * sizeof(*d_large_prime_starting_multiples), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_small_prime_offsets, small_prime_offsets, Cuda_sieve::m_small_prime_count * sizeof(*d_small_prime_offsets), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset(d_last_chain_index, 0, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMemset(d_prime_candidate_count, 0, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
        }

        void Cuda_sieve_impl::reset_stats()
        {
            checkCudaErrors(hipMemset(d_chain_histogram, 0, (Cuda_sieve::chain_histogram_max + 1) * sizeof(*d_chain_histogram)));
            checkCudaErrors(hipMemset(d_chain_stat_count, 0, sizeof(*d_chain_stat_count)));

        }

        void Cuda_sieve_impl::free_sieve()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_sieving_primes));
            checkCudaErrors(hipFree(d_large_primes));
            checkCudaErrors(hipFree(d_starting_multiples));
            checkCudaErrors(hipFree(d_multiples));
            checkCudaErrors(hipFree(d_sieve));
            checkCudaErrors(hipFree(d_chains));
            checkCudaErrors(hipFree(d_last_chain_index));
            checkCudaErrors(hipFree(d_long_chains));
            checkCudaErrors(hipFree(d_last_long_chain_index));
            checkCudaErrors(hipFree(d_good_chains));
            checkCudaErrors(hipFree(d_good_chain_index));
            checkCudaErrors(hipFree(d_chain_histogram));
            checkCudaErrors(hipFree(d_large_prime_buckets));
            checkCudaErrors(hipFree(d_bucket_indices));

        }
    }
}