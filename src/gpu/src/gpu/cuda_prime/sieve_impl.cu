#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sieve_impl.cuh"
#include "sieve.hpp"
#include "sieve_small_prime_constants.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>


#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

namespace nexusminer {
    namespace gpu {

        __device__ void cuda_chain_push_back(CudaChain& chain, uint16_t offset);
        __device__ void cuda_chain_open(CudaChain& chain, uint64_t base_offset);

        __device__ const unsigned int sieve30_offsets[]{ 1,7,11,13,17,19,23,29 };

        __device__ const unsigned int sieve30_gaps[]{ 6,4,2,4,2,4,6,2 };

        __device__ const unsigned int sieve30_index[]
        { 0,0,1,1,1,1,1, 1, 2, 2, 2, 2, 3, 3, 4, 4, 4, 4, 5, 5, 6, 6, 6, 6, 7, 7, 7, 7, 7, 7 };  //reverse lookup table (offset mod 30 to index)

        //__device__ const unsigned int sieve120_index[]
        //{    0, 0, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 4, 4, 4, 4, 5, 5, 6, 6, 6, 6, 7, 7, 7, 7, 7, 7, 
        //     8, 8, 9, 9, 9, 9, 9, 9,10,10,10,10,11,11,12,12,12,12,13,13,14,14,14,14,15,15,15,15,15,15,
        //    16,16,17,17,17,17,17,17,18,18,18,18,19,19,20,20,20,20,21,21,22,22,22,22,23,23,23,23,23,23,
        //    24,24,25,25,25,25,25,25,26,26,26,26,27,27,28,28,28,28,29,29,30,30,30,30,31,31,31,31,31,31
        //};  //reverse lookup table (offset mod 120 to index)


        //__device__  const Cuda_sieve::sieve_word_t unset_bit_mask[]{
        //    ~(1u << 0),  ~(1u << 1),  ~(1u << 2),  ~(1u << 3),  ~(1u << 4),  ~(1u << 5),  ~(1u << 6),  ~(1u << 7), 
        //    ~(1u << 8),  ~(1u << 9),  ~(1u << 10), ~(1u << 11), ~(1u << 12), ~(1u << 13), ~(1u << 14), ~(1u << 15),
        //    ~(1u << 16), ~(1u << 17), ~(1u << 18), ~(1u << 19), ~(1u << 20), ~(1u << 21), ~(1u << 22), ~(1u << 23),
        //    ~(1u << 24), ~(1u << 25), ~(1u << 26), ~(1u << 27), ~(1u << 28), ~(1u << 29), ~(1u << 30), ~(1u << 31)
        //};
        
        // cross off small primes.  These primes hit the sieve often.  We iterate through the sieve words and cross them off using 
        // precalculated constants.  start is offset from the sieve start 
        __global__ void sieveSmallPrimes(Cuda_sieve::sieve_word_t* sieve, uint64_t start, uint32_t* small_prime_offsets)
        {

            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;

            const uint32_t increment = Cuda_sieve::m_sieve_word_range;

            //#pragma unroll
            for (uint64_t i = index; i < Cuda_sieve::m_sieve_total_size; i += stride) 
            {
                
                //the offset for the sieve word in process
                uint64_t inc = i * increment;
                //get the correct rotation for the prime mask
                //primes for reference 7,11,13,17,19,23,29,31,37,41,43,47,53,59,61,67,71,73,79,83,89,97,101
                //                     1, 2, 3, 4, 5, 6, 7, 8, 9,10,11,12,13,14,15,16,17,18,19,20,21,22, 23  

                uint16_t index7 = (start + small_prime_offsets[0] + inc) % 7;
                uint16_t index11 = (start + small_prime_offsets[1] + inc) % 11;
                uint16_t index13 = (start + small_prime_offsets[2] + inc) % 13;
                uint16_t index17 = (start + small_prime_offsets[3] + inc) % 17;
                uint16_t index19 = (start + small_prime_offsets[4] + inc) % 19;
                uint16_t index23 = (start + small_prime_offsets[5] + inc) % 23;
                uint16_t index29 = (start + small_prime_offsets[6] + inc) % 29;
                uint16_t index31 = (start + small_prime_offsets[7] + inc) % 31;
                uint16_t index37 = (start + small_prime_offsets[8] + inc) % 37;
                uint16_t index41 = (start + small_prime_offsets[9] + inc) % 41;
                uint16_t index43 = (start + small_prime_offsets[10] + inc) % 43;
                uint16_t index47 = (start + small_prime_offsets[11] + inc) % 47;
                uint16_t index53 = (start + small_prime_offsets[12] + inc) % 53;
                uint16_t index59 = (start + small_prime_offsets[13] + inc) % 59;
                uint16_t index61 = (start + small_prime_offsets[14] + inc) % 61;
                uint16_t index67 = (start + small_prime_offsets[15] + inc) % 67;
                uint16_t index71 = (start + small_prime_offsets[16] + inc) % 71;
                uint16_t index73 = (start + small_prime_offsets[17] + inc) % 73;
                uint16_t index79 = (start + small_prime_offsets[18] + inc) % 79;
                uint16_t index83 = (start + small_prime_offsets[19] + inc) % 83;
                uint16_t index89 = (start + small_prime_offsets[20] + inc) % 89;
                uint16_t index97 = (start + small_prime_offsets[21] + inc) % 97;
                uint16_t index101 = (start + small_prime_offsets[22] + inc) % 101;

               

                //apply the mask.  the mask for the first prime 7 is also used to initialize the sieve (hence no &).
                Cuda_sieve::sieve_word_t word;
                word = p7[index7];
                word &= p11[index11];
                word &= p13[index13];
                word &= p17[index17];
                word &= p19[index19];
                word &= p23[index23];
                word &= p29[index29];
                word &= p31[index31];
                word &= p37[index37];
                word &= p41[index41];
                word &= p43[index43];
                word &= p47[index47];
                word &= p53[index53];
                word &= p59[index59];
                word &= p61[index61];
                word &= p67[index67];
                word &= p71[index71];
                word &= p73[index73];
                word &= p79[index79];
                word &= p83[index83];
                word &= p89[index89];
                word &= p97[index97];
                word &= p101[index101];

                //save to global memory
                sieve[i] = word;

            }
        }

        //return the offset from x to the next integer multiple of n greater than x that is not divisible by 2, 3, or 5.  
       //x must be a multiple of the primorial 30 and n must be a prime greater than 5.
        template <typename T1, typename T2>
        __device__ __forceinline__ T2 get_offset_to_next_multiple(T1 x, T2 n)
        {
            T2 m = n - static_cast<T2>(x % n);
            if (m % 2 == 0)
            {
                m += n;
            }
            while (m % 3 == 0 || m % 5 == 0)
            {
                m += 2 * n;
            }
            return m;
        }

        //large primes hit the sieve infrequently with large gaps (> 1 segment) between hits.  The optimizations for
        //medium primes hurts more than helps.  For large primes we simply iterate through multiples of the primes and cross 
        //them off one by one in global memory using atomicAnd.  The memory conflicts between primes should be few because
        // of the infrequency of the hits to the sieve. 
        __global__ void sieveLargePrimes(uint64_t sieve_start_offset, uint32_t* sieving_primes, uint32_t sieving_prime_count,
            uint32_t* starting_multiples, uint8_t* prime_mod_inverses, Cuda_sieve::sieve_word_t* sieve)
        {

            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            uint64_t wheel_index;
            unsigned int next_wheel_gap;
            uint64_t j;
            uint64_t k;
            
            //iterate through each prime starting at the large prime cutoff prime
            for (uint32_t i = index + Cuda_sieve::m_large_prime_cutoff_index; i < sieving_prime_count; i += stride)
            {
                //calculate the starting offset for the current prime
                j = starting_multiples[i];
                if (sieve_start_offset >= j)
                    j = get_offset_to_next_multiple(sieve_start_offset - j, sieving_primes[i]);
                else
                    j -= sieve_start_offset;
                k = sieving_primes[i];
                wheel_index = sieve30_index[(prime_mod_inverses[i] * j) % 30];
                next_wheel_gap = sieve30_gaps[wheel_index];

                while (j < Cuda_sieve::m_sieve_range)
                {
                    //cross off a multiple of the sieving prime
                    uint64_t sieve_index = j / Cuda_sieve::m_sieve_word_range;
                    Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                        (sieve30_index[j % 30] + (8 * (j / Cuda_sieve::m_sieve_byte_range % Cuda_sieve::m_sieve_word_byte_count))));

                    //todo: test lookup table version
                    //Cuda_sieve::sieve_word_t bitmask2 = unset_bit_mask[sieve120_index[j % 120u]];

                    //printf("%" PRIu64 " %u\n", j, bitmask);

                    
                    atomicAnd(&sieve[sieve_index], bitmask);

                    //increment the next multiple of the current prime (rotate the wheel).
                    j += k * next_wheel_gap;
                    wheel_index++;
                    next_wheel_gap = sieve30_gaps[wheel_index % 8];
                }

            }

        }
        
        //get the nth bit from the sieve.
        __device__ __forceinline__ bool get_bit(uint64_t bit_position, Cuda_sieve::sieve_word_t* sieve)
        {
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            
            uint64_t word = bit_position / sieve_bits_per_word;
            unsigned bit_position_in_word = bit_position % sieve_bits_per_word;
            return ((sieve[word] >> bit_position_in_word) & 1) == 1;

        }

        //search the sieve for chains that meet the minimum length requirement.  
        __global__ void find_chain_kernel(Cuda_sieve::sieve_word_t* sieve, CudaChain* chains, uint32_t* chain_index, uint64_t sieve_start_offset)
        {

            //const uint64_t sieve_size = Cuda_sieve::m_sieve_total_size;
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            const uint64_t sieve_total_bits = Cuda_sieve::m_sieve_total_size * sieve_bits_per_word;

            
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            unsigned int sieve_offset;
            unsigned int gap;
            uint64_t chain_start, prime_candidate_offset;
            
            
            if (index == 0)
                *chain_index = 0;
            __syncthreads();
           
            //search each sieve location for a possible chain
            for (uint64_t i = index; i < sieve_total_bits; i += stride)
            {
              
                //gross checks to ensure its possible to form a chain
                uint64_t word = i / sieve_bits_per_word;
                if (sieve[word] == 0)
                    return;
                //check if the next 4 bytes (4*30 = range of 120 integers) has enough prime candidates to form a chain 
                //this is only valid up to min chain length 9.  above 9 requires 5 bytes.
                if (word < Cuda_sieve::m_sieve_total_size - 1)
                {
                    unsigned int next_4_bytes = 0;
                    unsigned int byte_index = (i/8) % 4;
                    next_4_bytes = (sieve[word] >> (byte_index * 8)) & 0xFF;
                    next_4_bytes |= (((sieve[word + (byte_index >= 3 ? 1 : 0)] >> ((byte_index + 1) % 4) * 8) & 0xFF) << 8);
                    next_4_bytes |= (((sieve[word + (byte_index >= 2 ? 1 : 0)] >> ((byte_index + 2) % 4) * 8) & 0xFF) << 16);
                    next_4_bytes |= (((sieve[word + (byte_index >= 1 ? 1 : 0)] >> ((byte_index + 3) % 4) * 8) & 0xFF) << 24);

                    int popc = __popc(next_4_bytes);
                    if (popc < Cuda_sieve::m_min_chain_length)
                        return;
                }

                //chain must start with a prime
                if (!get_bit(i, sieve))
                {
                    return;
                }
                //search left for another prime less than max gap away
                uint64_t j = i - 1;
                gap = sieve30_gaps[j % 8];
                while (j < i && gap <= maxGap)
                {
                    if (get_bit(j, sieve))
                    {
                        //there is a valid element to the left.  this is not the first element in a chain. abort.
                        return;
                    }
                    j--;
                    gap += sieve30_gaps[j % 8];
                }
                   
                //this is the start of a possible chain.  search right
                //where are we in the wheel
                sieve_offset = sieve30_offsets[i % 8u];
                chain_start = sieve_start_offset + i / 8 * 30 + sieve_offset;
                CudaChain current_chain;
                cuda_chain_open(current_chain, chain_start);
                j = i;
                gap = sieve30_gaps[j % 8u];
                j++;
                while (j < sieve_total_bits && gap <= maxGap)
                {
                    if (get_bit(j, sieve))
                    {
                        //another possible candidate.  add it to the chain
                        gap = 0;
                        sieve_offset = sieve30_offsets[j % 8u];
                        prime_candidate_offset = sieve_start_offset + j / 8 * 30 + sieve_offset;
                        uint16_t offset = prime_candidate_offset - chain_start;
                        //printf("%" PRIu64 " %u\n", chain_start, prime_candidate_offset);
                        cuda_chain_push_back(current_chain, offset);
                    }
                    gap += sieve30_gaps[j % 8u];
                    j++;
                        
                }
                //we reached the end of the chain.  check if it meets the length requirement
                if (current_chain.m_offset_count >= Cuda_sieve::m_min_chain_length)
                {
                    //increment the chain list index
                    uint32_t chain_idx = atomicInc(chain_index, Cuda_sieve::m_max_chains);
                    //copy the current chain to the global list
                    chains[chain_idx] = current_chain;
                }
               
            }
            

        }

       
       

        //medium prime sieve.  We use a block of shared memory to sieve in segments.  Each block sieves a different range. 
        //the final results are merged with the global sieve at the end using atomicAnd. 
        __global__ void do_sieve(uint64_t sieve_start_offset, uint32_t* sieving_primes, uint32_t sieving_prime_count,
            uint32_t* starting_multiples, uint8_t* prime_mod_inverses, Cuda_sieve::sieve_word_t* sieve_results, uint32_t* multiples)
        {

            const uint32_t segment_size = Cuda_sieve::m_kernel_sieve_size_bytes * Cuda_sieve::m_sieve_byte_range;

            //local shared copy of the sieve
            __shared__ Cuda_sieve::sieve_word_t sieve[Cuda_sieve::m_kernel_sieve_size_words];

            uint64_t block_id = blockIdx.x;
            uint64_t index = threadIdx.x;
            uint64_t stride = blockDim.x;
           
            const uint64_t segments = Cuda_sieve::m_kernel_segments_per_block;
            uint64_t sieve_results_index = block_id * Cuda_sieve::m_kernel_sieve_size_words_per_block;

            //each block sieves a different region
            uint64_t start_offset = sieve_start_offset + block_id * Cuda_sieve::m_kernel_sieve_size_words_per_block * Cuda_sieve::m_sieve_word_range;
            
            uint64_t wheel_index;
            unsigned int next_wheel_gap;
            uint64_t j;
            uint64_t k;
            uint32_t max_prime_index = min(sieving_prime_count, Cuda_sieve::m_large_prime_cutoff_index);
            for (int s = 0; s < segments; s++)
            {
                
                //everyone in the block initialize part of the shared sieve
                for (int j1 = index; j1 < Cuda_sieve::m_kernel_sieve_size_words; j1 += stride)
                {
                    sieve[j1] = ~0;
                }

                __syncthreads();
                for (uint32_t i = index; i < max_prime_index; i += stride)
                {
                    
                    k = sieving_primes[i];
                    //get aligned to this region
                    if (s == 0)
                    {
                        j = starting_multiples[i];
                        //the first time through we need to calculate the starting offsets
                        if (start_offset >= j)
                            j = get_offset_to_next_multiple(start_offset - j, sieving_primes[i]);
                        else
                            j -= start_offset;
                        
                    }
                    else
                    {
                        j = multiples[block_id* sieving_prime_count +i];
                        //calculating the wheel index each time is faster than saving and retrieving it from global memory each loop
                        //wheel_index = wheel_indices[block_id * sieving_prime_count + i];
                    }
                    wheel_index = sieve30_index[(prime_mod_inverses[i] * j) % 30];
                    next_wheel_gap = sieve30_gaps[wheel_index];
                        
                    while (j < segment_size)
                    {
                        //cross off a multiple of the sieving prime
                        uint64_t sieve_index = j / Cuda_sieve::m_sieve_word_range;
                        Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                            (sieve30_index[j % 30] + (8 * (j/Cuda_sieve::m_sieve_byte_range % Cuda_sieve::m_sieve_word_byte_count))));
                        
                        //using this lookup table is a bit slower than the calculated version.
                        //Cuda_sieve::sieve_word_t bitmask2 = unset_bit_mask[sieve120_index[j % 120u]];
                        
                        //printf("%" PRIu64 " %u\n", j, bitmask);
                            
                        atomicAnd(&sieve[sieve_index], bitmask);
                        
                        //increment the next multiple of the current prime (rotate the wheel).
                        j += k * next_wheel_gap;
                        wheel_index++;
                        next_wheel_gap = sieve30_gaps[wheel_index % 8];
                    }
                    //save the starting multiple for this prime for the next segment
                    multiples[block_id * sieving_prime_count + i] = j - segment_size;
                    
                }
                __syncthreads();
                

                //merge the sieve results back to global memory
                
                for (uint32_t j2 = index; j2 < Cuda_sieve::m_kernel_sieve_size_words; j2 += stride)
                {
                    if (j2 < Cuda_sieve::m_kernel_sieve_size_words)
                    {
                        sieve_results[sieve_results_index + j2] &= sieve[j2];

                    }
                }
                
                sieve_results_index += Cuda_sieve::m_kernel_sieve_size_words;
            }

        }

        //count the prime candidates in the global sieve
        __global__ void count_prime_candidates(Cuda_sieve::sieve_word_t* sieve, unsigned long long* prime_candidate_count)
        {
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            
            uint64_t count = 0;
            if (index == 0)
                *prime_candidate_count = 0;
            __syncthreads();

            for (uint64_t i = index; i < Cuda_sieve::m_sieve_total_size; i += stride)
            {
                count += __popcll(sieve[i]);
            }
            atomicAdd(prime_candidate_count, count);

        }

        void Cuda_sieve_impl::run_large_prime_sieve(uint64_t sieve_start_offset)
        {
            const int threads = 256;
            const int primes_per_block = 1;
            uint32_t large_prime_count = m_sieving_prime_count - Cuda_sieve::m_large_prime_cutoff_index;
            int blocks = (large_prime_count/ primes_per_block + threads - 1) / threads;
            if (Cuda_sieve::m_large_prime_cutoff_index < m_sieving_prime_count)
            {
                sieveLargePrimes << <blocks, threads >> > (sieve_start_offset, d_sieving_primes, m_sieving_prime_count,
                    d_starting_multiples, d_prime_mod_inverses, d_sieve);
                checkCudaErrors(hipDeviceSynchronize());
            }

        }

        void Cuda_sieve_impl::run_small_prime_sieve(uint64_t sieve_start_offset)
        {
            const int threads = 256;
            const int blocks = (Cuda_sieve::m_sieve_total_size + threads - 1)/threads;
            
            sieveSmallPrimes << <blocks, threads >> > (d_sieve, sieve_start_offset, d_small_prime_offsets);

            checkCudaErrors(hipDeviceSynchronize());
        }

        void Cuda_sieve_impl::run_sieve(uint64_t sieve_start_offset)
        {
            m_sieve_start_offset = sieve_start_offset;
            
            do_sieve <<<Cuda_sieve::m_num_blocks, Cuda_sieve::m_threads_per_block >>> (sieve_start_offset, d_sieving_primes, m_sieving_prime_count,
                d_starting_multiples, d_prime_mod_inverses, d_sieve, d_multiples);

            checkCudaErrors(hipDeviceSynchronize());
        }

        void Cuda_sieve_impl::get_sieve(Cuda_sieve::sieve_word_t sieve[])
        {
            checkCudaErrors(hipMemcpy(sieve, d_sieve, Cuda_sieve::m_sieve_total_size * sizeof(*d_sieve), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::get_prime_candidate_count(uint64_t& prime_candidate_count)
        {
            const int threads = 256;
            const int blocks = 1; // (Cuda_sieve::m_sieve_total_size + threads - 1) / threads;
            count_prime_candidates << <blocks, threads >> > (d_sieve, d_prime_candidate_count);
            checkCudaErrors(hipDeviceSynchronize());
            
            checkCudaErrors(hipMemcpy(&prime_candidate_count, d_prime_candidate_count, sizeof(*d_prime_candidate_count), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::find_chains(CudaChain chains[], uint32_t& chain_count)
        {
            const int sieve_threads = 128;
            const int checks_per_block = 1;
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            const uint64_t sieve_total_bits = Cuda_sieve::m_sieve_total_size * sieve_bits_per_word;
            const int sieve_blocks = (sieve_total_bits /checks_per_block + sieve_threads - 1)/ sieve_threads;
            
            //run the kernel
            find_chain_kernel << <sieve_blocks, sieve_threads >> > (d_sieve, d_chains, d_chain_index, m_sieve_start_offset);

            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_index, sizeof(*d_chain_index), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(chains, d_chains, chain_count * sizeof(*d_chains), hipMemcpyDeviceToHost));
        }

        //allocate global memory and load values used by the sieve to the gpu 
        void Cuda_sieve_impl::load_sieve(uint32_t primes[], uint32_t prime_count, uint32_t starting_multiples_host[],
            uint8_t prime_mod_inverses_host[], uint32_t small_prime_offsets[], uint32_t sieve_size, uint16_t device)
        {
          
            m_sieving_prime_count = prime_count;
            checkCudaErrors(hipSetDevice(device));
            //allocate memory on the gpu
            checkCudaErrors(hipMalloc(&d_sieving_primes, prime_count * sizeof(*d_sieving_primes)));
            checkCudaErrors(hipMalloc(&d_starting_multiples, prime_count * sizeof(*d_starting_multiples)));
            checkCudaErrors(hipMalloc(&d_prime_mod_inverses, prime_count * sizeof(*d_prime_mod_inverses)));
            checkCudaErrors(hipMalloc(&d_small_prime_offsets, Cuda_sieve::m_small_prime_count * sizeof(*d_small_prime_offsets)));
            checkCudaErrors(hipMalloc(&d_sieve, sieve_size * sizeof(*d_sieve)));
            checkCudaErrors(hipMalloc(&d_multiples, prime_count * Cuda_sieve::m_num_blocks * sizeof(*d_multiples)));
            //checkCudaErrors(hipMalloc(&d_wheel_indices, prime_count * Cuda_sieve::m_num_blocks * sizeof(*d_wheel_indices)));
            checkCudaErrors(hipMalloc(&d_chains, Cuda_sieve::m_max_chains * sizeof(*d_chains)));
            checkCudaErrors(hipMalloc(&d_chain_index, sizeof(*d_chain_index)));
            checkCudaErrors(hipMalloc(&d_prime_candidate_count, sizeof(*d_prime_candidate_count)));


            //copy data to the gpu
            checkCudaErrors(hipMemcpy(d_sieving_primes, primes, prime_count * sizeof(*d_sieving_primes), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_starting_multiples, starting_multiples_host, prime_count * sizeof(*d_starting_multiples), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_small_prime_offsets, small_prime_offsets, Cuda_sieve::m_small_prime_count * sizeof(*d_small_prime_offsets), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_prime_mod_inverses, prime_mod_inverses_host, prime_count * sizeof(*d_prime_mod_inverses), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset(d_chain_index, 0, sizeof(*d_chain_index)));
            checkCudaErrors(hipMemset(d_prime_candidate_count, 0, sizeof(*d_prime_candidate_count)));


        }

        void Cuda_sieve_impl::free_sieve()
        {
            checkCudaErrors(hipFree(d_sieving_primes));
            checkCudaErrors(hipFree(d_starting_multiples));
            //checkCudaErrors(hipFree(d_wheel_indices));
            checkCudaErrors(hipFree(d_multiples));
            checkCudaErrors(hipFree(d_prime_mod_inverses));
            checkCudaErrors(hipFree(d_sieve));
            checkCudaErrors(hipFree(d_chains));
            checkCudaErrors(hipFree(d_chain_index));
        }
    }
}