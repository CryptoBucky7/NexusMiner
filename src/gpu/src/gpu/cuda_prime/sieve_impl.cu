#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sieve_impl.cuh"
#include "sieve.hpp"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

namespace nexusminer {
    namespace gpu {

        __device__ void cuda_chain_push_back(CudaChain& chain, int offset);
        __device__ void cuda_chain_open(CudaChain& chain, uint64_t base_offset);

        __constant__ const int sieve30_offsets[]{ 1,7,11,13,17,19,23,29 };

        __constant__ const int sieve30_gaps[]{ 6,4,2,4,2,4,6,2 };

        __constant__ const int sieve30_index[]
        { -1,0,-1,-1,-1,-1,-1, 1, -1, -1, -1, 2, -1, 3, -1, -1, -1, 4, -1, 5, -1, -1, -1, 6, -1, -1, -1, -1, -1, 7 };  //reverse lookup table (offset mod 30 to index)


        //search the sieve for chains that meet the minimum length requirement.  
        __global__ void find_chain_kernel(uint8_t* sieve, CudaChain* chains, uint32_t* chain_index, uint64_t sieve_start_offset)
        {
            
            uint64_t sieve_size = Cuda_sieve::m_sieve_total_size;
            CudaChain current_chain;
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            int sieve_offset;
            int gap;
            uint64_t chain_start, prime_candidate_offset;
            if (index == 0)
                *chain_index = 0;
            __syncthreads();
            //search each sieve location for a possible chain
            for (uint64_t i = index; i < sieve_size; i += stride)
            {
                if (i < sieve_size)
                {
                    //chain must start with a prime
                    if (sieve[i] == 0)
                    {
                        return;
                    }
                    //search left for another prime less than max gap away
                    gap = 0;
                    int64_t j = i;
                    j--;
                    while (j >= 0 && gap <= maxGap)
                    {
                        gap += sieve30_gaps[j % 8];
                        if (gap <= maxGap && sieve[j] == 1)
                        {
                            //there is a valid element to the left.  this is not the first element in a chain. abort.
                            return;
                        }
                        
                        j--;
                    }
                    //this is the start of a possible chain.  search right
                    //where are we in the wheel
                    sieve_offset = sieve30_offsets[i % 8];
                    chain_start = sieve_start_offset + i / 8 * 30 + sieve_offset;
                    cuda_chain_open(current_chain, chain_start);
                    gap = 0;
                    j = i;
                    j++;
                    while (j < sieve_size && gap <= maxGap)
                    {
                        gap += sieve30_gaps[j % 8];
                        if (gap <= maxGap && sieve[j] == 1)
                        {
                            //another possible candidate.  add it to the chain
                            gap = 0;
                            sieve_offset = sieve30_offsets[j % 8];
                            prime_candidate_offset = sieve_start_offset + j / 8 * 30 + sieve_offset;
                            cuda_chain_push_back(current_chain, prime_candidate_offset - chain_start);
                        }
                        j++;
                    }
                    //we reached the end of the chain.  check if it meets the length requirement
                    if (current_chain.m_offset_count >= Cuda_sieve::m_min_chain_length)
                    {
                        //increment the chain list index
                        uint32_t chain_idx = atomicInc(chain_index, Cuda_sieve::m_max_chains);
                        //copy the current chain to the global list
                        chains[chain_idx] = current_chain;
                    }
                }
            }
            

        }

       
        //return the offset from x to the next integer multiple of n greater than x that is not divisible by 2, 3, or 5.  
        //x must be a multiple of the primorial 30 and n must be a prime greater than 5.
        template <typename T1, typename T2>
        __device__ T2 get_offset_to_next_multiple(T1 x, T2 n)
        {
            T2 m = n - static_cast<T2>(x % n);
            if (m % 2 == 0)
            {
                m += n;
            }
            while (m % 3 == 0 || m % 5 == 0)
            {
                m += 2 * n;
            }
            return m;
        }

        

        //seive kernel

        __global__ void do_sieve(uint64_t sieve_start_offset, uint32_t* sieving_primes, uint32_t sieving_prime_count,
            uint32_t* starting_multiples, uint32_t* prime_mod_inverses, uint8_t* sieve_results, uint32_t* multiples,
            uint8_t* wheel_indices)
        {
            uint32_t segment_size = Cuda_sieve::m_kernel_sieve_size / 8 * 30;

            //local shared copy of the sieve
            __shared__ uint8_t sieve[Cuda_sieve::m_kernel_sieve_size];

            uint64_t block_id = blockIdx.x;
            uint64_t index = threadIdx.x;
            uint64_t stride = blockDim.x;
            //uint64_t num_blocks = gridDim.x;
           
            uint64_t segments = Cuda_sieve::m_kernel_segments_per_block;
            uint64_t sieve_results_index = block_id * Cuda_sieve::m_kernel_sieve_size_per_block;
            //uint64_t primes_per_block = (sieving_prime_count + stride - 1) / stride;
            
            //each block sieves a different region
            uint64_t start_offset = sieve_start_offset + block_id * Cuda_sieve::m_kernel_sieve_size_per_block / 8 * 30;
            
            int wheel_index;
            int next_wheel_gap;
            uint64_t j;
            uint64_t k;
            for (int s = 0; s < segments; s++)
            {
                //everyone in the block initialize part of the shared sieve
                for (int j1 = index; j1 < Cuda_sieve::m_kernel_sieve_size; j1 += stride)
                {
                    if (j1 < Cuda_sieve::m_kernel_sieve_size)
                        sieve[j1] = 1;
                }

                __syncthreads();
                for (uint32_t i = index; i < sieving_prime_count; i += stride)
                {
                    if (i < sieving_prime_count)
                    {
                        
                        k = sieving_primes[i];
                        //get aligned to this region
                        if (s == 0)
                        {
                            j = starting_multiples[i];
                            //the first time through we need to calculate the starting offsets
                            if (start_offset >= j)
                                j = get_offset_to_next_multiple(start_offset - j, sieving_primes[i]);
                            else
                                j -= start_offset;
                            
                            //where are we in the wheel
                            wheel_index = sieve30_index[(prime_mod_inverses[i] * j) % 30];
                        }
                        else
                        {
                            j = multiples[block_id* sieving_prime_count +i];
                            wheel_index = wheel_indices[block_id * sieving_prime_count + i];
                        }
                        next_wheel_gap = sieve30_gaps[wheel_index];
                        
                        while (j < segment_size)
                        {
                            //cross off a multiple of the sieving prime
                            uint64_t sieve_index = (j / 30) * 8 + sieve30_index[j % 30];
                            sieve[sieve_index] = 0;
                            //increment the next multiple of the current prime (rotate the wheel).
                            j += k * next_wheel_gap;
                            wheel_index = (wheel_index + 1) % 8;
                            next_wheel_gap = sieve30_gaps[wheel_index];
                        }
                        //save the starting multiple and wheel index for the next segment
                        multiples[block_id * sieving_prime_count + i] = j - segment_size;
                        wheel_indices[block_id * sieving_prime_count + i] = wheel_index;
                    }
                }
                __syncthreads();
                //copy the sieve results to global memory
                
                for (uint32_t j2 = index; j2 < Cuda_sieve::m_kernel_sieve_size; j2 += stride)
                {
                    if (j2 < Cuda_sieve::m_kernel_sieve_size)
                    {
                        sieve_results[sieve_results_index + j2] = sieve[j2];
                    }
                }
                
                sieve_results_index += Cuda_sieve::m_kernel_sieve_size;
                __syncthreads();
                
            }
        }


        void Cuda_sieve_impl::run_sieve(uint64_t sieve_start_offset, uint8_t sieve[])
        {
            m_sieve_start_offset = sieve_start_offset;
            
            //run the kernel
            do_sieve <<<Cuda_sieve::m_num_blocks, Cuda_sieve::m_threads_per_block >>> (sieve_start_offset, d_sieving_primes, m_sieving_prime_count,
                d_starting_multiples, d_prime_mod_inverses, d_sieve, d_multiples, d_wheel_indices);

            checkCudaErrors(hipDeviceSynchronize());
            //checkCudaErrors(hipMemcpy(sieve, d_sieve, m_sieve_total_size * sizeof(uint8_t), hipMemcpyDeviceToHost));
        }

        void Cuda_sieve_impl::find_chains(CudaChain chains[], uint32_t& chain_count)
        {
            int sieve_threads = 256;
            int sieve_blocks = (Cuda_sieve::m_sieve_total_size + sieve_threads - 1)/ sieve_threads;
            //reset the chain index
            //checkCudaErrors(hipMemset(d_chain_index, 0, sizeof(uint32_t)));
            //run the kernel
            find_chain_kernel << <sieve_blocks, sieve_threads >> > (d_sieve, d_chains, d_chain_index, m_sieve_start_offset);

            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_index, sizeof(uint32_t), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(chains, d_chains, chain_count * sizeof(CudaChain), hipMemcpyDeviceToHost));
        }

        //allocate global memory and load values used by the sieve to the gpu 
        void Cuda_sieve_impl::load_sieve(uint32_t primes[], uint32_t prime_count, uint32_t starting_multiples_host[],
            uint32_t prime_mod_inverses_host[], uint32_t sieve_size, uint16_t device)
        {
          
            m_sieving_prime_count = prime_count;
            checkCudaErrors(hipSetDevice(device));
            //allocate memory on the gpu
            checkCudaErrors(hipMalloc(&d_sieving_primes, prime_count * sizeof(uint32_t)));
            checkCudaErrors(hipMalloc(&d_starting_multiples, prime_count * sizeof(uint32_t)));
            checkCudaErrors(hipMalloc(&d_prime_mod_inverses, prime_count * sizeof(uint32_t)));
            checkCudaErrors(hipMalloc(&d_sieve, sieve_size * sizeof(uint8_t)));
            checkCudaErrors(hipMalloc(&d_multiples, prime_count * Cuda_sieve::m_num_blocks * sizeof(uint32_t)));
            checkCudaErrors(hipMalloc(&d_wheel_indices, prime_count * Cuda_sieve::m_num_blocks * sizeof(uint8_t)));
            checkCudaErrors(hipMalloc(&d_chains, Cuda_sieve::m_max_chains * sizeof(CudaChain)));
            checkCudaErrors(hipMalloc(&d_chain_index, sizeof(uint32_t)));


            //copy data to the gpu
            checkCudaErrors(hipMemcpy(d_sieving_primes, primes, prime_count * sizeof(uint32_t), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_starting_multiples, starting_multiples_host, prime_count * sizeof(uint32_t), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_prime_mod_inverses, prime_mod_inverses_host, prime_count * sizeof(uint32_t), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset(d_chain_index, 0, sizeof(uint32_t)));

        }

        void Cuda_sieve_impl::free_sieve()
        {
            checkCudaErrors(hipFree(d_sieving_primes));
            checkCudaErrors(hipFree(d_starting_multiples));
            checkCudaErrors(hipFree(d_wheel_indices));
            checkCudaErrors(hipFree(d_multiples));
            checkCudaErrors(hipFree(d_prime_mod_inverses));
            checkCudaErrors(hipFree(d_sieve));
            checkCudaErrors(hipFree(d_chains));
            checkCudaErrors(hipFree(d_chain_index));
        }
    }
}