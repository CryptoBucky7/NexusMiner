#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sieve_impl.cuh"
#include "sieve.hpp"
#include "find_chain.cuh"
#include "sieve_small_prime_constants.cuh"
#include "sieve_lookup_tables.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>


#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

namespace nexusminer {
    namespace gpu {

        int round_up(int num, int factor)
        {
            return num + factor - 1 - (num + factor - 1) % factor;
        }
        
        // cross off small primes.  These primes hit the sieve often.  Primes up to 59 can hit a single sieve word two or more times.  
        // We iterate through the sieve words and cross them off using 
        // precalculated constants.  start is the offset from the sieve start 
        __global__ void sieveSmallPrimes(Cuda_sieve::sieve_word_t* sieve, uint64_t start, uint16_t* small_prime_offsets, uint32_t* masks, 
            uint8_t* small_primes, Cuda_sieve::Cuda_sieve_properties sieve_properties)
        {

            uint32_t num_blocks = gridDim.x;
            uint32_t num_threads = blockDim.x;
            uint32_t block_id = blockIdx.x;
            uint32_t index = block_id * num_threads + threadIdx.x;
            uint32_t stride = num_blocks * num_threads;
            __shared__ uint16_t offsets[Cuda_sieve::m_small_prime_count];

            const uint32_t increment = Cuda_sieve::m_sieve_word_range;
            //this loop is faster than a big switch case block with the primes hardcoded.  
            for (uint32_t i = threadIdx.x; i < Cuda_sieve::m_small_prime_count; i += num_threads)
            {
                uint8_t start_offset = start % small_primes[i];
                offsets[i] = small_prime_offsets[i] + start_offset;
            }

            __syncthreads();
            uint32_t inc = increment * stride;
            
            //initialize the table indices
            //we mod intermediate values to avoid needing a 64 bit multiplication which is slow 
            //there is some risk of overflow if index gets too big.  max index is dependent on the sieve size.
            uint8_t index7 = (offsets[0] + index * (increment % 7)) % 7;  // 120 % 7 == 1.  
            uint8_t index11 = (offsets[1] + index * (increment % 11)) % 11;
            uint8_t index13 = (offsets[2] + index * (increment % 13)) % 13;  //120 % 13 == 3.  
            uint8_t index17 = (offsets[3] + index * (increment % 17)) % 17;  //120 % 17 == 1
            uint8_t index19 = (offsets[4] + index * (increment % 19)) % 19;
            uint8_t index23 = (offsets[5] + index * (increment % 23)) % 23;
            uint8_t index29 = (offsets[6] + index * (increment % 29)) % 29;
            uint8_t index31 = (offsets[7] + index * (increment % 31)) % 31;
            uint8_t index37 = (offsets[8] + index * (increment % 37)) % 37;
            uint8_t index41 = (offsets[9] + index * (increment % 41)) % 41;
            uint8_t index43 = (offsets[10] + index * (increment % 43)) % 43;
            uint8_t index47 = (offsets[11] + index * (increment % 47)) % 47;
            uint8_t index53 = (offsets[12] + index * (increment % 53)) % 53;
            uint8_t index59 = (offsets[13] + index * (increment % 59)) % 59;  //120 % 59 == 2. 

            //apply the masks the first time  
            uint32_t word = p7[index7] & p11[index11] & p13[index13] & p17[index17] & p19[index19] & p23[index23] & p29[index29] & p31[index31] &
                p37[index37] & p41[index41] & p43[index43] & p47[index47] & p53[index53] & p59[index59];

            //save the first sieve word to global memory
            sieve[index] = word;

            for (uint32_t i = index+stride; i < sieve_properties.m_sieve_total_size; i += stride)
            {
                //update the lookup table indices
                index7 = (index7 + inc) % 7;
                index11 = (index11 + inc) % 11;
                index13 = (index13 + inc) % 13;
                index17 = (index17 + inc) % 17;
                index19 = (index19 + inc) % 19;
                index23 = (index23 + inc) % 23;
                index29 = (index29 + inc) % 29;
                index31 = (index31 + inc) % 31;
                index37 = (index37 + inc) % 37;
                index41 = (index41 + inc) % 41;
                index43 = (index43 + inc) % 43;
                index47 = (index47 + inc) % 47;
                index53 = (index53 + inc) % 53;
                index59 = (index59 + inc) % 59;
                
                //apply the masks.  
                word = p7[index7] & p11[index11] & p13[index13] & p17[index17] & p19[index19] & p23[index23] & p29[index29] & p31[index31] &
                    p37[index37] & p41[index41] & p43[index43] & p47[index47] & p53[index53] & p59[index59];

                //save the sieve word to global memory
                sieve[i] = word;

            }
        }


        //large primes hit the sieve no more than once per segment.  The large prime kernel works on a shared copy 
        //of the sieve one segment at a time.  The word and bit where the primes hit the segment are stored in the bucket array. 
        //The buckets must be filled prior to calling this kernel. We iterate through the hits in the bucket and cross off composites.  
        __global__ void sieveLargePrimes(uint32_t* large_prime_buckets, uint32_t* bucket_indices, Cuda_sieve::sieve_word_t* sieve_results,
            Cuda_sieve::Cuda_sieve_properties sieve_properties)
        {
            //each kernel block works on one segment of the sieve.  
            unsigned int num_threads = blockDim.x;
            unsigned int block_id = blockIdx.x/Cuda_sieve::m_kernel_segments_per_block;
            unsigned int segment_id = blockIdx.x % Cuda_sieve::m_kernel_segments_per_block;
            unsigned int index = threadIdx.x;
            unsigned int stride = num_threads;

            if (block_id >= Cuda_sieve::m_num_blocks)
                return;

            //local shared copy of one segment of the sieve
            extern __shared__ Cuda_sieve::sieve_word_t sieve[];
            
            uint32_t sieve_results_index = blockIdx.x * sieve_properties.m_kernel_sieve_size_words;
            //each thread in the block initialize part of the shared sieve
            for (int j = index; j < sieve_properties.m_kernel_sieve_size_words; j += stride)
            {
                sieve[j] = sieve_results[sieve_results_index + j];
            }
            
            //the number of sieve hits in this segment
            unsigned int sieve_hits = bucket_indices[block_id * Cuda_sieve::m_kernel_segments_per_block + segment_id];
            uint32_t z = block_id;
            uint32_t y = segment_id;
            uint32_t x;
            const uint32_t ymax = Cuda_sieve::m_kernel_segments_per_block;
            const uint32_t xmax = sieve_properties.m_large_prime_bucket_size;
            __syncthreads();
            //iterate through the sieve hits
            for (unsigned int i = index; i < sieve_hits; i+=stride)
            {
                //unpack the sieve word and bit from the bucket data
                x = i;
                uint32_t bucket_data = large_prime_buckets[z * xmax * ymax + y * xmax + x];
                uint32_t sieve_word = (bucket_data >> 16) & 0x0000FFFF;
                uint32_t sieve_bit = bucket_data & 0x0000FFFF;
                //cross off the bit in the shared sieve
                uint32_t bit_mask = ~(1u << sieve_bit);
                atomicAnd(&sieve[sieve_word], bit_mask);
                //if (block_id == 1)
                //    printf("block %u segment %u bucket index %u word %u bit %u\n", block_id, segment_id, i, sieve_word, sieve_bit);
            }

            __syncthreads();

            //merge the sieve results back to global memory
            for (unsigned int j = index; j < sieve_properties.m_kernel_sieve_size_words; j += stride)
            {
                sieve_results[sieve_results_index + j] = sieve[j];
            }

        }
        

        //medium prime sieve.  We use a block of shared memory to sieve in segments.  Each block sieves a different range. 
        //the final results are merged with the global sieve at the end using atomicAnd. 
        __global__ void medium_sieve(uint64_t sieve_start_offset, uint32_t* sieving_primes, uint32_t sieving_prime_count,
            uint32_t* starting_multiples, Cuda_sieve::sieve_word_t* sieve_results, uint32_t* multiples, Cuda_sieve::Cuda_sieve_properties sieve_properties)
        {

            const uint32_t segment_size = sieve_properties.m_kernel_sieve_size_bytes * Cuda_sieve::m_sieve_byte_range;

            //dymamically allocated shared memory
            extern __shared__ uint32_t shared_mem[];
            uint32_t* sieve = shared_mem;                        
            uint8_t* sieve120_index_shared = (uint8_t*)&sieve[sieve_properties.m_kernel_sieve_size_words]; // starts at the end of sieve
            uint8_t* sieve30_gaps_shared = (uint8_t*)&sieve120_index_shared[120]; 
            unsigned int* prime_index = (unsigned int*)&sieve30_gaps_shared[8];

            //statically allocated shared memory
            //local shared copy of the sieve
            //__shared__ Cuda_sieve::sieve_word_t sieve[Cuda_sieve::m_kernel_sieve_size_words];
            //shared mem lookup tables
            //__shared__ uint8_t sieve120_index_shared[120];
            //__shared__  Cuda_sieve::sieve_word_t unset_bit_mask_shared[32];
            //mod 30 wheel
            //__shared__ uint8_t sieve30_gaps_shared[8];
            //__shared__ unsigned int sieve30_index_shared[30];
            //__shared__ unsigned int prime_mod30_inverse_shared[30];
            //__shared__ unsigned int next_multiple_mod30_offset_shared[30];
            //mod 210 wheel
            //__shared__ uint8_t wheel210_gaps_shared[48];
            //__shared__ uint8_t wheel210_index_shared[210];
            //__shared__ uint8_t prime_mod210_inverse_shared[210];
            //__shared__ uint8_t next_multiple_mod210_offset_shared[210];

            //__shared__ unsigned int prime_index;

            uint32_t block_id = blockIdx.x;
            uint32_t index = threadIdx.x;
            uint32_t stride = blockDim.x;
            uint32_t num_threads = blockDim.x;
            //unsigned int block_id = blockIdx.x / Cuda_sieve::m_kernel_segments_per_block;
            //unsigned int segment_id = blockIdx.x % Cuda_sieve::m_kernel_segments_per_block;

            //initialize shared lookup tables.  lookup tables in shared memory are faster than global memory lookup tables.
            for (int i = index; i < 120; i += stride)
            {
                sieve120_index_shared[i] = sieve120_index[i];
            }
            /*for (int i = index; i < 32; i += stride)
            {
                unset_bit_mask_shared[i] = unset_bit_mask[i];
            }*/
            for (int i = index; i < 8; i += stride)
            {
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            for (int i = index; i < 30; i += stride)
            {
                //sieve30_index_shared[i] = sieve30_index[i];
                //prime_mod30_inverse_shared[i] = prime_mod30_inverse[i];
                //next_multiple_mod30_offset_shared[i] = next_multiple_mod30_offset[i];
            }
            //for (int i = index; i < 48; i += stride)
           //{
                //wheel210_gaps_shared[i] = wheel210_gaps[i];
            //}
            //for (int i = index; i < 210; i += stride)
            //{
                //wheel210_index_shared[i] = wheel210_index[i];
                //prime_mod210_inverse_shared[i] = prime_mod210_inverse[i];
                //next_multiple_mod210_offset_shared[i] = next_multiple_mod210_offset[i];
            //}


            const uint32_t segments = Cuda_sieve::m_kernel_segments_per_block;
            uint32_t sieve_results_index = blockIdx.x * sieve_properties.m_kernel_sieve_size_words * segments;
            //each block sieves a different region
            uint64_t start_offset = sieve_start_offset + static_cast<uint64_t>(blockIdx.x) * sieve_properties.m_segment_range * segments;

            uint8_t wheel_index;
            uint8_t next_wheel_gap;
            uint32_t j;
            uint32_t k;
            uint8_t prime_mod_inv;
            for (int s = 0; s < segments; s++)
            {
                //everyone in the block initialize part of the shared sieve
                for (int sieve_index = index; sieve_index < sieve_properties.m_kernel_sieve_size_words; sieve_index += stride)
                {
                    //sieve[sieve_index] = ~0;
                    sieve[sieve_index] = sieve_results[sieve_results_index + sieve_index];
                }
                if (index == 0)
                {
                    *prime_index = num_threads;
                }
                __syncthreads();
                for (uint32_t i = index; i < sieving_prime_count; i = atomicInc(prime_index, 0xFFFFFFFF))
                {
                    k = sieving_primes[i];

                    //get aligned to this region
                    if (s == 0)
                    {
                        j = starting_multiples[i];
                        //the first time through we need to calculate the starting offsets
                        if (start_offset > 0)
                        {
                            uint64_t x = start_offset - j;
                            //offset to the first integer multiple of the prime above the starting offset
                            uint32_t m = k - (x % k);
                            //find the next integer multiple of the prime that is not divisible by 2,3 or 5
                            m += (m % 2 == 0) ? k : 0;
                            m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                            m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                            j = m;
                            //this does the same thing as above - it gets the next multiple using prime inverse mod 30 and a lookup table. 
                            //prime_mod_inv = prime_mod30_inverse[k % 30];
                            //j = m + k * next_multiple_mod30_offset[((m % 30) * prime_mod_inv) % 30];
                        }
                        //else
                            //j -= start_offset;
                    }
                    else
                    {
                        j = multiples[block_id * sieving_prime_count + i];
                        //calculating the wheel index each time is faster than saving and retrieving it from global memory each loop
                    }
                    prime_mod_inv = prime_mod30_inverse[k % 30];
                    wheel_index = sieve30_index[(prime_mod_inv * j) % 30];
                    next_wheel_gap = sieve30_gaps_shared[wheel_index];

                    while (j < segment_size)
                    {
                        //cross off a multiple of the sieving prime
                        uint32_t sieve_index = j / Cuda_sieve::m_sieve_word_range;

                        Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                            sieve120_index_shared[j % Cuda_sieve::m_sieve_word_range]);

                        atomicAnd(&sieve[sieve_index], bitmask);

                        //increment the next multiple of the current prime (rotate the wheel).
                        j += k * next_wheel_gap;
                        wheel_index = (wheel_index + 1) % 8;
                        next_wheel_gap = sieve30_gaps_shared[wheel_index];

                    }

                    //save the starting multiple for this prime for the next segment
                    multiples[block_id * sieving_prime_count + i] = j - segment_size;
                }
                __syncthreads();


                //merge the sieve results back to global memory
                for (uint32_t sieve_index = index; sieve_index < sieve_properties.m_kernel_sieve_size_words; sieve_index += stride)
                {
                    sieve_results[sieve_results_index + sieve_index] = sieve[sieve_index];
                }

                sieve_results_index += sieve_properties.m_kernel_sieve_size_words;
                start_offset += segment_size;
            }

        }

        //This is a sieve optimized for primes in the neighborhood of 100 - 1000.  Primes in this range hit each sieve segment hundreds of times
        //and hit each sieve word no more than once.  Each prime is processed by one full warp which helps minimize thread divergence.  
        // There are enough hits per segment to keep a full warp busy with a single prime.  The sieve is stored in shared memory. 
        // At the end the results are merged with the global sieve. 
        __global__ void medium_small_sieve(uint64_t sieve_start_offset, uint32_t* sieving_primes, 
            uint32_t* starting_multiples, Cuda_sieve::sieve_word_t* sieve_results, Cuda_sieve::Cuda_sieve_properties sieve_properties)
        {
            const uint32_t segment_size = sieve_properties.m_kernel_sieve_size_bytes * Cuda_sieve::m_sieve_byte_range;

            //dymamically allocated shared memory
            extern __shared__ uint32_t shared_mem[];
            uint32_t* sieve = shared_mem;
            uint8_t* sieve120_index_shared = (uint8_t*)&sieve[sieve_properties.m_kernel_sieve_size_words]; // starts at the end of sieve
            uint8_t* sieve30_gaps_shared = (uint8_t*)&sieve120_index_shared[120];
            uint8_t* sieve30_index_shared = (uint8_t*)&sieve30_gaps_shared[8];
            uint8_t* prime_mod30_inverse_shared = (uint8_t*)&sieve30_index_shared[30];
            //local shared copy of the sieve
            //__shared__ Cuda_sieve::sieve_word_t sieve[Cuda_sieve::m_kernel_sieve_size_words];
            //shared mem lookup tables
            //__shared__ uint8_t sieve120_index_shared[120];
            //__shared__ uint8_t sieve30_gaps_shared[8];
            //__shared__ uint8_t sieve30_index_shared[30];
            //__shared__ uint8_t prime_mod30_inverse_shared[30];

            uint32_t index = threadIdx.x;
            uint32_t stride = blockDim.x;
            unsigned int block_id = blockIdx.x;
            unsigned int warp_id = threadIdx.x / 32;
            unsigned int lane_id = threadIdx.x % 32;

            //initialize shared lookup tables.  lookup tables in shared memory are faster than global memory lookup tables.
            for (int i = index; i < 120; i += stride)
            {
                sieve120_index_shared[i] = sieve120_index[i];
            }
            for (int i = index; i < 8; i += stride)
            {
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            for (int i = index; i < 30; i += stride)
            {
                sieve30_index_shared[i] = sieve30_index[i];
                prime_mod30_inverse_shared[i] = prime_mod30_inverse[i];
            }

            const uint32_t segments = Cuda_sieve::m_kernel_segments_per_block;
            uint32_t sieve_results_index = block_id * sieve_properties.m_kernel_sieve_size_words_per_block;
            uint64_t start_offset = sieve_start_offset +
                static_cast<uint64_t>(block_id) * sieve_properties.m_kernel_sieve_size_words_per_block * Cuda_sieve::m_sieve_word_range;
            uint8_t wheel_index;
            unsigned int next_wheel_gap;
            uint32_t j;
            uint32_t k;
            uint32_t prime_mod_inv;
            for (int s = 0; s < segments; s++)
            {
                //everyone in the block initialize part of the shared sieve
                for (unsigned int sieve_index = index; sieve_index < sieve_properties.m_kernel_sieve_size_words; sieve_index += stride)
                {
                    //sieve[sieve_index] = ~0;
                    sieve[sieve_index] = sieve_results[sieve_results_index + sieve_index];
                }

                __syncthreads();
                for (uint32_t i = warp_id; i < Cuda_sieve::m_medium_small_prime_count; i += stride/32)
                {
                    k = sieving_primes[i];
                    j = starting_multiples[i];

                    //the first time through we need to calculate the starting offsets
                    if (start_offset > 0)
                    {
                        uint64_t x = start_offset - j;
                        //offset to the first integer multiple of the prime above the starting offset
                        uint32_t m = k - (x % k);
                        //find the next integer multiple of the prime that is not divisible by 2,3 or 5
                        m += (m % 2 == 0) ? k : 0;
                        m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                        m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                        j = m;
                        //this does the same thing as above - it gets the next multiple using prime inverse mod 30 and a lookup table. 
                        //prime_mod_inv = prime_mod30_inverse_shared[k % 30];
                        //j = m + k * next_multiple_mod30_offset_shared[((m % 30) * prime_mod_inv) % 30];
                    }
                    //else
                    //    j -= start_offset;
                   
                    prime_mod_inv = prime_mod30_inverse_shared[k % 30];
                    unsigned int full_wheels = lane_id / 8;
                    wheel_index = sieve30_index_shared[(prime_mod_inv * j) % 30];
                    next_wheel_gap = 0;
                    j += full_wheels * 30 * k;
                    for (auto id = 0; id < lane_id % 8; id++)
                    {
                        next_wheel_gap += sieve30_gaps_shared[wheel_index % 8];
                        wheel_index++;
                    }
                    j += k * next_wheel_gap;
                    
                    uint32_t sieve_index = j / Cuda_sieve::m_sieve_word_range;
                    Cuda_sieve::sieve_word_t bitmask = ~(static_cast<Cuda_sieve::sieve_word_t>(1) <<
                        sieve120_index_shared[j % Cuda_sieve::m_sieve_word_range]);
                    //each lane always crosses off the same spot on the wheel (the same bit in the word)
                    while(sieve_index < sieve_properties.m_kernel_sieve_size_words)
                    {
                        //cross off a multiple of the sieving prime
                        atomicAnd(&sieve[sieve_index], bitmask);

                        //Normally this is where we add the next multiple of the current prime (rotate the wheel).  
                        //j += increment; //we don't need to keep track of the multiple of the prime, only the index of the sieve word.
                        //there are 32 lanes working on a prime.  There are also 32 bits in the sieve word and the mod30 wheel is 8 bits.
                        //Each lane works on multiples of one bit in the wheel.   
                        sieve_index += k; //this looks wierd but it works because each lane works on a multiple of 120 * prime.  
                       
                    }
                   
                }
                __syncthreads();


                //merge the sieve results back to global memory
                for (uint32_t sieve_index = index; sieve_index < sieve_properties.m_kernel_sieve_size_words; sieve_index += stride)
                {
                    sieve_results[sieve_results_index + sieve_index] = sieve[sieve_index];
                }

                sieve_results_index += sieve_properties.m_kernel_sieve_size_words;
                start_offset += segment_size;
            }

        }

        //count the prime candidates in the global sieve
        __global__ void count_prime_candidates(Cuda_sieve::sieve_word_t* sieve, unsigned long long* prime_candidate_count, Cuda_sieve::Cuda_sieve_properties sieve_properties)
        {
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            
            uint64_t count = 0;
            if (index == 0)
                *prime_candidate_count = 0;
            __syncthreads();

            for (uint64_t i = index; i < sieve_properties.m_sieve_total_size; i += stride)
            {
                count += __popcll(sieve[i]);
            }
            atomicAdd(prime_candidate_count, count);

        }

        

        //sort large primes into buckets by where they hit the sieve
        __global__ void sort_large_primes(uint64_t sieve_start_offset, uint32_t* large_primes, uint32_t sieving_prime_count,
            uint32_t* starting_multiples, uint32_t* large_prime_buckets, uint32_t* bucket_indices, Cuda_sieve::Cuda_sieve_properties sieve_properties)
        {
            int num_threads = blockDim.x;
            int block_id = blockIdx.x;
            int index = threadIdx.x;
            int stride = num_threads;
            
            const uint32_t segment_size = sieve_properties.m_kernel_sieve_size_bytes * Cuda_sieve::m_sieve_byte_range;
            const uint32_t segments = Cuda_sieve::m_kernel_segments_per_block * Cuda_sieve::m_num_blocks / gridDim.x;
            const uint32_t block_range = segments * segment_size;

            //each block sieves a different region
            uint64_t start_offset = sieve_start_offset + static_cast<uint64_t>(block_id) * block_range;

            //shared mem lookup tables
            __shared__ uint8_t sieve30_gaps_shared[8];
            __shared__ uint8_t sieve30_index_shared[30];
            __shared__ uint8_t prime_mod30_inverse_shared[30];
            __shared__ uint32_t sieve120_index_shared[120];
            __shared__ unsigned int prime_index;

            //shared copy of bucket index array
            //this local array could be smaller than the global array
            __shared__ uint32_t bucket_indices_shared[Cuda_sieve::m_kernel_segments_per_block * Cuda_sieve::m_num_blocks];
            uint32_t bucket_index = 0;
            __shared__ uint32_t max_bucket_index;

            //initialize shared lookup tables.  lookup tables in shared memory are faster than global memory lookup tables.
            for (int i = index; i < 8; i += stride)
            {
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            for (int i = index; i < 30; i += stride)
            {
                sieve30_index_shared[i] = sieve30_index[i];
                prime_mod30_inverse_shared[i] = prime_mod30_inverse[i];
            }
            for (int i = index; i < 120; i += stride)
            {
                sieve120_index_shared[i] = sieve120_index[i];
            }

            //reset the bucket indices
            for (int i = index; i < segments; i += stride)
            {
                bucket_indices_shared[block_id* segments + i] = 0;
            }
            if (index == 0)
            {
                prime_index = num_threads;
                max_bucket_index = 0;
            }
            __syncthreads();
            //iterate through the list of primes
            for (uint32_t i = index; i < sieving_prime_count; i = atomicInc(&prime_index, 0xFFFFFFFF))
            //for (uint32_t i = index; i < Cuda_sieve::m_large_prime_count; i += stride)
            {
                uint32_t k = large_primes[i];
                uint32_t j = starting_multiples[i];

                //calculate the starting offsets for this block
                if (start_offset >= j)
                {
                    uint64_t x = start_offset - j;
                    //offset to the first integer multiple of the prime above the starting offset
                    uint32_t m = k - (x % k);
                    //find the next integer multiple of the prime that is not divisible by 2,3 or 5
                    m += (m % 2 == 0) ? k : 0;
                    m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                    m += (m % 3 == 0 || m % 5 == 0) ? 2 * k : 0;
                    j = m;
                }
                else
                    j -= start_offset;
                
                uint8_t prime_mod_inv = prime_mod30_inverse_shared[k % 30];
                uint8_t wheel_index = sieve30_index_shared[(prime_mod_inv * j) % 30];
                uint8_t next_wheel_gap = sieve30_gaps_shared[wheel_index];
                uint32_t next_segment = j / segment_size;
                uint32_t segment_offset = j % segment_size;
               // uint32_t loop_count = 0;
                while (next_segment < segments)
                {
                    //which word within the segment does the prime hit
                    uint32_t sieve_word = segment_offset / Cuda_sieve::m_sieve_word_range;
                    //which bit within the word does the prime hit
                    uint32_t sieve_bit = sieve120_index_shared[segment_offset % Cuda_sieve::m_sieve_word_range];
                    //pack the word index and bit into one 32 bit word
                    uint32_t sieve_segment_hit = (sieve_word << 16) | sieve_bit;
                    //add the sieve hit to the segment's bucket
                    bucket_index = atomicInc(&bucket_indices_shared[block_id * segments + next_segment], 0xFFFFFFFF);
                    //we are indexing a 1D array as if it were a 3D array. 
                    uint32_t z = block_id;
                    uint32_t y = next_segment;
                    uint32_t x = bucket_index;
                    const uint32_t ymax = segments;
                    const uint32_t xmax = sieve_properties.m_large_prime_bucket_size;
                    large_prime_buckets[z*xmax*ymax + y*xmax + x] = sieve_segment_hit;
                   
                    //increment the next multiple of the current prime (rotate the wheel).
                    j += k * next_wheel_gap;
                    wheel_index = (wheel_index + 1) % 8;
                    next_wheel_gap = sieve30_gaps_shared[wheel_index];
                    next_segment = j / segment_size;
                    segment_offset = j % segment_size;
                }
               
            }
            __syncthreads();
            //copy bucket indices to global memory
            for (int i = index; i < segments; i += stride)
            {
                bucket_indices[block_id * segments + i] = bucket_indices_shared[block_id * segments + i];
                //max_bucket_index = max(max_bucket_index, bucket_indices_shared[block_id * segments + i]);
            }
            //for debugging max memory usage of the buckets. large sieves can overflow the buckets.
            //__syncthreads();
            //if (threadIdx.x == 0)
            //        printf("max bucket index %u\n", max_bucket_index);

        }

       
        void Cuda_sieve_impl::run_large_prime_sieve(uint64_t sieve_start_offset)
        {

            int threads = 1024;
            //one kernel block per sieve block
            int blocks = Cuda_sieve::m_num_blocks ;

            int split_denominator = 4;
            //with larger sieves we can run out of memory.  for larger sieves on cards with larger shared memory
            //we split the large primes differently to reduce max memory usage by the buckets 
            if (m_sieve_properties.m_shared_mem_size_kbytes > 64)
                split_denominator = 8;

            int split_numerator = split_denominator - 1;
            //warning this can use a lot of vram. it does not check for overflow of buckets between blocks. 
            sort_large_primes << <blocks, threads >> > (sieve_start_offset, d_large_primes, Cuda_sieve::m_large_prime_count/ split_denominator,
                d_large_prime_starting_multiples, d_large_prime_buckets, d_bucket_indices, m_sieve_properties);

            //one kernel block per sieve segment
            blocks = Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block;
            threads = 1024;
            hipFuncSetAttribute(reinterpret_cast<const void*>(sieveLargePrimes), hipFuncAttributeMaxDynamicSharedMemorySize, m_sieve_properties.m_shared_mem_size_bytes);
            sieveLargePrimes << <blocks, threads, m_sieve_properties.m_shared_mem_size_bytes >> > (d_large_prime_buckets,
                d_bucket_indices, d_sieve, m_sieve_properties);

            blocks = Cuda_sieve::m_num_blocks / 2;

            sort_large_primes << <blocks, threads >> > (sieve_start_offset, d_large_primes+ Cuda_sieve::m_large_prime_count / split_denominator,
                split_numerator *Cuda_sieve::m_large_prime_count/ split_denominator,
                d_large_prime_starting_multiples + Cuda_sieve::m_large_prime_count / split_denominator, d_large_prime_buckets, d_bucket_indices,
                m_sieve_properties);

            //one kernel block per sieve segment
            blocks = Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block;
            threads = 1024;
            sieveLargePrimes << <blocks, threads, m_sieve_properties.m_shared_mem_size_bytes >> > (d_large_prime_buckets, 
                d_bucket_indices, d_sieve, m_sieve_properties);

        }

        void Cuda_sieve_impl::run_small_prime_sieve(uint64_t sieve_start_offset)
        {
            const int threads = 256;
            const int loops_per_block = 32;
            const int blocks = (m_sieve_properties.m_sieve_total_size/loops_per_block + threads - 1)/threads;
            
            sieveSmallPrimes << <blocks, threads >> > (d_sieve, sieve_start_offset, d_small_prime_offsets, d_small_prime_masks,
                d_small_primes, m_sieve_properties);

        }

        //medium sieve
        void Cuda_sieve_impl::run_sieve(uint64_t sieve_start_offset)
        {
            int blocks = Cuda_sieve::m_num_blocks;// * Cuda_sieve::m_kernel_segments_per_block;
            int threads = 1024;
            m_sieve_start_offset = sieve_start_offset;
            
            hipFuncSetAttribute(reinterpret_cast<const void*>(medium_sieve), hipFuncAttributeMaxDynamicSharedMemorySize, m_sieve_properties.m_shared_mem_size_bytes);

            medium_sieve << <blocks, threads, m_sieve_properties.m_shared_mem_size_bytes >> > (sieve_start_offset, d_sieving_primes, m_sieving_prime_count,
                d_starting_multiples, d_sieve, d_multiples, m_sieve_properties);

        }

        void Cuda_sieve_impl::run_medium_small_prime_sieve(uint64_t sieve_start_offset)
        {

           hipFuncSetAttribute(reinterpret_cast<const void*>(medium_small_sieve), hipFuncAttributeMaxDynamicSharedMemorySize, m_sieve_properties.m_shared_mem_size_bytes);
           medium_small_sieve << <Cuda_sieve::m_num_blocks, Cuda_sieve::m_threads_per_block, m_sieve_properties.m_shared_mem_size_bytes >> >
               (sieve_start_offset, d_medium_small_primes, d_medium_small_prime_starting_multiples, d_sieve, m_sieve_properties);

        }

        void Cuda_sieve_impl::get_sieve(Cuda_sieve::sieve_word_t sieve[])
        {
            checkCudaErrors(hipMemcpy(sieve, d_sieve, m_sieve_properties.m_sieve_total_size * sizeof(*d_sieve), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::get_prime_candidate_count(uint64_t& prime_candidate_count)
        {
            const int threads = 256;
            const int blocks = 1; // (Cuda_sieve::m_sieve_total_size + threads - 1) / threads;
            count_prime_candidates << <blocks, threads >> > (d_sieve, d_prime_candidate_count, m_sieve_properties);
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipMemcpy(&prime_candidate_count, d_prime_candidate_count, sizeof(*d_prime_candidate_count), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::find_chains()
        {
            //const int sieve_threads = 64;
            //const int checks_per_block = 64;
            //const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            //const uint64_t sieve_total_bits = Cuda_sieve::m_sieve_total_size * sieve_bits_per_word;
            //const int sieve_blocks = (sieve_total_bits /checks_per_block + sieve_threads - 1)/ sieve_threads;
            //find_chain_kernel << <sieve_blocks, sieve_threads >> > (d_sieve, d_chains, d_last_chain_index, m_sieve_start_offset, d_chain_stat_count);

            const int blocks = Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block;
            const int search_regions_per_thread = 1;
            const unsigned int search_range = Cuda_sieve::m_sieve_chain_search_boundary * Cuda_sieve::m_sieve_word_byte_count;
            const unsigned int search_regions_per_segment = (m_sieve_properties.m_segment_range + search_range - 1) / search_range;
            const unsigned int threads = round_up((search_regions_per_segment + search_regions_per_thread - 1) / search_regions_per_thread,32);
            find_chain_kernel2 << <blocks, threads >> > (d_sieve, d_chains, d_last_chain_index, m_sieve_start_offset, d_chain_stat_count, m_sieve_properties);
            
        }

        void Cuda_sieve_impl::get_chains(CudaChain chains[], uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(chains, d_chains, chain_count * sizeof(*d_chains), hipMemcpyDeviceToHost));
        }

        void Cuda_sieve_impl::get_chain_count(uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToHost));
        }

        //get a pointer to the chain array.  fermat test uses the chain array as input. 
        void Cuda_sieve_impl::get_chain_pointer(CudaChain*& chains_ptr, uint32_t*& chain_count_ptr)
        {
            chains_ptr = d_chains;
            chain_count_ptr = d_last_chain_index;
        }

        //check the list of chains for winners.  save winners and remove losers
        void Cuda_sieve_impl::clean_chains()
        {
            const int threads = 256;
            uint32_t chain_count;
            get_chain_count(chain_count);
            int blocks = (chain_count + threads - 1) / threads;
            //copy surviving chains to a temporary location. 
            filter_busted_chains << <blocks, threads >> > (d_chains, d_last_chain_index, d_good_chains, d_good_chain_index,
                d_long_chains, d_last_long_chain_index, d_chain_histogram);
            uint32_t good_chain_count;
            //get the count of good chains from device memory
            checkCudaErrors(hipMemcpy(&good_chain_count, d_good_chain_index, sizeof(*d_good_chain_index), hipMemcpyDeviceToHost));
            //copy the temporary good chain list back to the chain list
            checkCudaErrors(hipMemcpyAsync(d_chains, d_good_chains, good_chain_count*sizeof(*d_chains), hipMemcpyDeviceToDevice));
            //update the chain count
            checkCudaErrors(hipMemcpy(d_last_chain_index, d_good_chain_index, sizeof(*d_last_chain_index), hipMemcpyDeviceToDevice));

        }

        void Cuda_sieve_impl::get_long_chains(CudaChain chains[], uint32_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(&chain_count, d_last_long_chain_index, sizeof(*d_last_long_chain_index), hipMemcpyDeviceToHost));
            if (chain_count > 0)
            {
                checkCudaErrors(hipMemcpy(chains, d_long_chains, chain_count * sizeof(*d_long_chains), hipMemcpyDeviceToHost));
                //clear the long chain list
                checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
            }
        }

        //read the histogram
        void Cuda_sieve_impl::get_stats(uint32_t chain_histogram[], uint64_t& chain_count)
        {
            checkCudaErrors(hipMemcpy(chain_histogram, d_chain_histogram, (Cuda_sieve::chain_histogram_max+1) * sizeof(*d_chain_histogram), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_stat_count, sizeof(*d_chain_stat_count), hipMemcpyDeviceToHost));

        }

        void Cuda_sieve_impl::synchronize()
        {
            checkCudaErrors(hipDeviceSynchronize());
        }


        //The size of the sieve is determined by the maximum amount of shared memory available to a kernel.
        //Many other sieve constants are set based on the size of the sieve.
        //Here we read the amount of shared memory available and set the size of the sieve.  Do this once when the miner starts. 
        void Cuda_sieve_impl::init_sieve_size(int device, Cuda_sieve::Cuda_sieve_properties& sieve_properties)
        {
            //get max shared memory available to each thread block
            int shared_memory_size;
            hipDeviceGetAttribute(&shared_memory_size, hipDeviceAttributeSharedMemPerBlockOptin, device);
            //printf("Max shared mem size %i\n", shared_memory_size);

            //get total gpu ram
            size_t free_mem, total_mem;
            hipSetDevice(device);
            hipMemGetInfo(&free_mem, &total_mem);
            //printf("Total gpu memory %zu\n", total_mem);
            if (total_mem < 8.0e9)
                sieve_properties.m_bucket_ram_budget = 4.5e9;  //bytes avaialble for storing bucket data
            else
                sieve_properties.m_bucket_ram_budget = 6.0e9;

            sieve_properties.m_large_prime_bucket_size = sieve_properties.m_bucket_ram_budget / (Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block) / 4;
            //shared_memory_size = 48 * 1024;
            sieve_properties.m_shared_mem_size_kbytes = shared_memory_size / 1024;
            sieve_properties.m_shared_mem_size_bytes = sieve_properties.m_shared_mem_size_kbytes * 1024;
            //The span of the primorial 30030 is represented by 30030/30 = 1001 bytes which conveniently is just below 1KB
            //We size the sieve segment to fill the block shared memory.  N.B. we have to keep a few hundred bytes of shared mem free for lookup tables. 
            //this is the size of the sieve segment in bytes. It should be a multiple of 4 for a 32 bit word sieve.
            sieve_properties.m_kernel_sieve_size_bytes = 1001 * (sieve_properties.m_shared_mem_size_kbytes / 4) * 4;  
            sieve_properties.m_kernel_sieve_size_words = sieve_properties.m_kernel_sieve_size_bytes / Cuda_sieve::m_sieve_word_byte_count;
            sieve_properties.m_segment_range = sieve_properties.m_kernel_sieve_size_words * Cuda_sieve::m_sieve_word_range;
            sieve_properties.m_kernel_sieve_size_words_per_block = sieve_properties.m_kernel_sieve_size_words * Cuda_sieve::m_kernel_segments_per_block;
            sieve_properties.m_block_range = sieve_properties.m_segment_range * Cuda_sieve::m_kernel_segments_per_block;
            sieve_properties.m_sieve_total_size = sieve_properties.m_kernel_sieve_size_words_per_block * Cuda_sieve::m_num_blocks; //size of the sieve in words
            sieve_properties.m_sieve_range = sieve_properties.m_sieve_total_size * Cuda_sieve::m_sieve_word_range;
            //keep a local cache of sieve properties
            m_sieve_properties = sieve_properties;
        }

        //allocate global memory and load values used by the sieve to the gpu 
        void Cuda_sieve_impl::load_sieve(uint32_t primes[], uint32_t prime_count, uint32_t large_primes[], uint32_t medium_small_primes[], 
            uint32_t small_prime_masks[], uint32_t small_prime_mask_count, uint8_t small_primes[], uint16_t device)
        {
          
            m_sieving_prime_count = prime_count;
            m_device = device;
            checkCudaErrors(hipSetDevice(device));

            //allocate memory on the gpu
            checkCudaErrors(hipMalloc(&d_sieving_primes, prime_count * sizeof(*d_sieving_primes)));
            checkCudaErrors(hipMalloc(&d_starting_multiples, prime_count * sizeof(*d_starting_multiples)));
            //checkCudaErrors(hipMalloc(&d_medium_primes, prime_count * sizeof(*d_medium_primes)));

            checkCudaErrors(hipMalloc(&d_small_prime_offsets, Cuda_sieve::m_small_prime_count * sizeof(*d_small_prime_offsets)));
            checkCudaErrors(hipMalloc(&d_medium_small_primes, Cuda_sieve::m_medium_small_prime_count * sizeof(*d_medium_small_primes)));
            checkCudaErrors(hipMalloc(&d_medium_small_prime_starting_multiples, 
                Cuda_sieve::m_medium_small_prime_count * sizeof(*d_medium_small_prime_starting_multiples)));
            checkCudaErrors(hipMalloc(&d_small_prime_masks, small_prime_mask_count * sizeof(*d_small_prime_masks)));
            checkCudaErrors(hipMalloc(&d_small_primes, Cuda_sieve::m_small_prime_count * sizeof(*d_small_primes)));
            checkCudaErrors(hipMalloc(&d_large_primes, Cuda_sieve::m_large_prime_count * sizeof(*d_large_primes)));
            checkCudaErrors(hipMalloc(&d_large_prime_starting_multiples, Cuda_sieve::m_large_prime_count * sizeof(*d_large_prime_starting_multiples)));
            checkCudaErrors(hipMalloc(&d_large_prime_buckets, Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block
                * m_sieve_properties.m_large_prime_bucket_size * sizeof(*d_large_prime_buckets)));
            checkCudaErrors(hipMalloc(&d_bucket_indices, Cuda_sieve::m_num_blocks * Cuda_sieve::m_kernel_segments_per_block * sizeof(*d_bucket_indices)));
            checkCudaErrors(hipMalloc(&d_sieve, m_sieve_properties.m_sieve_total_size * sizeof(*d_sieve)));
            checkCudaErrors(hipMalloc(&d_multiples, prime_count * Cuda_sieve::m_num_blocks * sizeof(*d_multiples)));
            checkCudaErrors(hipMalloc(&d_chains, Cuda_sieve::m_max_chains * sizeof(*d_chains)));
            checkCudaErrors(hipMalloc(&d_long_chains, Cuda_sieve::m_max_long_chains * sizeof(*d_long_chains)));
            checkCudaErrors(hipMalloc(&d_last_chain_index, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMalloc(&d_last_long_chain_index, sizeof(*d_last_long_chain_index)));
            checkCudaErrors(hipMalloc(&d_prime_candidate_count, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMalloc(&d_good_chain_index, sizeof(*d_good_chain_index)));
            checkCudaErrors(hipMalloc(&d_good_chains, Cuda_sieve::m_max_chains/2 * sizeof(*d_good_chains)));
            checkCudaErrors(hipMalloc(&d_chain_histogram, (Cuda_sieve::chain_histogram_max + 1) * sizeof(*d_chain_histogram)));
            checkCudaErrors(hipMalloc(&d_chain_stat_count, sizeof(*d_chain_stat_count)));

            //copy data to the gpu
            checkCudaErrors(hipMemcpy(d_small_primes, small_primes, Cuda_sieve::m_small_prime_count * sizeof(*d_small_primes), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_small_prime_masks, small_prime_masks, small_prime_mask_count * sizeof(*d_small_prime_masks), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_sieving_primes, primes, prime_count * sizeof(*d_sieving_primes), hipMemcpyHostToDevice));

            checkCudaErrors(hipMemcpy(d_large_primes, large_primes, Cuda_sieve::m_large_prime_count * sizeof(*d_large_primes), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_medium_small_primes, medium_small_primes,
                Cuda_sieve::m_medium_small_prime_count * sizeof(*d_medium_small_primes), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset(d_last_chain_index, 0, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMemset(d_prime_candidate_count, 0, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
            checkCudaErrors(hipMemset(d_chain_stat_count, 0, sizeof(*d_chain_stat_count)));
            reset_stats();

        }

        //reset sieve with new starting offsets
        void Cuda_sieve_impl::init_sieve(uint32_t starting_multiples[], uint16_t small_prime_offsets[], uint32_t large_prime_multiples[],
            uint32_t medium_small_prime_multiples[])
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipMemcpy(d_starting_multiples, starting_multiples, m_sieving_prime_count * sizeof(*d_starting_multiples), hipMemcpyHostToDevice));
            //checkCudaErrors(hipMemcpy(d_medium_primes, starting_multiples, m_sieving_prime_count * sizeof(*d_medium_primes), hipMemcpyHostToDevice));

            checkCudaErrors(hipMemcpy(d_large_prime_starting_multiples, large_prime_multiples, Cuda_sieve::m_large_prime_count * sizeof(*d_large_prime_starting_multiples), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_small_prime_offsets, small_prime_offsets, Cuda_sieve::m_small_prime_count * sizeof(*d_small_prime_offsets), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset(d_last_chain_index, 0, sizeof(*d_last_chain_index)));
            checkCudaErrors(hipMemset(d_prime_candidate_count, 0, sizeof(*d_prime_candidate_count)));
            checkCudaErrors(hipMemset(d_last_long_chain_index, 0, sizeof(*d_last_long_chain_index)));
            checkCudaErrors(hipMemcpy(d_medium_small_prime_starting_multiples, medium_small_prime_multiples,
                Cuda_sieve::m_medium_small_prime_count * sizeof(*d_medium_small_prime_starting_multiples), hipMemcpyHostToDevice));
        }

        void Cuda_sieve_impl::reset_stats()
        {
            checkCudaErrors(hipMemset(d_chain_histogram, 0, (Cuda_sieve::chain_histogram_max + 1) * sizeof(*d_chain_histogram)));
            checkCudaErrors(hipMemset(d_chain_stat_count, 0, sizeof(*d_chain_stat_count)));

        }

        void Cuda_sieve_impl::free_sieve()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_sieving_primes));
            checkCudaErrors(hipFree(d_large_primes));
            checkCudaErrors(hipFree(d_starting_multiples));
            //checkCudaErrors(hipFree(d_medium_primes));
            checkCudaErrors(hipFree(d_multiples));
            checkCudaErrors(hipFree(d_sieve));
            checkCudaErrors(hipFree(d_chains));
            checkCudaErrors(hipFree(d_last_chain_index));
            checkCudaErrors(hipFree(d_long_chains));
            checkCudaErrors(hipFree(d_last_long_chain_index));
            checkCudaErrors(hipFree(d_good_chains));
            checkCudaErrors(hipFree(d_good_chain_index));
            checkCudaErrors(hipFree(d_chain_histogram));
            checkCudaErrors(hipFree(d_large_prime_buckets));
            checkCudaErrors(hipFree(d_bucket_indices));
            checkCudaErrors(hipFree(d_medium_small_primes));
            checkCudaErrors(hipFree(d_medium_small_prime_starting_multiples));
            checkCudaErrors(hipFree(d_small_primes));
            checkCudaErrors(hipFree(d_small_prime_masks));



        }
    }
}
