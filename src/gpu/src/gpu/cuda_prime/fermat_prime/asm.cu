
#include <hip/hip_runtime.h>

namespace nexusminer {
    namespace gpu {
        __device__ __forceinline__ uint32_t add_cc(uint32_t a, uint32_t b) {
            uint32_t r;

            asm volatile ("add.cc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
            return r;
        }

        __device__ __forceinline__ uint32_t addc_cc(uint32_t a, uint32_t b) {
            uint32_t r;

            asm volatile ("addc.cc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
            return r;
        }

        __device__ __forceinline__ uint32_t addc(uint32_t a, uint32_t b) {
            uint32_t r;

            asm volatile ("addc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
            return r;
        }

        __device__ __forceinline__ uint32_t sub_cc(uint32_t a, uint32_t b) {
            uint32_t r;

            asm volatile ("sub.cc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
            return r;
        }

        __device__ __forceinline__ uint32_t subc_cc(uint32_t a, uint32_t b) {
            uint32_t r;

            asm volatile ("subc.cc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
            return r;
        }

        __device__ __forceinline__ uint32_t subc(uint32_t a, uint32_t b) {
            uint32_t r;

            asm volatile ("subc.u32 %0, %1, %2;" : "=r"(r) : "r"(a), "r"(b));
            return r;
        }

        __device__ __forceinline__ uint32_t madlo(uint32_t a, uint32_t b, uint32_t c) {
            uint32_t r;

            asm volatile ("mad.lo.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
            return r;
        }

        __device__ __forceinline__ uint32_t madlo_cc(uint32_t a, uint32_t b, uint32_t c) {
            uint32_t r;

            asm volatile ("mad.lo.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
            return r;
        }

        __device__ __forceinline__ uint32_t madloc_cc(uint32_t a, uint32_t b, uint32_t c) {
            uint32_t r;

            asm volatile ("madc.lo.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
            return r;
        }

        __device__ __forceinline__ uint32_t madloc(uint32_t a, uint32_t b, uint32_t c) {
            uint32_t r;

            asm volatile ("madc.lo.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
            return r;
        }

        __device__ __forceinline__ uint32_t madhi(uint32_t a, uint32_t b, uint32_t c) {
            uint32_t r;

            asm volatile ("mad.hi.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
            return r;
        }

        __device__ __forceinline__ uint32_t madhi_cc(uint32_t a, uint32_t b, uint32_t c) {
            uint32_t r;

            asm volatile ("mad.hi.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
            return r;
        }

        __device__ __forceinline__ uint32_t madhic_cc(uint32_t a, uint32_t b, uint32_t c) {
            uint32_t r;

            asm volatile ("madc.hi.cc.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
            return r;
        }

        __device__ __forceinline__ uint32_t madhic(uint32_t a, uint32_t b, uint32_t c) {
            uint32_t r;

            asm volatile ("madc.hi.u32 %0, %1, %2, %3;" : "=r"(r) : "r"(a), "r"(b), "r"(c));
            return r;
        }

        __device__ __forceinline__ uint64_t mad_wide(uint32_t a, uint32_t b, uint64_t c) {
            uint64_t r;

            asm volatile ("mad.wide.u32 %0, %1, %2, %3;" : "=l"(r) : "r"(a), "r"(b), "l"(c));
            return r;
        }
    }
}


