#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "fermat_prime_impl.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>
#include "../fermat_prime/fermat_utils.cuh"
#include "../cuda_chain.cuh"


#ifndef checkCudaErrors
#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)
#endif

namespace nexusminer {
    namespace gpu {

        __device__  bool get_next_fermat_candidate(CudaChain& chain, uint64_t& base_offset, int& offset);
        __device__  bool update_fermat_status(CudaChain& chain, bool is_prime);

        __global__ void
        //__launch_bounds__(256, 1)

            kernel_fermat(uint64_t* offsets, uint64_t* offset_count,
                Cump<1024>* base_int, uint8_t* results, unsigned long long* test_count, unsigned long long* pass_count)
        {
            const unsigned int num_threads = blockDim.x;
            const unsigned int block_id = blockIdx.x;
            const unsigned int thread_index = threadIdx.x;
            const int threads_per_instance = 1;

            const uint32_t index = block_id * num_threads/threads_per_instance + thread_index/threads_per_instance;
            

            if (index < *offset_count)
            {
                const bool is_prime = powm_2(*base_int, offsets[index]) == 1;
                if (thread_index % threads_per_instance == 0)
                {
                    if (is_prime)
                    {
                        atomicAdd(pass_count, 1);
                    }
                    results[index] = is_prime ? 1 : 0;
                    atomicAdd(test_count, 1);

                }

            }

        }

        void Fermat_prime_impl::fermat_run()
        {
            //changing thread count seems to have negligible impact on the throughput
            const int32_t threads_per_block = 32*2;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            int blocks = (m_offset_count + instances_per_block - 1) / instances_per_block;

           kernel_fermat << <blocks, threads_per_block >> > (d_offsets, d_offset_count, d_base_int,
                d_results, d_fermat_test_count, d_fermat_pass_count);

            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }


        __global__ void fermat_test_chains(CudaChain* chains, uint32_t* chain_count,
            Cump<1024>* base_int, uint8_t* results, unsigned long long* test_count, unsigned long long* pass_count) {
            
            const unsigned int num_threads = blockDim.x;
            const unsigned int block_id = blockIdx.x;
            const unsigned int thread_index = threadIdx.x;
            const int threads_per_instance = 1;
            const uint32_t index = block_id * num_threads / threads_per_instance + thread_index / threads_per_instance;


            if (index >= *chain_count)
                return;

            uint64_t offset64, base_offset;
            int relative_offset;
            get_next_fermat_candidate(chains[index], base_offset, relative_offset);
            offset64 = base_offset + relative_offset;
           
            const bool is_prime = powm_2(*base_int, offset64) == 1;
            update_fermat_status(chains[index], is_prime);
            if (thread_index % threads_per_instance == 0)
            {
                if (is_prime)
                {
                    atomicAdd(pass_count, 1);
                }
                results[index] = is_prime ? 1 : 0;
                atomicAdd(test_count, 1);

            }

        }


        void Fermat_prime_impl::fermat_chain_run()
        {
            const int32_t threads_per_block = 32 * 2;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            uint32_t chain_count;
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_count, sizeof(*d_chain_count), hipMemcpyDeviceToHost));
            
            int blocks = (chain_count + instances_per_block - 1) / instances_per_block;
            fermat_test_chains << <blocks, threads_per_block >> > (d_chains, d_chain_count, d_base_int,
                d_results, d_fermat_test_count, d_fermat_pass_count);

            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }

        //allocate device memory for gpu fermat testing.  we use a fixed maximum batch size and allocate device memory once at the beginning. 
        void Fermat_prime_impl::fermat_init(uint64_t batch_size, int device)
        {

            m_device = device;

            checkCudaErrors(hipSetDevice(device));
            checkCudaErrors(hipMalloc(&d_base_int, sizeof(*d_base_int)));
            checkCudaErrors(hipMalloc(&d_offsets, sizeof(*d_offsets) * batch_size));
            checkCudaErrors(hipMalloc(&d_results, sizeof(*d_results) * batch_size));
            checkCudaErrors(hipMalloc(&d_offset_count, sizeof(*d_offset_count)));
            checkCudaErrors(hipMalloc(&d_fermat_test_count, sizeof(*d_fermat_test_count)));
            checkCudaErrors(hipMalloc(&d_fermat_pass_count, sizeof(*d_fermat_pass_count)));
            reset_stats();

        }

        void Fermat_prime_impl::fermat_free()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_base_int));
            checkCudaErrors(hipFree(d_offsets));
            checkCudaErrors(hipFree(d_results));
            checkCudaErrors(hipFree(d_offset_count));
            checkCudaErrors(hipFree(d_fermat_test_count));
            checkCudaErrors(hipFree(d_fermat_pass_count));
        }

        void Fermat_prime_impl::set_base_int(mpz_t base_big_int)
        {
            checkCudaErrors(hipSetDevice(m_device));
            Cump<1024> cuda_base_big_int;
            cuda_base_big_int.from_mpz(base_big_int);
            checkCudaErrors(hipMemcpy(d_base_int, &cuda_base_big_int, sizeof(cuda_base_big_int), hipMemcpyHostToDevice));
            mpz_set(m_base_int, base_big_int);
        }

        void Fermat_prime_impl::set_offsets(uint64_t offsets[], uint64_t offset_count)
        {
            checkCudaErrors(hipMemcpy(d_offsets, offsets, sizeof(*offsets) * offset_count, hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_offset_count, &offset_count, sizeof(offset_count), hipMemcpyHostToDevice));
            m_offset_count = offset_count;
        }

        void Fermat_prime_impl::get_results(uint8_t results[])
        {
            checkCudaErrors(hipMemcpy(results, d_results, sizeof(uint8_t) * m_offset_count, hipMemcpyDeviceToHost));
        }

        void Fermat_prime_impl::get_stats(uint64_t& fermat_tests, uint64_t& fermat_passes)
        {
            checkCudaErrors(hipMemcpy(&fermat_tests, d_fermat_test_count, sizeof(*d_fermat_test_count), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(&fermat_passes, d_fermat_pass_count, sizeof(*d_fermat_pass_count), hipMemcpyDeviceToHost));
        }

        void Fermat_prime_impl::reset_stats()
        {
            checkCudaErrors(hipMemset(d_fermat_test_count, 0, sizeof(*d_fermat_test_count)));
            checkCudaErrors(hipMemset(d_fermat_pass_count, 0, sizeof(*d_fermat_pass_count)));
        }

        void Fermat_prime_impl::set_chain_ptr(CudaChain* chains, uint32_t* chain_count)
        {
            d_chains = chains;
            d_chain_count = chain_count;
            uint32_t chain_count_test;
            checkCudaErrors(hipMemcpy(&chain_count_test, d_chain_count, sizeof(*d_chain_count), hipMemcpyDeviceToHost));
        }

        void Fermat_prime_impl::synchronize()
        {
            checkCudaErrors(hipDeviceSynchronize());
        }

        void Fermat_prime_impl::test_init(uint64_t batch_size, int device)
        {
            m_device = device;
            checkCudaErrors(hipSetDevice(device));
            checkCudaErrors(hipMalloc(&d_test_a, sizeof(*d_test_a) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_b, sizeof(*d_test_b) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_results, sizeof(*d_test_results) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_vector_size, sizeof(*d_test_vector_size)));

        }

        void Fermat_prime_impl::test_free()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_test_a));
            checkCudaErrors(hipFree(d_test_b));
            checkCudaErrors(hipFree(d_test_results));
            checkCudaErrors(hipFree(d_test_vector_size));

        }

        void Fermat_prime_impl::set_input_a(mpz_t* a, uint64_t count)
        {
            m_test_vector_a_size = count;
            Cump<1024>* vector_a = new Cump<1024>[count];
            for (auto i = 0; i < count; i++)
            {
                vector_a[i].from_mpz(a[i]);
            }
            checkCudaErrors(hipMemcpy(d_test_a, vector_a, sizeof(*vector_a) * count, hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_test_vector_size, &count, sizeof(count), hipMemcpyHostToDevice));
            delete[] vector_a;
        }

        void Fermat_prime_impl::set_input_b(mpz_t* b, uint64_t count)
        {
            m_test_vector_b_size = count;
            Cump<1024>* vector_b = new Cump<1024>[count];
            for (auto i = 0; i < count; i++)
            {
                vector_b[i].from_mpz(b[i]);
            }
            checkCudaErrors(hipMemcpy(d_test_b, vector_b, sizeof(*vector_b) * count, hipMemcpyHostToDevice));
            delete[] vector_b;
        }

        

        void Fermat_prime_impl::get_test_results(mpz_t* test_results)
        {
            Cump<1024>* results = new Cump<1024>[m_test_vector_a_size];
            checkCudaErrors(hipMemcpy(results, d_test_results, sizeof(*d_test_results) * m_test_vector_a_size, hipMemcpyDeviceToHost));
            for (auto i = 0; i < m_test_vector_a_size; i++)
            {
                //mpz_init(test_results[i]);
                results[i].to_mpz(test_results[i]);
            }
            delete[] results;
        }

        
        
        __global__ void 
        //__launch_bounds__(128, 1)
        logic_test_kernel(Cump<1024>* a, Cump<1024>* b, Cump<1024>* results, uint64_t* test_vector_size)
        {
            unsigned int num_threads = blockDim.x;
            unsigned int block_id = blockIdx.x;
            unsigned int thread_index = threadIdx.x;

            uint32_t index = block_id * num_threads + thread_index;
            
            if (index < *test_vector_size)
            {
                //uint32_t m_primed = -mod_inverse_32(b[index].m_limbs[0]);
                //Cump<1024> Rmodm = b[index].R_mod_m();
                //results[index] = montgomery_square_2(Rmodm, b[index], m_primed);
                //results[index] = montgomery_square(Rmodm, b[index], m_primed);
                
                //results[index] = a[index].add_ptx(b[index]);
                //results[index] = powm_2(b[index]);

                //results[index] = results[index] - Rmodm;
                //results[index] += 1;

                

                

            }

        }

        void Fermat_prime_impl::logic_test()
        {
            const int32_t threads_per_block = 32 * 8;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            int blocks = (m_test_vector_a_size + instances_per_block - 1) / instances_per_block;
            logic_test_kernel << <blocks, threads_per_block >> > (d_test_a, d_test_b, d_test_results, d_test_vector_size);
            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }

        
    }
}
