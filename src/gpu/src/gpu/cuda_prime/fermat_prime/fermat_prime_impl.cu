#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include ""

#include "fermat_prime_impl.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>
#include "../fermat_prime/fermat_utils.cuh"
#include "../cuda_chain.cuh"

#ifndef checkCudaErrors
#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)
#endif

namespace nexusminer {
    namespace gpu {

        __device__  bool get_next_fermat_candidate(CudaChain& chain, uint64_t& base_offset, int& offset);
        __device__  bool update_fermat_status(CudaChain& chain, bool is_prime);

        __global__ void
        //__launch_bounds__(256, 1)

            kernel_fermat(uint64_t* offsets, uint64_t* offset_count,
                Cump<1024>* base_int, uint8_t* results, unsigned long long* test_count, unsigned long long* pass_count)
        {
            const unsigned int num_threads = blockDim.x;
            const unsigned int block_id = blockIdx.x;
            const unsigned int thread_index = threadIdx.x;
            const int threads_per_instance = 1;

            const uint32_t index = block_id * num_threads/threads_per_instance + thread_index/threads_per_instance;
            

            if (index < *offset_count)
            {
                const bool is_prime = powm_2(*base_int, offsets[index]) == 1;
                if (thread_index % threads_per_instance == 0)
                {
                    if (is_prime)
                    {
                        atomicAdd(pass_count, 1);
                    }
                    results[index] = is_prime ? 1 : 0;
                    atomicAdd(test_count, 1);

                }

            }

        }

        void Fermat_prime_impl::fermat_run()
        {
            //changing thread count seems to have negligible impact on the throughput
            const int32_t threads_per_block = 32*2;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            int blocks = (m_offset_count + instances_per_block - 1) / instances_per_block;

           kernel_fermat <<<blocks, threads_per_block >>> (d_offsets, d_offset_count, d_base_int,
                d_results, d_fermat_test_count, d_fermat_pass_count);

            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }


        __global__ void fermat_test_chains(CudaChain* chains, uint32_t* chain_count,
            Cump<1024>* base_int, uint8_t* results, unsigned long long* test_count, unsigned long long* pass_count) {
            
            const unsigned int num_threads = blockDim.x;
            const unsigned int block_id = blockIdx.x;
            const unsigned int thread_index = threadIdx.x;
            const int threads_per_instance = 1;
            const uint32_t index = block_id * num_threads / threads_per_instance + thread_index / threads_per_instance;


            if (index >= *chain_count)
                return;

            uint64_t offset64, base_offset;
            int relative_offset;
            get_next_fermat_candidate(chains[index], base_offset, relative_offset);
            offset64 = base_offset + relative_offset;
           
            const bool is_prime = powm_2(*base_int, offset64);
            update_fermat_status(chains[index], is_prime);
            if (thread_index % threads_per_instance == 0)
            {
                if (is_prime)
                {
                    atomicAdd(pass_count, 1);
                }
                results[index] = is_prime ? 1 : 0;
                atomicAdd(test_count, 1);

            }

        }


        void Fermat_prime_impl::fermat_chain_run()
        {
            const int32_t threads_per_block = 32 * 2;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            uint32_t chain_count;
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_count, sizeof(*d_chain_count), hipMemcpyDeviceToHost));
            
            int blocks = (chain_count + instances_per_block - 1) / instances_per_block;
            fermat_test_chains <<<blocks, threads_per_block >>> (d_chains, d_chain_count, d_base_int,
                d_results, d_fermat_test_count, d_fermat_pass_count);

            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }

        //allocate device memory for gpu fermat testing.  we use a fixed maximum batch size and allocate device memory once at the beginning. 
        void Fermat_prime_impl::fermat_init(uint64_t batch_size, int device)
        {

            m_device = device;

            checkCudaErrors(hipSetDevice(device));
            checkCudaErrors(hipMalloc(&d_base_int, sizeof(*d_base_int)));
            checkCudaErrors(hipMalloc(&d_offsets, sizeof(*d_offsets) * batch_size));
            checkCudaErrors(hipMalloc(&d_results, sizeof(*d_results) * batch_size));
            checkCudaErrors(hipMalloc(&d_offset_count, sizeof(*d_offset_count)));
            checkCudaErrors(hipMalloc(&d_fermat_test_count, sizeof(*d_fermat_test_count)));
            checkCudaErrors(hipMalloc(&d_fermat_pass_count, sizeof(*d_fermat_pass_count)));
            checkCudaErrors(hipMemset(d_fermat_test_count, 0, sizeof(*d_fermat_test_count)));
            checkCudaErrors(hipMemset(d_fermat_pass_count, 0, sizeof(*d_fermat_pass_count)));
            checkCudaErrors(hipMalloc(&d_trial_division_test_count, sizeof(*d_trial_division_test_count)));
            checkCudaErrors(hipMalloc(&d_trial_division_composite_count, sizeof(*d_trial_division_composite_count)));
            checkCudaErrors(hipMemset(d_trial_division_test_count, 0, sizeof(*d_trial_division_test_count)));
            checkCudaErrors(hipMemset(d_trial_division_composite_count, 0, sizeof(*d_trial_division_composite_count)));

        }

        void Fermat_prime_impl::fermat_free()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_base_int));
            checkCudaErrors(hipFree(d_offsets));
            checkCudaErrors(hipFree(d_results));
            checkCudaErrors(hipFree(d_offset_count));
            checkCudaErrors(hipFree(d_fermat_test_count));
            checkCudaErrors(hipFree(d_fermat_pass_count));
            checkCudaErrors(hipFree(d_trial_division_test_count));
            checkCudaErrors(hipFree(d_trial_division_composite_count));
        }

        void Fermat_prime_impl::set_base_int(mpz_t base_big_int)
        {
            checkCudaErrors(hipSetDevice(m_device));
            Cump<1024> cuda_base_big_int;
            cuda_base_big_int.from_mpz(base_big_int);
            checkCudaErrors(hipMemcpy(d_base_int, &cuda_base_big_int, sizeof(cuda_base_big_int), hipMemcpyHostToDevice));
            mpz_set(m_base_int, base_big_int);
        }

        void Fermat_prime_impl::set_offsets(uint64_t offsets[], uint64_t offset_count)
        {
            checkCudaErrors(hipMemcpy(d_offsets, offsets, sizeof(*offsets) * offset_count, hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_offset_count, &offset_count, sizeof(offset_count), hipMemcpyHostToDevice));
            m_offset_count = offset_count;
        }

        void Fermat_prime_impl::get_results(uint8_t results[])
        {
            checkCudaErrors(hipMemcpy(results, d_results, sizeof(uint8_t) * m_offset_count, hipMemcpyDeviceToHost));
        }

        void Fermat_prime_impl::get_stats(uint64_t& fermat_tests, uint64_t& fermat_passes,
            uint64_t& trial_division_tests, uint64_t& trial_division_composites)
        {
            checkCudaErrors(hipMemcpy(&fermat_tests, d_fermat_test_count, sizeof(*d_fermat_test_count), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(&fermat_passes, d_fermat_pass_count, sizeof(*d_fermat_pass_count), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(&trial_division_tests, d_trial_division_test_count, sizeof(*d_trial_division_test_count), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(&trial_division_composites, d_trial_division_composite_count, sizeof(*d_trial_division_composite_count), hipMemcpyDeviceToHost));
        }

        void Fermat_prime_impl::reset_stats()
        {
            checkCudaErrors(hipMemset(d_fermat_test_count, 0, sizeof(*d_fermat_test_count)));
            checkCudaErrors(hipMemset(d_fermat_pass_count, 0, sizeof(*d_fermat_pass_count)));
            checkCudaErrors(hipMemset(d_trial_division_test_count, 0, sizeof(*d_trial_division_test_count)));
            checkCudaErrors(hipMemset(d_trial_division_composite_count, 0, sizeof(*d_trial_division_composite_count)));
        }

        void Fermat_prime_impl::set_chain_ptr(CudaChain* chains, uint32_t* chain_count)
        {
            d_chains = chains;
            d_chain_count = chain_count;
            uint32_t chain_count_test;
            checkCudaErrors(hipMemcpy(&chain_count_test, d_chain_count, sizeof(*d_chain_count), hipMemcpyDeviceToHost));
        }

        void Fermat_prime_impl::synchronize()
        {
            checkCudaErrors(hipDeviceSynchronize());
        }

        __global__ void trial_division_chains(CudaChain* chains, uint32_t* chain_count, trial_divisors_uint32_t* trial_divisors,
            uint32_t* trial_divisor_count, unsigned long long* test_count, unsigned long long* composite_count) {

            const unsigned int num_threads = blockDim.x;
            const unsigned int block_id = blockIdx.x;
            const unsigned int thread_index = threadIdx.x;
            const int threads_per_instance = 1;
            const uint32_t index = block_id * num_threads / threads_per_instance + thread_index / threads_per_instance;


            if (index >= *chain_count)
                return;

            uint64_t offset64, base_offset, prime_offset;
            int relative_offset;
            get_next_fermat_candidate(chains[index], base_offset, relative_offset);
            offset64 = base_offset + relative_offset;
            bool is_composite = false;
            for (int i = 0; i < *trial_divisor_count; i++)
            {
                prime_offset = trial_divisors[i].starting_multiple + offset64;
                if (prime_offset % trial_divisors[i].divisor == 0)
                {
                    is_composite = true;
                    break;
                }
            }

            if (is_composite)
                update_fermat_status(chains[index], false);
            if (thread_index % threads_per_instance == 0)
            {
                if (is_composite)
                {
                    atomicAdd(composite_count, 1);
                }
                atomicAdd(test_count, *trial_divisor_count);

            }

        }

        void Fermat_prime_impl::trial_division_chain_run()
        {
            const int32_t threads_per_block = 1024;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            uint32_t chain_count;
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_count, sizeof(*d_chain_count), hipMemcpyDeviceToHost));

            int blocks = (chain_count + instances_per_block - 1) / instances_per_block;
            trial_division_chains <<<blocks, threads_per_block >>> (d_chains, d_chain_count, d_trial_divisors, 
                d_trial_divisor_count, d_trial_division_test_count, d_trial_division_composite_count);

            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }

        void Fermat_prime_impl::trial_division_init(uint32_t trial_divisor_count, trial_divisors_uint32_t trial_divisors[],
            int device)
        {
            checkCudaErrors(hipSetDevice(device));
            checkCudaErrors(hipMalloc(&d_trial_divisor_count, sizeof(*d_trial_divisor_count)));
            checkCudaErrors(hipMalloc(&d_trial_divisors, trial_divisor_count * sizeof(*d_trial_divisors)));
            
            checkCudaErrors(hipMemcpy(d_trial_divisors, trial_divisors, trial_divisor_count * sizeof(*d_trial_divisors), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_trial_divisor_count, &trial_divisor_count, sizeof(*d_trial_divisor_count), hipMemcpyHostToDevice));

        }

        void Fermat_prime_impl::trial_division_free()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_trial_divisor_count));
            checkCudaErrors(hipFree(d_trial_divisors));
            

        }

        void Fermat_prime_impl::test_init(uint64_t batch_size, int device)
        {
            m_device = device;
            checkCudaErrors(hipSetDevice(device));
            checkCudaErrors(hipMalloc(&d_test_a, sizeof(*d_test_a) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_b, sizeof(*d_test_b) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_results, sizeof(*d_test_results) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_vector_size, sizeof(*d_test_vector_size)));

        }

        void Fermat_prime_impl::test_free()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_test_a));
            checkCudaErrors(hipFree(d_test_b));
            checkCudaErrors(hipFree(d_test_results));
            checkCudaErrors(hipFree(d_test_vector_size));

        }

        void Fermat_prime_impl::set_input_a(mpz_t* a, uint64_t count)
        {
            m_test_vector_a_size = count;
            Cump<1024>* vector_a = new Cump<1024>[count];
            for (auto i = 0; i < count; i++)
            {
                vector_a[i].from_mpz(a[i]);
            }
            checkCudaErrors(hipMemcpy(d_test_a, vector_a, sizeof(*vector_a) * count, hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_test_vector_size, &count, sizeof(count), hipMemcpyHostToDevice));
            delete[] vector_a;
        }

        void Fermat_prime_impl::set_input_b(mpz_t* b, uint64_t count)
        {
            m_test_vector_b_size = count;
            Cump<1024>* vector_b = new Cump<1024>[count];
            for (auto i = 0; i < count; i++)
            {
                vector_b[i].from_mpz(b[i]);
            }
            checkCudaErrors(hipMemcpy(d_test_b, vector_b, sizeof(*vector_b) * count, hipMemcpyHostToDevice));
            delete[] vector_b;
        }

        

        void Fermat_prime_impl::get_test_results(mpz_t* test_results)
        {
            Cump<1024>* results = new Cump<1024>[m_test_vector_a_size];
            checkCudaErrors(hipMemcpy(results, d_test_results, sizeof(*d_test_results) * m_test_vector_a_size, hipMemcpyDeviceToHost));
            for (auto i = 0; i < m_test_vector_a_size; i++)
            {
                //mpz_init(test_results[i]);
                results[i].to_mpz(test_results[i]);
            }
            delete[] results;
        }

        
        //this is a generic test kernel for evaluating big int math functions
        __global__ void 
        //__launch_bounds__(128, 1)
        logic_test_kernel(Cump<1024>* a, Cump<1024>* b, Cump<1024>* results, uint64_t* test_vector_size)
        {
            unsigned int num_threads = blockDim.x;
            unsigned int block_id = blockIdx.x;
            unsigned int thread_index = threadIdx.x;

            uint32_t index = block_id * num_threads + thread_index;
            
            if (index < *test_vector_size)
            {
                //uint32_t m_primed = -mod_inverse_32(b[index].m_limbs[0]);
                //Cump<1024> Rmodm = b[index].R_mod_m();
                //results[index] = montgomery_square_2(Rmodm, b[index], m_primed);
                //results[index] = montgomery_square(Rmodm, b[index], m_primed);
                
                //results[index] = a[index].add_ptx(b[index]);
                //results[index] = powm_2(b[index]);

                //results[index] = results[index] - Rmodm;
                //results[index] += 1;

                

                

            }

        }

        void Fermat_prime_impl::logic_test()
        {
            const int32_t threads_per_block = 32 * 8;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            int blocks = (m_test_vector_a_size + instances_per_block - 1) / instances_per_block;
            logic_test_kernel <<<blocks, threads_per_block >>> (d_test_a, d_test_b, d_test_results, d_test_vector_size);
            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }

        
    }
}
