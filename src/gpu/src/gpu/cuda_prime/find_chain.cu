#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include ""
#include "sieve_impl.cuh"
#include "sieve.hpp"
#include "find_chain.cuh"
#include "sieve_lookup_tables.cuh"
#include "cuda_chain.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>


namespace nexusminer {
    namespace gpu {

        __device__ void cuda_chain_push_back(CudaChain& chain, uint16_t offset);
        __device__ void cuda_chain_open(CudaChain& chain, uint64_t base_offset);
        __device__  bool is_there_still_hope(CudaChain& chain);
        __device__  void get_best_fermat_chain(const CudaChain& chain, uint64_t& base_offset, int& offset, int& best_length);


        //get the nth bit from the sieve.
        __device__ __forceinline__ bool get_bit(uint64_t bit_position, Cuda_sieve::sieve_word_t* sieve)
        {
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            
            uint64_t word = bit_position / sieve_bits_per_word;
            unsigned bit_position_in_word = bit_position % sieve_bits_per_word;
            return ((sieve[word] >> bit_position_in_word) & 1) == 1;

        }

        //search the sieve for chains that meet the minimum length requirement.  
        __global__ void find_chain_kernel(Cuda_sieve::sieve_word_t* sieve, CudaChain* chains, uint32_t* chain_index, uint64_t sieve_start_offset,
            unsigned long long* chain_stat_count, Cuda_sieve::Cuda_sieve_properties sieve_properties)
        {

            //const uint64_t sieve_size = Cuda_sieve::m_sieve_total_size;
            const uint32_t sieve_bits_per_word = Cuda_sieve::m_sieve_word_byte_count * 8;
            const uint64_t sieve_total_bits = sieve_properties.m_sieve_total_size * sieve_bits_per_word;
            uint64_t num_blocks = gridDim.x;
            uint64_t num_threads = blockDim.x;
            uint64_t block_id = blockIdx.x;
            uint64_t index = block_id * num_threads + threadIdx.x;
            uint64_t stride = num_blocks * num_threads;
            unsigned int sieve_offset;
            unsigned int gap;
            uint64_t chain_start, prime_candidate_offset;

            //shared copies of lookup tables
            __shared__ unsigned int sieve30_offsets_shared[8];
            __shared__ unsigned int sieve30_gaps_shared[8];
            //local stats
            __shared__ uint32_t chain_count_shared;
            
            if (threadIdx.x < 8)
            {
                int i = threadIdx.x;
                sieve30_offsets_shared[i] = sieve30_offsets[i];
                sieve30_gaps_shared[i] = sieve30_gaps[i];
            }
            
            if (threadIdx.x == 0)
                chain_count_shared = 0;
            __syncthreads();
           
            //search each sieve location for a possible chain
            for (uint64_t i = index; i < sieve_total_bits; i += stride)
            {
              
                //gross checks to ensure its possible to form a chain
                uint64_t word = i / sieve_bits_per_word;
                if (sieve[word] == 0)
                    continue;
                //check if the next 4 bytes (4*30 = range of 120 integers) has enough prime candidates to form a chain 
                //this is only valid up to min chain length 9.  above 9 requires 5 bytes.
                if (word < sieve_properties.m_sieve_total_size - 1)
                {
                    unsigned int next_4_bytes = 0;
                    unsigned int byte_index = (i/8) % 4;
                    next_4_bytes = (sieve[word] >> (byte_index * 8)) & 0xFF;
                    next_4_bytes |= (((sieve[word + (byte_index >= 3 ? 1 : 0)] >> ((byte_index + 1) % 4) * 8) & 0xFF) << 8);
                    next_4_bytes |= (((sieve[word + (byte_index >= 2 ? 1 : 0)] >> ((byte_index + 2) % 4) * 8) & 0xFF) << 16);
                    next_4_bytes |= (((sieve[word + (byte_index >= 1 ? 1 : 0)] >> ((byte_index + 3) % 4) * 8) & 0xFF) << 24);

                    int popc = __popc(next_4_bytes);
                    if (popc < Cuda_sieve::m_min_chain_length)
                        continue;
                }

                //chain must start with a prime
                if (!get_bit(i, sieve))
                {
                    continue;
                }
                //search left for another prime less than max gap away
                uint64_t j = i - 1;
                gap = sieve30_gaps_shared[j % 8];
                while (j < i && gap <= maxGap)
                {
                    if (get_bit(j, sieve))
                    {
                        //there is a valid element to the left.  this is not the first element in a chain. abort.
                        break;
                    }
                    j--;
                    gap += sieve30_gaps_shared[j % 8];
                }
                if (gap <= maxGap)
                    continue;
                //this is the start of a possible chain.  search right
                //where are we in the wheel
                sieve_offset = sieve30_offsets_shared[i % 8u];
                chain_start = sieve_start_offset + i / 8 * 30 + sieve_offset;
                CudaChain current_chain;
                cuda_chain_open(current_chain, chain_start);
                j = i;
                gap = sieve30_gaps_shared[j % 8u];
                j++;
                while (j < sieve_total_bits && gap <= maxGap)
                {
                    if (get_bit(j, sieve))
                    {
                        //another possible candidate.  add it to the chain
                        gap = 0;
                        sieve_offset = sieve30_offsets_shared[j % 8u];
                        prime_candidate_offset = sieve_start_offset + j / 8 * 30 + sieve_offset;
                        uint16_t offset = prime_candidate_offset - chain_start;
                        //printf("%" PRIu64 " %u\n", chain_start, prime_candidate_offset);
                        cuda_chain_push_back(current_chain, offset);
                    }
                    gap += sieve30_gaps_shared[j % 8u];
                    j++;
                        
                }
                //we reached the end of the chain.  check if it meets the length requirement
                if (current_chain.m_offset_count >= Cuda_sieve::m_min_chain_length)
                {
                    //increment the chain list index
                    uint32_t chain_idx = atomicInc(chain_index, Cuda_sieve::m_max_chains);
                    //copy the current chain to the global list
                    chains[chain_idx] = current_chain;
                    //updated block level stats
                    atomicInc(&chain_count_shared, 0xFFFFFFFF);
                }
            }
            //update global chain stats
            __syncthreads();
            if (threadIdx.x == 0)
                atomicAdd(chain_stat_count, chain_count_shared);
        }

        __device__ void close_chain(const CudaChain& chain, uint32_t* chain_index, CudaChain* chains, uint32_t* chain_count_shared)
        {
            //We reached the end of the chain.  
            if (chain.m_offset_count >= Cuda_sieve::m_min_chain_length)
            {
                //increment the chain list index
                uint32_t chain_idx = atomicInc(chain_index, Cuda_sieve::m_max_chains);
                //copy the current chain to the global list
                chains[chain_idx] = chain;
                //updated block level stats
                atomicInc(chain_count_shared, 0xFFFFFFFF);
            }
        }

        //alternative chain finder
        //each kernel block is a sieve segment.  Each thread searches a range of 2310*4 within a segment.   
        __global__ void find_chain_kernel2(Cuda_sieve::sieve_word_t* sieve, CudaChain* chains, uint32_t* chain_index, uint64_t sieve_start_offset,
            unsigned long long* chain_stat_count, Cuda_sieve::Cuda_sieve_properties sieve_properties)
        {
            const unsigned int search_range = Cuda_sieve::m_sieve_chain_search_boundary * Cuda_sieve::m_sieve_word_byte_count;
            const unsigned int search_words = search_range / Cuda_sieve::m_sieve_word_range;
            unsigned int total_search_regions = sieve_properties.m_sieve_range / search_range;
            unsigned int num_blocks = gridDim.x;
            unsigned int block_id = blockIdx.x / Cuda_sieve::m_kernel_segments_per_block;
            unsigned int segment_id = blockIdx.x % Cuda_sieve::m_kernel_segments_per_block;
            unsigned int index = threadIdx.x;
            unsigned int search_regions_per_kernel_block = (total_search_regions + num_blocks - 1) / num_blocks;
            unsigned int stride = blockDim.x;
            unsigned int gap = 0;
            uint32_t chain_start;
            uint64_t segment_offset = sieve_start_offset + block_id * sieve_properties.m_block_range + segment_id * sieve_properties.m_segment_range;
            uint32_t sieve_segment_index = block_id * sieve_properties.m_kernel_sieve_size_words_per_block + segment_id * sieve_properties.m_kernel_sieve_size_words;
            uint32_t sieve_index;
            //shared copies of lookup tables
            __shared__ unsigned int sieve30_offsets_shared[8];
            //local stats
            __shared__ uint32_t chain_count_shared;
            //local shared copy of the sieve
            //__shared__ Cuda_sieve::sieve_word_t sieve_shared[Cuda_sieve::m_kernel_sieve_size_words];

            if (threadIdx.x < 8)
            {
                int i = threadIdx.x;
                sieve30_offsets_shared[i] = sieve30_offsets[i];
            }

            if (threadIdx.x == 0)
            {
                chain_count_shared = 0;
            }
                
            __syncthreads();
           
            sieve_index = sieve_segment_index + index * search_words;
            for (unsigned int region = index; region < search_regions_per_kernel_block; region += stride)
            {
                bool chain_in_process = false;
                CudaChain current_chain;
                uint64_t region_offset = segment_offset + region * search_range;
                chain_start = 0;
                //sieve_index = region * search_words + sieve_segment_index;
                //sieve_index += sieve_index_increment;
                uint32_t last_offset = 0;
                //iterate through each word in the search region
                bool previous_word_last_bit_set = false;
                for (unsigned int word = 0; word < search_words; word++, sieve_index++)
                {
                    uint32_t sieve_word = sieve[sieve_index];
                    uint32_t next_word = word >= search_words ? 0 : sieve[sieve_index + 1];
                    bool next_word_first_bit_set = (next_word & 1) == 1;
                    if (chain_in_process)
                    {
                        uint32_t word_start = sieve_word & 0x7;
                        bool first_bit_set = (sieve_word & 1) == 1;
                        //if the first 3 bits are zeros any in process chain is broken
                        //if the last bit of the previous word and the first bit of the currnet word are both 0 the chain is broken
                        if (word_start == 0 || (!previous_word_last_bit_set && !first_bit_set))
                        {
                            //We reached the end of the chain.  
                            close_chain(current_chain, chain_index, chains, &chain_count_shared);
                            chain_in_process = false;
                        }
                    }
                    bool last_bit_set = sieve_word >= 0x80000000;
                    previous_word_last_bit_set = last_bit_set;
                    //gross check to ensure there are enough set bits in the word to make it worthwhile to process
                    if (!chain_in_process)
                    {
                        //if bits 7 and 8 are both zero, no chain originating in the first byte can make it through.  discard any set bits in the first byte
                        uint32_t first_byte_transition = sieve_word & 0x000000180;
                        if (first_byte_transition == 0)
                            sieve_word &= 0xFFFFFF00;
                        uint32_t second_byte_transition = sieve_word & 0x000018000;
                        if (second_byte_transition == 0 && __popc(sieve_word & 0x0000FFFF) < Cuda_sieve::m_min_chain_length)
                            sieve_word &= 0xFFFF0000;
                        uint32_t third_byte_transition = sieve_word & 0x001800000;
                        if (third_byte_transition == 0 && __popc(sieve_word & 0x00FFFFFF) < Cuda_sieve::m_min_chain_length)
                            sieve_word &= 0xFF000000;

                        //if the last 3 bits are all zero, or the last bit is zero and first bit of the next word is zero, any chain must end at the current word
                        uint32_t word_end = sieve_word & 0xE0000000;
                        bool chain_must_end = (word_end == 0 || (!last_bit_set && !next_word_first_bit_set));
                        int popc = __popc(sieve_word);
                        if (chain_must_end && popc < Cuda_sieve::m_min_chain_length)
                            continue;
                        
                    }

                    //iterate through each set bit in the sieve word
                    for (unsigned int b = sieve_word; b > 0; b &= b - 1)
                    {
                        //determine the position of the set bit in the sieve word.
                        uint8_t lowest_set_bit = __ffs(b) - 1;  //__ffs is a cuda primitive that finds the index of the lowest set bit in a word (ones based).
                        uint8_t byte_index = lowest_set_bit / 8;
                        unsigned int sieve30_offset = sieve30_offsets_shared[lowest_set_bit % 8];
                        uint32_t local_offset = word * Cuda_sieve::m_sieve_word_range +
                            byte_index * Cuda_sieve::m_sieve_byte_range + sieve30_offset;
                        gap = local_offset - last_offset;
                        /*if (region_offset + local_offset == 2055301)
                            printf("sieve word %u %x region offset %llu local offset % u\n", sieve_index + word, sieve[sieve_index + word], region_offset, local_offset);*/
                        if (chain_in_process)
                        {
                            if (gap > maxGap)
                            {
                                //We reached the end of the chain.  
                                close_chain(current_chain, chain_index, chains, &chain_count_shared);
                                //start a new chain
                                cuda_chain_open(current_chain, region_offset + local_offset);
                                chain_start = local_offset;
                                last_offset = local_offset;
                            }
                            else
                            {
                                //grow the chain
                                uint16_t offset_from_chain_start = local_offset - chain_start;
                                cuda_chain_push_back(current_chain, offset_from_chain_start);
                                last_offset = local_offset;
                            }
                        }
                        else
                        {
                            //start a new chain
                            cuda_chain_open(current_chain, region_offset + local_offset);
                            last_offset = local_offset;
                            chain_start = local_offset;
                            chain_in_process = true;
                        }
                    }
                }
                //we reached the end of the search region.  do a final check on the chain in process
                if (chain_in_process)
                {
                    close_chain(current_chain, chain_index, chains, &chain_count_shared);
                }
            }
            
            //update global chain stats
            __syncthreads();
            if (threadIdx.x == 0)
            {
                atomicAdd(chain_stat_count, chain_count_shared);
            }
            
        }

        //go through the list of chains.  copy winners to the long chain list.  copy survivors to a temporary chain
        __global__ void filter_busted_chains(CudaChain* chains, uint32_t* chain_index, CudaChain* surviving_chains,
            uint32_t* surviving_chain_index, CudaChain* long_chains, uint32_t* long_chain_index, uint32_t* histogram)
        {
            uint32_t num_threads = blockDim.x;
            uint32_t block_id = blockIdx.x;
            uint32_t index = block_id * num_threads + threadIdx.x;

            if (index >= *chain_index)
                return;
            if (index == 0)
            {
                *surviving_chain_index = 0;
            }
            __syncthreads();
            //printf("%" PRIu64 " %u\n", index, *chain_index);
            if (!is_there_still_hope(chains[index]))
            {
                //this chain is busted.  check how long it is
                //collect stats
                //only count chains 3 or longer to minimize memory accesses
                if (chains[index].m_prime_count >= 3)
                {
                    int chain_length, local_offset;
                    uint64_t base_offset;
                    get_best_fermat_chain(chains[index], base_offset, local_offset, chain_length);
                    uint32_t histogram_chain_length = min(chain_length, Cuda_sieve::chain_histogram_max);
                    if (chain_length >= 3)
                        atomicInc(&histogram[histogram_chain_length], 0xFFFFFFFF);

                    //check for winners
                    if (chain_length >= chains[index].m_min_chain_report_length)
                    {
                        //chain is long. save it. 
                        uint32_t last_long_chain_index = atomicInc(long_chain_index, Cuda_sieve::m_max_long_chains);
                        long_chains[last_long_chain_index] = chains[index];
                    }
                }
            }
            else
            {
                //copy chain to the survival list
                uint32_t last_surviving_chain_index = atomicInc(surviving_chain_index, Cuda_sieve::m_max_chains);
                surviving_chains[last_surviving_chain_index] = chains[index];
            }
        }
        
    }
}
