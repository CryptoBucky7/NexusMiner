#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "big_int_impl.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>
#include "../big_int/fermat_utils.cuh"


#ifndef checkCudaErrors
#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)
#endif

namespace nexusminer {
    namespace gpu {

        __global__ void
            kernel_fermat(uint64_t* offsets, uint64_t* offset_count,
                Cump<1024>* base_int, uint8_t* results, unsigned long long* test_count, unsigned long long* pass_count)
        {
            unsigned int num_threads = blockDim.x;
            unsigned int block_id = blockIdx.x;
            unsigned int thread_index = threadIdx.x;

            uint32_t index = block_id * num_threads + thread_index;

            if (index < *offset_count)
            {
                Cump<1024> prime_candidate = *base_int + offsets[index];
                uint32_t m_primed = -mod_inverse_32(prime_candidate.m_limbs[0]);
                Cump<1024> Rmodm = prime_candidate.R_mod_m();
                
                bool is_prime = powm_2(prime_candidate, Rmodm, m_primed) == 1;
                if (is_prime)
                {
                    atomicAdd(pass_count, 1);
                }
                results[index] = is_prime ? 1 : 0;
                atomicAdd(test_count, 1);
                
            }

        }

        void Big_int_impl::fermat_run()
        {
            //changing thread count seems to have negligible impact on the throughput
            const int32_t threads_per_block = 128;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            int blocks = (m_offset_count + instances_per_block - 1) / instances_per_block;

           kernel_fermat << <blocks, threads_per_block >> > (d_offsets, d_offset_count, d_base_int,
                d_results, d_fermat_test_count, d_fermat_pass_count);


            checkCudaErrors(hipDeviceSynchronize());
        }

        void Big_int_impl::fermat_chain_run()
        {

            int32_t threads_per_block = 256;
            int32_t threads_per_instance = 1;
            int32_t instances_per_block = threads_per_block / threads_per_instance;

            uint32_t chain_count;
            checkCudaErrors(hipMemcpy(&chain_count, d_chain_count, sizeof(*d_chain_count), hipMemcpyDeviceToHost));
            int blocks = (chain_count + instances_per_block - 1) / instances_per_block;

            /*fermat_test_chains << <blocks, threads_per_block >> > (d_report, d_chains, d_chain_count, d_base_int,
                d_results, d_fermat_test_count, d_fermat_pass_count);*/

            //checkCudaErrors(hipDeviceSynchronize());
        }

        //allocate device memory for gpu fermat testing.  we use a fixed maximum batch size and allocate device memory once at the beginning. 
        void Big_int_impl::fermat_init(uint64_t batch_size, int device)
        {

            m_device = device;

            checkCudaErrors(hipSetDevice(device));
            checkCudaErrors(hipMalloc(&d_base_int, sizeof(*d_base_int)));
            checkCudaErrors(hipMalloc(&d_offsets, sizeof(*d_offsets) * batch_size));
            checkCudaErrors(hipMalloc(&d_results, sizeof(*d_results) * batch_size));
            checkCudaErrors(hipMalloc(&d_offset_count, sizeof(*d_offset_count)));
            checkCudaErrors(hipMalloc(&d_fermat_test_count, sizeof(*d_fermat_test_count)));
            checkCudaErrors(hipMalloc(&d_fermat_pass_count, sizeof(*d_fermat_pass_count)));
            reset_stats();

        }

        void Big_int_impl::fermat_free()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_base_int));
            checkCudaErrors(hipFree(d_offsets));
            checkCudaErrors(hipFree(d_results));
            checkCudaErrors(hipFree(d_offset_count));
            checkCudaErrors(hipFree(d_fermat_test_count));
            checkCudaErrors(hipFree(d_fermat_pass_count));
        }

        void Big_int_impl::set_base_int(mpz_t base_big_int)
        {
            checkCudaErrors(hipSetDevice(m_device));
            Cump<1024> cuda_base_big_int;
            cuda_base_big_int.from_mpz(base_big_int);
            checkCudaErrors(hipMemcpy(d_base_int, &cuda_base_big_int, sizeof(cuda_base_big_int), hipMemcpyHostToDevice));
            mpz_set(m_base_int, base_big_int);
        }

        void Big_int_impl::set_offsets(uint64_t offsets[], uint64_t offset_count)
        {
            checkCudaErrors(hipMemcpy(d_offsets, offsets, sizeof(*offsets) * offset_count, hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_offset_count, &offset_count, sizeof(offset_count), hipMemcpyHostToDevice));
            m_offset_count = offset_count;
        }

        void Big_int_impl::get_results(uint8_t results[])
        {
            checkCudaErrors(hipMemcpy(results, d_results, sizeof(uint8_t) * m_offset_count, hipMemcpyDeviceToHost));
        }

        void Big_int_impl::get_stats(uint64_t& fermat_tests, uint64_t& fermat_passes)
        {
            checkCudaErrors(hipMemcpy(&fermat_tests, d_fermat_test_count, sizeof(*d_fermat_test_count), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(&fermat_passes, d_fermat_pass_count, sizeof(*d_fermat_pass_count), hipMemcpyDeviceToHost));
        }

        void Big_int_impl::reset_stats()
        {
            checkCudaErrors(hipMemset(d_fermat_test_count, 0, sizeof(*d_fermat_test_count)));
            checkCudaErrors(hipMemset(d_fermat_pass_count, 0, sizeof(*d_fermat_pass_count)));
        }

        void Big_int_impl::set_chain_ptr(CudaChain* chains, uint32_t* chain_count)
        {
            d_chains = chains;
            d_chain_count = chain_count;
            uint32_t chain_count_test;
            checkCudaErrors(hipMemcpy(&chain_count_test, d_chain_count, sizeof(*d_chain_count), hipMemcpyDeviceToHost));
        }

        void Big_int_impl::synchronize()
        {
            checkCudaErrors(hipDeviceSynchronize());
        }

        void Big_int_impl::test_init(uint64_t batch_size, int device)
        {
            m_device = device;
            checkCudaErrors(hipSetDevice(device));
            checkCudaErrors(hipMalloc(&d_test_a, sizeof(*d_test_a) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_b, sizeof(*d_test_b) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_results, sizeof(*d_test_results) * batch_size));
            checkCudaErrors(hipMalloc(&d_test_vector_size, sizeof(*d_test_vector_size)));

        }

        void Big_int_impl::test_free()
        {
            checkCudaErrors(hipSetDevice(m_device));
            checkCudaErrors(hipFree(d_test_a));
            checkCudaErrors(hipFree(d_test_b));
            checkCudaErrors(hipFree(d_test_results));
            checkCudaErrors(hipFree(d_test_vector_size));

        }

        void Big_int_impl::set_input_a(mpz_t* a, uint64_t count)
        {
            m_test_vector_a_size = count;
            Cump<1024>* vector_a = new Cump<1024>[count];
            for (auto i = 0; i < count; i++)
            {
                vector_a[i].from_mpz(a[i]);
            }
            checkCudaErrors(hipMemcpy(d_test_a, vector_a, sizeof(*vector_a) * count, hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_test_vector_size, &count, sizeof(count), hipMemcpyHostToDevice));
            delete[] vector_a;
        }

        void Big_int_impl::set_input_b(mpz_t* b, uint64_t count)
        {
            m_test_vector_b_size = count;
            Cump<1024>* vector_b = new Cump<1024>[count];
            for (auto i = 0; i < count; i++)
            {
                vector_b[i].from_mpz(b[i]);
            }
            checkCudaErrors(hipMemcpy(d_test_b, vector_b, sizeof(*vector_b) * count, hipMemcpyHostToDevice));
            delete[] vector_b;
        }

        

        void Big_int_impl::get_test_results(mpz_t* test_results)
        {
            Cump<1024>* results = new Cump<1024>[m_test_vector_a_size];
            checkCudaErrors(hipMemcpy(results, d_test_results, sizeof(*d_test_results) * m_test_vector_a_size, hipMemcpyDeviceToHost));
            for (auto i = 0; i < m_test_vector_a_size; i++)
            {
                //mpz_init(test_results[i]);
                results[i].to_mpz(test_results[i]);
            }
            delete[] results;
        }

        //__global__ void add_kernel(Cump<1024>* a, Cump<1024>* b, Cump<1024>* results, uint64_t* test_vector_size)
        //{
        //    unsigned int num_threads = blockDim.x;
        //    unsigned int block_id = blockIdx.x;
        //    unsigned int thread_index = threadIdx.x;
        //    
        //    uint32_t index = block_id * num_threads + thread_index;
        //    //printf("index: %u\n", index);
        //    if (index < *test_vector_size)
        //    {
        //        results[index] = a[index] + b[index];
        //        //char s[400];
        //        //results[index].to_cstr(s);
        //        //printf("%s\n", s);
        //    }
        //    
        //}

        void Big_int_impl::add()
        {
            
            const int32_t threads_per_block = 32*16;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            int blocks = (m_test_vector_a_size + instances_per_block - 1) / instances_per_block;
            //add_kernel <<<blocks, threads_per_block >>> (d_test_a, d_test_b, d_test_results, d_test_vector_size);
            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }

        /*__global__ void subtract_kernel(Cump<1024>* a, Cump<1024>* b, Cump<1024>* results, uint64_t* test_vector_size)
        {
            unsigned int num_threads = blockDim.x;
            unsigned int block_id = blockIdx.x;
            unsigned int thread_index = threadIdx.x;

            uint32_t index = block_id * num_threads + thread_index;
            if (index < *test_vector_size)
            {
                results[index] = a[index] - b[index];
            }

        }*/

        void Big_int_impl::subtract()
        {
            const int32_t threads_per_block = 32 * 12;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            int blocks = (m_test_vector_a_size + instances_per_block - 1) / instances_per_block;
            //subtract_kernel << <blocks, threads_per_block >> > (d_test_a, d_test_b, d_test_results, d_test_vector_size);
            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }

        
        __global__ void 
        //__launch_bounds__(128, 1)
        logic_test_kernel(Cump<1024>* a, Cump<1024>* b, Cump<1024>* results, uint64_t* test_vector_size)
        {
            unsigned int num_threads = blockDim.x;
            unsigned int block_id = blockIdx.x;
            unsigned int thread_index = threadIdx.x;

            uint32_t index = block_id * num_threads + thread_index;
            
            if (index < *test_vector_size)
            {
                uint32_t m_primed = -mod_inverse_32(b[index].m_limbs[0]);
                Cump<1024> Rmodm = b[index].R_mod_m();
                //results[index] = montgomery_square_2(Rmodm, b[index], m_primed);
                //results[index] = montgomery_square(Rmodm, b[index], m_primed);
                
                //results[index] = a[index].add_ptx(b[index]);
                results[index] = powm_2(b[index], Rmodm, m_primed);

                //results[index] = results[index] - Rmodm;
                //results[index] += 1;

                

                

            }

        }

        void Big_int_impl::logic_test()
        {
            const int32_t threads_per_block = 32 * 8;
            const int32_t threads_per_instance = 1;
            const int32_t instances_per_block = threads_per_block / threads_per_instance;

            int blocks = (m_test_vector_a_size + instances_per_block - 1) / instances_per_block;
            logic_test_kernel << <blocks, threads_per_block >> > (d_test_a, d_test_b, d_test_results, d_test_vector_size);
            checkCudaErrors(hipPeekAtLastError());
            checkCudaErrors(hipDeviceSynchronize());
        }

        
    }
}
