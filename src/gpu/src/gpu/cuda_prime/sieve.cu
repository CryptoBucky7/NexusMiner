#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sieve.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


#define checkCudaErrors(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

namespace nexusminer {
    namespace gpu {

        
        
        //global variables used by all kernels
        __device__ uint32_t* sieving_primes;
        __device__ uint32_t sieving_prime_count[1];
        __device__ uint32_t* multiples;
        __device__ uint32_t* starting_multiples;
        __device__ uint32_t* prime_mod_inverses;
        __device__ uint8_t* wheel_indices;
        __device__ uint8_t* sieve_global;  //the result of the sieve is stored here
        __device__ uint32_t sieve_global_size[1];  //the size of the sieve in bytes
        uint32_t sieving_prime_count_host;


        //return the offset from x to the next integer multiple of n greater than x that is not divisible by 2, 3, or 5.  
        //x must be a multiple of the primorial 30 and n must be a prime greater than 5.
        template <typename T1, typename T2>
        __device__ T2 get_offset_to_next_multiple(T1 x, T2 n)
        {
            T2 m = n - static_cast<T2>(x % n);
            if (m % 2 == 0)
            {
                m += n;
            }
            while (m % 3 == 0 || m % 5 == 0)
            {
                m += 2 * n;
            }
            return m;
        }

        __constant__ const int sieve30_gaps[]{ 6,4,2,4,2,4,6,2 };

        __constant__ const int sieve30_index[]
            { -1,0,-1,-1,-1,-1,-1, 1, -1, -1, -1, 2, -1, 3, -1, -1, -1, 4, -1, 5, -1, -1, -1, 6, -1, -1, -1, -1, -1, 7 };  //reverse lookup table (offset mod 30 to index)


        //seive kernel

        __global__ void do_sieve(uint64_t sieve_start_offset)
        {
            uint32_t segment_size = kernel_sieve_size / 8 * 30;

            //local shared copy of the sieve
            __shared__ uint8_t sieve[kernel_sieve_size];

            uint64_t block_id = blockIdx.x;
            uint64_t index = threadIdx.x;
            uint64_t stride = blockDim.x;
            uint64_t num_blocks = gridDim.x;
           
            uint64_t segments = kernel_segments_per_block;
            uint64_t sieve_results_index = block_id * kernel_sieve_size_per_block;

            uint64_t primes_per_block = (sieving_prime_count[0] + stride - 1) / stride;
            
            //each block sieves a different region
            uint64_t start_offset = sieve_start_offset + block_id * kernel_sieve_size_per_block / 8 * 30;
            
            int wheel_index;
            int next_wheel_gap;
            uint64_t j;
            uint64_t k;
            for (int s = 0; s < segments; s++)
            {
                //everyone in the block initialize part of the shared sieve
                for (int j1 = index; j1 < kernel_sieve_size; j1 += stride)
                {
                    if (j1 < kernel_sieve_size)
                        sieve[j1] = 1;
                }

                __syncthreads();
                for (uint32_t i = index; i < sieving_prime_count[0]; i += stride)
                {
                    if (i < sieving_prime_count[0])
                    {
                        
                        k = sieving_primes[i];
                        //get aligned to this region
                        if (s == 0)
                        {
                            j = starting_multiples[i];
                            //the first time through we need to calculate the starting offsets
                            if (start_offset >= j)
                                j = get_offset_to_next_multiple(start_offset - j, sieving_primes[i]);
                            else
                                j -= start_offset;
                            
                            //where are we in the wheel
                            wheel_index = sieve30_index[(prime_mod_inverses[i] * j) % 30];
                        }
                        else
                        {
                            j = multiples[block_id* sieving_prime_count[0] +i];
                            wheel_index = wheel_indices[block_id * sieving_prime_count[0] + i];
                        }
                        next_wheel_gap = sieve30_gaps[wheel_index];
                        
                        while (j < segment_size)
                        {
                            //cross off a multiple of the sieving prime
                            uint64_t sieve_index = (j / 30) * 8 + sieve30_index[j % 30];
                            sieve[sieve_index] = 0;
                            //increment the next multiple of the current prime (rotate the wheel).
                            j += k * next_wheel_gap;
                            wheel_index = (wheel_index + 1) % 8;
                            next_wheel_gap = sieve30_gaps[wheel_index];
                        }
                        //save the starting multiple and wheel index for the next segment
                        multiples[block_id * sieving_prime_count[0] + i] = j - segment_size;
                        wheel_indices[block_id * sieving_prime_count[0] + i] = wheel_index;
                    }
                }
                __syncthreads();
                //copy the sieve results to global memory
                
                for (int j2 = index; j2 < kernel_sieve_size; j2 += stride)
                {
                    if (j2 < kernel_sieve_size)
                    {
                        sieve_global[sieve_results_index + j2] = sieve[j2];
                    }
                }
                
                sieve_results_index += kernel_sieve_size;
                __syncthreads();
                
            }
        }


        void run_sieve(uint64_t sieve_start_offset, uint8_t sieve[])
        {
            
            uint8_t* d_sieve;

            //run the kernel
            do_sieve <<<num_blocks, threads_per_block >>> (sieve_start_offset);

            checkCudaErrors(hipDeviceSynchronize());

            checkCudaErrors(hipMemcpyFromSymbol(&d_sieve, HIP_SYMBOL(sieve_global), sizeof(uint8_t*)));
            checkCudaErrors(hipMemcpy(sieve, d_sieve, sieve_total_size * sizeof(uint8_t), hipMemcpyDeviceToHost));
        }

        //allocate global memory and load values used by the sieve to the gpu 
        void load_sieve(uint32_t primes[], uint32_t prime_count, uint32_t starting_multiples_host[],
            uint32_t prime_mod_inverses_host[], uint32_t sieve_size)
        {
            //device memory pointers
            uint32_t* d_sieving_primes;
            uint32_t* d_starting_multiples;
            uint32_t* d_prime_mod_inverses;
            uint8_t* d_sieve;
            uint32_t* d_multiples;
            uint8_t* d_wheel_indices;
            
            sieving_prime_count_host = prime_count;
            checkCudaErrors(hipSetDevice(0));
            //allocate memory on the gpu
            checkCudaErrors(hipMalloc(&d_sieving_primes, prime_count * sizeof(uint32_t)));
            checkCudaErrors(hipMalloc(&d_starting_multiples, prime_count * sizeof(uint32_t)));
            checkCudaErrors(hipMalloc(&d_prime_mod_inverses, prime_count * sizeof(uint32_t)));
            checkCudaErrors(hipMalloc(&d_sieve, sieve_size * sizeof(uint8_t)));
            checkCudaErrors(hipMalloc(&d_multiples, prime_count * num_blocks * sizeof(uint32_t)));
            checkCudaErrors(hipMalloc(&d_wheel_indices, prime_count * num_blocks * sizeof(uint8_t)));


            //copy data to the gpu
            checkCudaErrors(hipMemcpy(d_sieving_primes, primes, prime_count * sizeof(uint32_t), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_starting_multiples, starting_multiples_host, prime_count * sizeof(uint32_t), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_prime_mod_inverses, prime_mod_inverses_host, prime_count * sizeof(uint32_t), hipMemcpyHostToDevice));

            //point the global device variable to the allocated memory
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sieving_primes), &d_sieving_primes, sizeof(uint32_t*)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(starting_multiples), &d_starting_multiples, sizeof(uint32_t*)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(prime_mod_inverses), &d_prime_mod_inverses, sizeof(uint32_t*)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sieve_global), &d_sieve, sizeof(uint8_t*)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sieving_prime_count), &prime_count, sizeof(uint32_t)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sieve_global_size), &sieve_size, sizeof(uint32_t)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wheel_indices), &d_wheel_indices, sizeof(uint8_t*)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(multiples), &d_multiples, sizeof(uint32_t*)));

        
        }

        void free_sieve()
        {
            uint32_t* d_sieving_primes;
            uint32_t* d_starting_multiples;
            uint32_t* d_prime_mod_inverses;
            uint32_t* d_multiples;
            uint8_t* d_wheel_indices;
            uint8_t* d_sieve;

            checkCudaErrors(hipMemcpyFromSymbol(&d_sieving_primes, HIP_SYMBOL(sieving_primes), sizeof(uint32_t*)));
            checkCudaErrors(hipMemcpyFromSymbol(&d_starting_multiples, HIP_SYMBOL(starting_multiples), sizeof(uint32_t*)));
            checkCudaErrors(hipMemcpyFromSymbol(&d_prime_mod_inverses, HIP_SYMBOL(prime_mod_inverses), sizeof(uint32_t*)));
            checkCudaErrors(hipMemcpyFromSymbol(&d_multiples, HIP_SYMBOL(multiples), sizeof(uint32_t*)));
            checkCudaErrors(hipMemcpyFromSymbol(&d_wheel_indices, HIP_SYMBOL(wheel_indices), sizeof(uint8_t*)));
            checkCudaErrors(hipMemcpyFromSymbol(&d_sieve, HIP_SYMBOL(sieve_global), sizeof(uint8_t*)));

            checkCudaErrors(hipFree(d_sieving_primes));
            checkCudaErrors(hipFree(d_starting_multiples));
            checkCudaErrors(hipFree(d_wheel_indices));
            checkCudaErrors(hipFree(d_multiples));
            checkCudaErrors(hipFree(d_prime_mod_inverses));
            checkCudaErrors(hipFree(d_sieve));


        }
    }
}